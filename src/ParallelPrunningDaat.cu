#include "hip/hip_runtime.h"
/*
 * ParallelPrunningDaat.cu
 *
 *  Created on: 08/12/2017
 *      Author: roussian
 */
#include "ParallelPrunningDaat.cuh"
#include "DeviceFunctions.cuh"
#include "UnityTest.cuh"
#include <stdio.h>


__device__ volatile float globalThreshold = 0.0;
__device__ volatile float globalThresholdBatch[500];
__device__ int globalCount=0;

__global__ void mergeTopkLists_v3(float *dTopkScoreList,
								  int *iTopkDocList,
								  int iTopk,
								  int iMergeNumber,
								  int iSkipTopkBetweenMerges,
								  int iSkipTopkBetweenBlocks,
								  int iTotalElementos){

//	if(blockIdx.x != 74)
//		return;

	//Peguei o doc idblock 4 skipBlock 32 skipMerges 16 na posição 18559 !
//	if(iSkipTopkBetweenBlocks == 4 && iSkipTopkBetweenMerges == 2)
//	if(blockIdx.x != 87)
//		return;

	__shared__ documentTopkList documentTopkSharedList_1;
	__shared__ documentTopkList documentTopkSharedList_2;
	__shared__ documentTopkList documentTopkSharedList_Partial;

	__shared__ short int halfNumberTopk;// = iTopk >> 1;
	__shared__ short int iTopkPosition;// = iTopk - 1; // Começa no índice 0 e vai iTopk - 1
	__shared__ short int halfPositionNumberTopk;// = iTopkPosition >> 1;

	if(THREAD_MASTER){
		halfNumberTopk = iTopk >> 1;
	}else if(THREAD_MASTER_2){
		iTopkPosition = iTopk - 1;
		halfPositionNumberTopk = iTopkPosition >> 1;
	}

	int warpIndex = threadIdx.x >> 5;
	int threadWarpId =  threadIdx.x - (warpIndex << 5); // (threadIdx.x & 0x1f); //threadIdx.x % 32;
	int isOdd = warpIndex & 1; //Verifica se o IdWarp é ímpar
	int numberThreadsInList = ((blockDim.x >> 6) << 5); // (#Block/Tamanho da Warp--2⁵) / 2; ---> isso pq metade do # de warps trabalham sobre uma lista
	warpIndex = warpIndex >> 1; //Isso pois as warps são divididas por impar e par. Então, se o idWarp é 5, então o novo id é 2

	__syncthreads();
//	   int proportion = iTopk / blockDim.x; //K é múltiplo do numero de threads por bloco

	int offset = iTopkPosition; //a Posição que cada thread irá inserir o seu elemento
	//A posição das threads nas listas --- half + (pos. da warp * #threads dentro da warp) + id
	int indexInMemShared = halfNumberTopk + ( warpIndex << 5) + threadWarpId;//(iTopk >> 1) + ((warpId >> 1) << 5) + threadWarpId;///half + (pos. da warp * #threads na warp) + id

	float score_1, score_2;
	float *ownScorePtr, *workListPtr;
	int *ownDocId;

	int position;
	int index_1, index_2, indexLocal;
//	int isEndPart;

	//Obtém a posição inicial que a thread irá inserir na lista final.
	offset -=  (iTopkPosition - indexInMemShared ) << 1;//A multiplicação por 2 é por causa das duas listas

//	if(blockIdx.x == 203)
//		printf("Oi!\n");
	__syncthreads();

	//As listas estão alinhadas em uma lista, por isso que o indice tem que seguir para
	//a próxima parte não processada referente ao bloco
	//Um merge pega 2 listas ou 1 lista + Resultado anterior
	index_1 = blockIdx.x * iTopk * iSkipTopkBetweenBlocks + threadIdx.x;//blockIdx.x * iTopk * (iMergeNumber + 1) * iSkipBetweenMerge + threadIdx.x;
	index_2 = index_1 + iTopk * iSkipTopkBetweenMerges;//index_1 + iTopk * iSkipTopkBetweenMerges;
//	isEndPart = 0;//index_2 > totalElements;

	indexLocal = threadIdx.x;
	//O número de threads por bloco pode ser menor que K
	while(indexLocal < iTopk){

		documentTopkSharedList_1.id[indexLocal] = iTopkDocList[index_1];
		documentTopkSharedList_1.score[indexLocal] = dTopkScoreList[index_1];

//		if(iTopkDocList[index_1] == 46517642)
//			printf("Peguei o doc idblock %d skipBlock %d skipMerges %d na posição %d !\n",
//					blockIdx.x, iSkipTopkBetweenBlocks,iSkipTopkBetweenMerges,index_1);

		index_1 += blockDim.x;
		indexLocal += blockDim.x;
	}



//	if(THREAD_MASTER && blockIdx.x == 0 && iSkipTopkBetweenBlocks >= 2048)
//		printf("idblock %d skipBlock %d skipMerges %d na posição inicial %d %d!\n",
//				blockIdx.x, iSkipTopkBetweenBlocks,iSkipTopkBetweenMerges,blockIdx.x * iTopk * iSkipTopkBetweenBlocks,
//				blockIdx.x * iTopk * iSkipTopkBetweenBlocks + iTopk * iSkipTopkBetweenMerges);

	__syncthreads();

//	if(THREAD_MASTER && blockIdx.x == 0){
//		printf("First List - ");
//		for (int i = 0; i < iTopk; ++i) {
//			printf(" %.2f ", documentTopkSharedList_1.score[i]);
//		}
//		printf("\n");
//	}

	for (int globalRound = 0; globalRound < iMergeNumber; ++globalRound) {
		//O número de threads por bloco pode ser menor que K
		indexLocal = threadIdx.x;
		while(indexLocal < iTopk){
			if(index_2 >= iTotalElementos || index_2 < 0){
				documentTopkSharedList_2.id[indexLocal] = 0;
				documentTopkSharedList_2.score[indexLocal] = 0;
			}else{
				documentTopkSharedList_2.id[indexLocal] = iTopkDocList[index_2];
				documentTopkSharedList_2.score[indexLocal] = dTopkScoreList[index_2];
//				if(iTopkDocList[index_2] == 46517642)
//					printf("Peguei o doc idblock %d skipBlock %d skipMerges %d na posição %d !\n",
//							blockIdx.x, iSkipTopkBetweenBlocks,iSkipTopkBetweenMerges,index_2);
			}
			index_2 += blockDim.x;
			indexLocal += blockDim.x;
		}

		__syncthreads();

		if(!isOdd){//As threads das Warps com ids par trabalham sobre os maiores elementos da mesma posição.

			do {//Esse bloco de instruções trabalha somente com dados que estão na memória compartilhada
				score_1 = documentTopkSharedList_1.score[indexInMemShared];
				score_2 = documentTopkSharedList_2.score[indexInMemShared];
				//Escolhe o maior elemento de uma mesma posição e a lista, a que tiver o menor elemento, que irá pecorrer.
				if(score_1 >= score_2){
					ownScorePtr = &score_1;
					ownDocId = &documentTopkSharedList_1.id[indexInMemShared];

//					if(*ownDocId == 46517642){// && blockIdx.x == 5
//						printf("1 - blockId %d threadId %d\n", blockIdx.x, threadIdx.x);
//					}

					workListPtr = documentTopkSharedList_2.score; //A lista de trabalho sempre é a lista do menor elemento
				}
				else{
					ownScorePtr = &score_2;
					ownDocId = &documentTopkSharedList_2.id[indexInMemShared];

//					if(*ownDocId == 46517642){// && blockIdx.x == 5){
//						printf("1.1 - blockId %d threadId %d\n", blockIdx.x, threadIdx.x);
//					}
					workListPtr = documentTopkSharedList_1.score;
				}

				if(score_1 != score_2){
					//Busca atualizar o offset, i.e., procura o 1º elemento maior
					position = indexInMemShared;//Define a posição início para fazer as comparações (//Se workscore está em A[i] e A[i] < B[i], então A[i] < B[i + (1,2,3...)])
					while( (position+1 < iTopk) && (*ownScorePtr > workListPtr[position+1]) ){
						offset++; //Ao encontrar um elemento menor, ele irá aumentar a posição que irá inserir o seu elemento
						position++;
					}
				}

				//Insere os maiores elementos das listas, i.e., os elementos mais a direita da lista dos top-k
				documentTopkSharedList_Partial.score[offset] = *ownScorePtr;
				documentTopkSharedList_Partial.id[offset] = *ownDocId;

				//Redefine as variáveis para inicializar outro bloco de dados que está na memória compartilhada
				indexInMemShared += numberThreadsInList;
				offset = iTopkPosition - ((iTopkPosition - indexInMemShared ) << 1); //Reinicia o offset

			} while (indexInMemShared < iTopk);

		}else{
			int count; //Quantos elementos irá buscar;
			float *ownScoreListPtr;

			do{//Esse bloco de instrução trabalha somente com dados que estão na memória compartilhada
				offset--;//É o menor elemento entre dois elementos (mesmo índice)

				score_1 = documentTopkSharedList_1.score[indexInMemShared];
				score_2 = documentTopkSharedList_2.score[indexInMemShared];
				//Escolhe o menor elemento de uma mesma posição e a lista, a que tiver o maior elemento, que irá pecorrer.
				if(score_1 < score_2){
					ownScorePtr = &score_1;
					ownDocId = &documentTopkSharedList_1.id[indexInMemShared];
//					if(*ownDocId == 46517642){
//						printf("2 -  blockId %d threadId %d\n", blockIdx.x, threadIdx.x);
//					}
					workListPtr = documentTopkSharedList_2.score;
					ownScoreListPtr = documentTopkSharedList_1.score;
				}
				else{//Entra igual(se for igual, entao o score_2 é selecionado) ou menor
					ownScorePtr = &score_2;
					ownDocId = &documentTopkSharedList_2.id[indexInMemShared];

//					if(*ownDocId == 46517642){
//						printf("2.1 - blockId %d threadId %d\n", blockIdx.x, threadIdx.x);
//					}
					workListPtr = documentTopkSharedList_1.score;
					ownScoreListPtr = documentTopkSharedList_2.score;
				}
				//Duas possibilidades podem ocorrer: (1) O elemento adquirido está entre os k maiores elementos
				//(2) o elemento não está entre os k maiore elementos
				if(*ownScorePtr >= workListPtr[halfPositionNumberTopk]){//Compara-se com o elemento que está na metade//if(*ownScorePtr > workListPtr[iTopkPosition >> 1]){//Compara-se com o elemento que está na metade

					position = indexInMemShared;// - 1;
					while(  (position - 1 > 0) && (*ownScorePtr < workListPtr[position-1]) ){
						offset--;
						position--;
					}

					documentTopkSharedList_Partial.score[offset] = *ownScorePtr;
					documentTopkSharedList_Partial.id[offset] = *ownDocId;

				}else{

					offset -= indexInMemShared - halfNumberTopk;//(iTopk >> 1); Subtrai da metade do número das posições e não do índice máx, pois já ouve uma subtração do conjunto dos maiores elementos
					count = halfPositionNumberTopk - offset; //Quantos elementos irá buscar;

					float *aux;
					int posWork, posOwn;
					int *docIdOwn, *docIdWork;
					if(ownScoreListPtr[iTopkPosition] >= workListPtr[halfPositionNumberTopk] ){
						ownScorePtr = &ownScoreListPtr[iTopkPosition];
						posOwn = iTopkPosition;
						posWork = halfPositionNumberTopk;
						if(ownScoreListPtr == documentTopkSharedList_2.score){
							docIdOwn = documentTopkSharedList_2.id;
							docIdWork = documentTopkSharedList_1.id;
						}else{
							docIdOwn = documentTopkSharedList_1.id;
							docIdWork = documentTopkSharedList_2.id;
						}
					}else{
						ownScorePtr = &workListPtr[halfPositionNumberTopk];
						aux = ownScoreListPtr;
						ownScoreListPtr = workListPtr;
						workListPtr = aux;
						posOwn = halfPositionNumberTopk;
						posWork = iTopkPosition;

						if(workListPtr == documentTopkSharedList_2.score){
							docIdWork = documentTopkSharedList_2.id;
							docIdOwn = documentTopkSharedList_1.id;
						}else{
							docIdWork = documentTopkSharedList_1.id;
							docIdOwn = documentTopkSharedList_2.id;
						}
					}

					while(count > 0){

						while((workListPtr[posWork] <= ownScoreListPtr[posOwn]) && (count > 0)){
							posOwn--;
							count--;
						}
//						posOwn++;

						if(count == 0){
							ownScorePtr = &ownScoreListPtr[posOwn];
							ownDocId = &docIdOwn[posOwn];
						}else{

							while((ownScoreListPtr[posOwn] <= workListPtr[posWork]) && count > 0){
								posWork--;
								count--;
							}
//							posWork++;
							if(count == 0){
								ownScorePtr = &workListPtr[posWork];
								ownDocId = &docIdWork[posWork];
							}
						}
					}

					documentTopkSharedList_Partial.score[offset] = *ownScorePtr;
					documentTopkSharedList_Partial.id[offset] = *ownDocId;
				}

				indexInMemShared += numberThreadsInList;
				offset = iTopkPosition - ((iTopkPosition - indexInMemShared ) << 1); //Reinicia o offset

			} while(indexInMemShared < iTopk);

		}//IF-ELSE ODD

		__syncthreads();

		indexLocal = threadIdx.x;
		while(indexLocal < iTopk){

			documentTopkSharedList_1.id[indexLocal] = documentTopkSharedList_Partial.id[indexLocal];
			documentTopkSharedList_1.score[indexLocal] = documentTopkSharedList_Partial.score[indexLocal];

			indexLocal += blockDim.x;
		}
		// -1 por causa do avanço realizado pelas threads para o próximo bloco de topk documentos no último loop
		index_2 += iTopk * (iSkipTopkBetweenMerges - 1);
		indexInMemShared = halfNumberTopk + ( warpIndex << 5) + threadWarpId;
		offset = iTopkPosition - ((iTopkPosition - indexInMemShared ) << 1); //Reinicia o offset

//		checkMerge_Sorting_Documents(documentTopkSharedList_Partial, iSkipTopkBetweenMerges, iSkipTopkBetweenBlocks, iTopk);
	}

	__syncthreads();

	index_1 = blockIdx.x * iTopk * iSkipTopkBetweenBlocks + threadIdx.x;
	indexLocal = threadIdx.x;
	while(indexLocal < iTopk){
//		if(isEndPart)
//			break;
//		if(documentTopkSharedList_Partial.id[indexLocal] == 46517642)
//			printf("Entregando o doc idblock %d skipBlock %d skipMerges %d em %d!\n",
//					blockIdx.x, iSkipTopkBetweenBlocks,iSkipTopkBetweenMerges,index_1);

		if(documentTopkSharedList_Partial.score[indexLocal] != 0.0){
			iTopkDocList[index_1] = documentTopkSharedList_Partial.id[indexLocal];
			dTopkScoreList[index_1] = documentTopkSharedList_Partial.score[indexLocal];
		}

		indexLocal += blockDim.x;
		index_1 += blockDim.x;
	}

//	__syncthreads();
//
//	if(THREAD_MASTER && blockIdx.x == 0){
//		printf("Final List - ");
//		for (int i = 0; i < iTopk; ++i) {
//			printf(" %.2f ", documentTopkSharedList_Partial.score[i]);
//		}
//		printf("\n");
//	}

}

__global__ void matchWandParallel_FIXED_2(const int* iDocIdList, const unsigned short int* iFreqList,
								  const float *dUBlist, const float *dIdfList,
								  const int *iDocLenghtList,
								  const short int iTermNumber, int *iTopkDocListGlobal,
								  float *dTopkScoreListGlobal, const float dAverageDocumentLength,
								  const int iGlobalRoundNumber,// const int iBlockRoundNumber,
								  const short int iTopK,
								  const float iInitialThreshold,const int* d_iDocNumberByTermList){

		int count=0;

		__shared__ pivot sharedPivot;
		__shared__ finger fingers[TERM_NUMBER];

		__shared__ postingList postingLists[TERM_NUMBER];
		__shared__ documentTopkList documentTopk;

		__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];

		__shared__ int iGlobalInitialPosition;

		__shared__ float score;
		__shared__ bool isValidCandidate;

		int positionInitialInTermPostingList;
		float thresholdLocal = iInitialThreshold;
		int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
		int localIndex;

		if(THREAD_MASTER){
			iGlobalInitialPosition = blockDim.x  * blockIdx.x * iGlobalRoundNumber;
			documentTopk.padding = iTopK;
		}

		if(thresholdLocal < globalThreshold)
			thresholdLocal = globalThreshold;

		//Inicializa a lista de Score e Documentos dos Topk
		//Considero que o Top_K seja um número múltiplo do tamanho do bloco
		for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
			documentTopk.id[localIndex] = -1;
			documentTopk.score[localIndex] = 0.0;
		}

		__syncthreads();

		for (int globalRound = 0; globalRound < iGlobalRoundNumber; ++globalRound) {
			positionInitialInTermPostingList = 0;
			for (int termIndex = 0; termIndex < iTermNumber; ++termIndex) {
				localIndex = threadIdx.x;
				globalIndex = positionInitialInTermPostingList + iGlobalInitialPosition + localIndex;
				while(localIndex < DOC_QUANTITY_IN_MEMORY){//(globalIndex < d_iDocNumberByTermList[termIndex] && localIndex < DOC_QUANTITY_IN_MEMORY){

					if(globalIndex < d_iDocNumberByTermList[termIndex] + positionInitialInTermPostingList){
						postingLists[termIndex].docId[localIndex] =  iDocIdList[globalIndex];//[positionInitialInTermPostingList + globalIndex];
						postingLists[termIndex].freq[localIndex] = iFreqList[globalIndex];
						postingLists[termIndex].docLenght[localIndex] = iDocLenghtList[globalIndex];
					}
					else{
						postingLists[termIndex].docId[localIndex] = NO_MORE_DOC;
					}
					localIndex += blockDim.x;
					globalIndex += blockDim.x;
				}

				if(THREAD_MASTER){
					fingers[termIndex].docId = postingLists[termIndex].docId[0];
					fingers[termIndex].position = (fingers[termIndex].docId == NO_MORE_DOC) ? NO_VALID_POSITION : 0 ;
				}

				positionInitialInTermPostingList += d_iDocNumberByTermList[termIndex];
			}

			__syncthreads();

//			if(fingers[0].docId == 16563866)
//				printf("Oi!");

			//Sort the terms in non decreasing order of DID
			sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

//			__syncthreads();

			//Select term pivot
			if(THREAD_MASTER){
				selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			}else if(THREAD_MASTER_2){
				score = 0.0;
			}

			__syncthreads();

			int padding;
			int threadIdInWarp = (threadIdx.x & 0x1f);
			int idWarp = ((blockDim.x >> 5) == 1 ) ? 1 :  threadIdx.x >> 5;

			while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

				if(THREAD_MASTER)
					isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);

				count++;
				padding = documentTopk.padding;

				__syncthreads();

				if(isValidCandidate){
					//Avaliação Completa
					if(threadIdx.x < iTermNumber){
						fullScore_3_1(&score, fingers[sharedPivot.idTerm].docId, iOrderedTermSharedList,
									  fingers,postingLists, dIdfList, dAverageDocumentLength);
					}

					__syncthreads();

//					if(padding != 0 || thresholdLocal < score){
					if(thresholdLocal < score){
						thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId,score,padding);
//						thresholdLocal = documentTopk.score[0];
					}

					if(idWarp == 1 && threadIdInWarp < iTermNumber ){
						advancePivoTermFinger_4(sharedPivot,fingers, postingLists,DOC_QUANTITY_IN_MEMORY,threadIdInWarp);
					}
				}
				else{
					 advanceDocIdOfPredecessorTerm_4(postingLists,
												   iOrderedTermSharedList,
												   fingers,sharedPivot,fingers[sharedPivot.idTerm].docId,
												   DOC_QUANTITY_IN_MEMORY);
				}

//				if(fingers[0].docId == 16563866)
//					printf("Oi!");

				__syncthreads();

				sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

//				__syncthreads();

				//Select term sharedPivot
				if(THREAD_MASTER){
					selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
				}else if(THREAD_MASTER_2){
					score = 0.0;
				}

				__syncthreads();
			}

			if(THREAD_MASTER){
				iGlobalInitialPosition += DOC_QUANTITY_IN_MEMORY;
			}

			if (SHAREDTHESHOLD == 1){//SHARED_READ
				if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > globalThreshold){
//					atomicMax(&globalThreshold,thresholdLocal);
					globalThreshold = thresholdLocal;
				}

				if(thresholdLocal < globalThreshold){
					thresholdLocal = globalThreshold;
				}
			}else if (SHAREDTHESHOLD == 2){ //TSHARED_WRITEREAD
				if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > globalThreshold){
//					atomicMaxD(&globalThreshold,thresholdLocal);
					globalThreshold = thresholdLocal;
				}

				if((documentTopk.padding < (iTopK >> 1)))
					if(thresholdLocal < globalThreshold){
						thresholdLocal = globalThreshold;
					}
			}

			__syncthreads();
		}

		sortLocalTopkDocAndStoreInGlobal(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);

		if(THREAD_MASTER)
				atomicAdd(&globalCount,count);

		if(THREAD_MASTER)
			printf("-----%d----", globalCount);

}

__global__ void matchWandParallel_VARIABLE_Batch_Block_3(const int* iDocIdList, const unsigned short int* iFreqList,
														const float *dUBlistGlobal, const float *dIdfListGlobal, const int *iDocLenghtList,
														const short int* iTermNumberByQuery, int *iTopkDocListGlobal,
														float *dTopkScoreListGlobal, const float dAverageDocumentLengthGlobal,
														const short int iTopK, const float iInitialThreshold,
														const int* iQueryTerms, const long long* ptrInitPostingList,
														int* ptrQueryPositions,int *iDocNumberByTermListGlobal){





	__shared__ int queryPosition;
	__shared__ pivot sharedPivot;
	__shared__ finger fingers[TERM_NUMBER];

	__shared__ documentTopkList documentTopk;


	__shared__ postingList2 postings[TERM_NUMBER];
	__shared__ int positionInShared[TERM_NUMBER];

	__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];
	__shared__ long long iSharedPositionInitialInList[TERM_NUMBER];
	__shared__ float dUBlist[TERM_NUMBER];
	__shared__ float dIdfList[TERM_NUMBER];
	__shared__ float dAverageDocumentLength;

	__shared__ int iDocNumberByTermList[TERM_NUMBER];

	__shared__ float score;
	__shared__ bool isValidCandidate;
	__shared__ int docCurrent;
	__shared__ limitDocId limitDoc;
	__shared__ short int iTermNumber;

	int count;
	int padding;

 	float thresholdLocal;// = iInitialThreshold;
 	thresholdLocal = iInitialThreshold;

	int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
	int localIndex;

//	int count = 0;

	if(THREAD_MASTER){
		documentTopk.padding = iTopK;
		dAverageDocumentLength = dAverageDocumentLengthGlobal;
		limitDoc.secondMaxDocId = -1;
		iTermNumber = iTermNumberByQuery[blockIdx.x];
	}

	__syncthreads();

	if(threadIdx.x < iTermNumber){
		queryPosition = ptrQueryPositions[blockIdx.x];
		int idTerm = iQueryTerms[queryPosition + threadIdx.x];

		iDocNumberByTermList[threadIdx.x] = iDocNumberByTermListGlobal[idTerm];

		dUBlist[threadIdx.x] = dUBlistGlobal[idTerm]*1.0;//[threadIdx.x];
		dIdfList[threadIdx.x] = dIdfListGlobal[idTerm];//[threadIdx.x];

		iSharedPositionInitialInList[threadIdx.x] = ptrInitPostingList[idTerm];
		positionInShared[threadIdx.x] = -1;
	}

	//Inicializa a lista de Score e Documentos dos Topk
	//Considero que o Top_K seja um número múltiplo do tamanho do bloco
	#pragma unroll 4
	for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
		documentTopk.id[localIndex] = -1;
		documentTopk.score[localIndex] = 0.0;
	}

	//Define o max e o min
	if(threadIdx.x < iTermNumber){
		int docAmount = iDocNumberByTermList[threadIdx.x];
		globalIndex = 0;

		int maxDoc;

		if(THREAD_MASTER) limitDoc.minDocId = 0;

		globalIndex = docAmount;

		maxDoc =  iDocIdList[iSharedPositionInitialInList[threadIdx.x] +  globalIndex - 1];

		atomicMax(&(limitDoc.secondMaxDocId), maxDoc);

		fingers[threadIdx.x].docId = iDocIdList[iSharedPositionInitialInList[threadIdx.x]];
		fingers[threadIdx.x].position = iSharedPositionInitialInList[threadIdx.x];
	}

//	__syncthreads();

	int pos;
	for (int termId = 0; termId < iTermNumber; ++termId) {
		for (int localIndex = threadIdx.x; localIndex < DOC_QUANTITY_IN_MEMORY; localIndex+=blockDim.x) {
			pos = fingers[termId].position+localIndex+1;
			if(pos < iSharedPositionInitialInList[termId] + iDocNumberByTermList[termId]){
				postings[termId].docId[localIndex] = iDocIdList[pos];
			}else{
				postings[termId].docId[localIndex] = NO_MORE_DOC;
			}
		}
	}

	sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

	if(THREAD_MASTER){
		selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
		docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
		score = 0.0;
	}

	__syncthreads();

	while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

		isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
		count++;

//		if(count == 2559)
//			printf("Oi");

		if(isValidCandidate){
			if(threadIdx.x < iTermNumber){
		 		int termId = iOrderedTermSharedList[threadIdx.x];
		 		float scoreL = 0.0;
		 		if(fingers[termId].docId == fingers[sharedPivot.idTerm].docId){
		 			scoreL = scoreTf_Idf(iFreqList[fingers[termId].position],
										iDocLenghtList[fingers[termId].position],
										dIdfList[termId],dAverageDocumentLength,1.0);
		 		}

		 		float aux = 0;
		 		for (int i = 0; i < TERM_NUMBER; ++i) {
		 			aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
		 		}

		 		if(THREAD_MASTER) score = aux;
//		 		atomicAdd(&score,scoreL);
			}

			padding = documentTopk.padding;

			__syncthreads();

/*				If the heap is not full
			the candidate is inserted into the heap. If the heap is full
			and the new score is larger than the minimum score in the
			heap, the new document is inserted into the heap, replacing
			the one with the minimum score.

*/
			if(padding != 0 || thresholdLocal < score ){
				thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
			}

			for (int idTerm = 0; idTerm < iTermNumber; ++idTerm) {
				if(THREAD_MASTER && positionInShared[idTerm] == -1)
					fingers[idTerm].position++;

				if(docCurrent == fingers[idTerm].docId){
					fingers[idTerm].docId = NO_MORE_DOC;

					int docIdLocal, localIndex=0;
					for (localIndex = threadIdx.x + positionInShared[idTerm]; localIndex < DOC_QUANTITY_IN_MEMORY; localIndex+=blockDim.x) {
						docIdLocal = postings[idTerm].docId[localIndex];
						if(docIdLocal > docCurrent && docIdLocal != NO_MORE_DOC){
							if(localIndex == 0 || (postings[idTerm].docId[localIndex-1] <= docCurrent)){
								fingers[idTerm].docId = docIdLocal;
								fingers[idTerm].position += localIndex - positionInShared[idTerm];
								positionInShared[idTerm] = localIndex;
							}
							break;
						}
					}
				}
			}
		}
		else{
			int idTerm;
			for (int j = 0; j < sharedPivot.positionInOrderedList; ++j) {
				idTerm = iOrderedTermSharedList[sharedPivot.positionInOrderedList];

				if(docCurrent == fingers[threadIdx.x].docId)
					break;

				fingers[idTerm].docId = NO_MORE_DOC;

				if(THREAD_MASTER && positionInShared[idTerm] == -1)
					fingers[idTerm].position++;

				int docIdLocal, localIndex=0;
				for (localIndex = threadIdx.x+positionInShared[idTerm]; localIndex < DOC_QUANTITY_IN_MEMORY; localIndex+=blockDim.x) {
					docIdLocal = postings[idTerm].docId[localIndex];
					if(docIdLocal >= docCurrent && docIdLocal != NO_MORE_DOC){
						if(localIndex == 0 || (postings[idTerm].docId[localIndex-1] < docCurrent)){
							fingers[idTerm].docId = docIdLocal;
							fingers[idTerm].position += localIndex - positionInShared[idTerm];
							positionInShared[idTerm] =  localIndex;
						}
						break;
					}
				}
			}
		}

		__syncthreads();

		for (int termId = 0; termId < iTermNumber; ++termId) {
			if(fingers[termId].docId == NO_MORE_DOC && fingers[termId].position != NO_VALID_POSITION){
				int pos, localIndex;
				for (localIndex = threadIdx.x; localIndex < DOC_QUANTITY_IN_MEMORY; localIndex+=blockDim.x) {
					pos = fingers[termId].position+localIndex+1;
					if(pos < iSharedPositionInitialInList[termId] + iDocNumberByTermList[termId]){
						postings[termId].docId[localIndex] = iDocIdList[pos];
					}else{
						postings[termId].docId[localIndex] = NO_MORE_DOC;
					}
				}

				if(THREAD_MASTER && postings[termId].docId[0] == NO_MORE_DOC)
					fingers[termId].position = NO_VALID_POSITION;
				else{
					fingers[termId].docId = postings[termId].docId[0];
					positionInShared[termId] = -1;
					fingers[termId].position = pos;
				}
			}
		}
		__syncthreads();

		//Sort the terms in non decreasing order of DID
		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

		//Select term pivot
		if(THREAD_MASTER){
			selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
			score = 0.0;
		}
		__syncthreads();
	}

	if(blockIdx.x==499 && THREAD_MASTER)
		printf("-----%d %d----", blockIdx.x, count);



	sortLocalTopkDocAndStoreInGlobal((float*)&(dTopkScoreListGlobal[blockIdx.x*iTopK]),(int*)&(iTopkDocListGlobal[blockIdx.x*iTopK]),iTopK,&documentTopk);

//		if(THREAD_MASTER)
////			atomicAdd(&globalCount,count);
////
//		if


}

__global__ void matchWandParallel_VARIABLE_Batch_Block_Test(const int* iDocIdList, const unsigned short int* iFreqList,
														const float *dUBlistGlobal, const float *dIdfListGlobal, const int *iDocLenghtList,
														const short int* iTermNumberByQuery, int *iTopkDocListGlobal,
														float *dTopkScoreListGlobal, const float dAverageDocumentLengthGlobal,
														const short int iTopK, const float iInitialThreshold,
														const int* iQueryTerms, const long long* ptrInitPostingList,
														int* ptrQueryPositions,int *iDocNumberByTermListGlobal,
														const int* iOrderQueryList){

	__shared__ int queryPosition;
	__shared__ pivot sharedPivot;
	__shared__ finger fingers[TERM_NUMBER];

	__shared__ documentTopkList documentTopk;

	__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];
	__shared__ long long iSharedPositionInitialInList[TERM_NUMBER];
	__shared__ float dUBlist[TERM_NUMBER];
	__shared__ float dIdfList[TERM_NUMBER];
	__shared__ float dAverageDocumentLength;

	__shared__ int iDocNumberByTermList[TERM_NUMBER];

	__shared__ float score;
	__shared__ bool isValidCandidate;
	__shared__ int docCurrent;
	__shared__ limitDocId limitDoc;
	__shared__ short int iTermNumber;

	int padding;

	float thresholdLocal;// = iInitialThreshold;
	thresholdLocal = iInitialThreshold;

	int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
	int localIndex;

//	int count = 0;

	if(THREAD_MASTER){
		documentTopk.padding = iTopK;
		dAverageDocumentLength = dAverageDocumentLengthGlobal;
//	}else if(THREAD_MASTER_2){
//		limitDoc.minDocId = -1;
		limitDoc.secondMaxDocId = -1;

		iTermNumber = iTermNumberByQuery[blockIdx.x];
	}

	__syncthreads();

	if(threadIdx.x < iTermNumber){
		queryPosition = ptrQueryPositions[iOrderQueryList[blockIdx.x]];
		int idTerm = iQueryTerms[queryPosition + threadIdx.x];

		iDocNumberByTermList[threadIdx.x] = iDocNumberByTermListGlobal[idTerm];

		dUBlist[threadIdx.x] = dUBlistGlobal[idTerm];//[threadIdx.x];
		dIdfList[threadIdx.x] = dIdfListGlobal[idTerm];//[threadIdx.x];

//		printf(" %.2f ",dUBlist[threadIdx.x]);
		iSharedPositionInitialInList[threadIdx.x] = ptrInitPostingList[idTerm];
	}

	//Inicializa a lista de Score e Documentos dos Topk
	//Considero que o Top_K seja um número múltiplo do tamanho do bloco
	#pragma unroll 4
	for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
		documentTopk.id[localIndex] = -1;
		documentTopk.score[localIndex] = 0.0;
	}

	//Define o max e o min
	if(threadIdx.x < iTermNumber){
		int docAmount = iDocNumberByTermList[threadIdx.x];
//		fingers[threadIdx.x].final = 0;
//		limitDoc.extraPosition[threadIdx.x] = 0;

		globalIndex = 0;

		int maxDoc;

		if(THREAD_MASTER) limitDoc.minDocId = 0;

		globalIndex = docAmount-1;

		maxDoc =  iDocIdList[iSharedPositionInitialInList[threadIdx.x] +  globalIndex];

		atomicMax(&(limitDoc.secondMaxDocId), maxDoc);

		fingers[threadIdx.x].docId = iDocIdList[iSharedPositionInitialInList[threadIdx.x]];
		fingers[threadIdx.x].position = iSharedPositionInitialInList[threadIdx.x];
	}

	__syncthreads();


	sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

	if(THREAD_MASTER){
		selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
		docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
//	}else if(THREAD_MASTER_2){
		score = 0.0;
	}

	__syncthreads();

	while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

		isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
//		count++;
//
//		if(fingers[sharedPivot.idTerm].docId == 38182)
//			printf("Oi");

		if(isValidCandidate){
			if(threadIdx.x < iTermNumber){
				int termId = iOrderedTermSharedList[threadIdx.x];
				float scoreL = 0.0;
				if(fingers[termId].docId == fingers[sharedPivot.idTerm].docId){
					scoreL = scoreTf_Idf(iFreqList[fingers[termId].position],
										iDocLenghtList[fingers[termId].position],
										dIdfList[termId],dAverageDocumentLength,1.0);
				}

				float aux = 0;
				for (int i = 0; i < TERM_NUMBER; ++i) {
					aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
				}

				if(THREAD_MASTER) score = aux;
//		 		atomicAdd(&score,scoreL);
			}

			padding = documentTopk.padding;

			__syncthreads();

/*				If the heap is not full
			the candidate is inserted into the heap. If the heap is full
			and the new score is larger than the minimum score in the
			heap, the new document is inserted into the heap, replacing
			the one with the minimum score.

*/
			if(padding != 0 || thresholdLocal < score ){
				thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
			}

			if(threadIdx.x < iTermNumber ){
				int docPivot = fingers[sharedPivot.idTerm].docId;
				if(fingers[threadIdx.x].docId ==  docPivot){
					fingers[threadIdx.x].position++;
					if(fingers[threadIdx.x].position >= (iDocNumberByTermList[threadIdx.x]+iSharedPositionInitialInList[threadIdx.x])){//Não Válido
						fingers[threadIdx.x].docId = NO_MORE_DOC;
						fingers[threadIdx.x].position = NO_VALID_POSITION;
					}else{
						fingers[threadIdx.x].docId = iDocIdList[fingers[threadIdx.x].position];
//			 			if(fingers[threadIdx.x].docId > limitDoc.secondMaxDocId){
//			 				fingers[threadIdx.x].docId = NO_MORE_DOC;
//			 				fingers[threadIdx.x].position = NO_VALID_POSITION;
//			 			}
					}
				}
			}
		}
		else{
			int pivotDoc = docCurrent;
			long long position;
			int docLocal;
			int idTerm;
			for (int j = 0; j < sharedPivot.positionInOrderedList; ++j) {
				idTerm = iOrderedTermSharedList[j];

				if(fingers[idTerm].docId == fingers[sharedPivot.idTerm].docId)//Até alcançar um finger q aponte a um documento pivo
					break;

				fingers[idTerm].docId = NO_MORE_DOC;
				position = fingers[idTerm].position + 1 + threadIdx.x;
				docLocal = -1;
				while(position < (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])
						&& docLocal < pivotDoc){
					docLocal = iDocIdList[position];
					position += blockDim.x;

				}
				position -= blockDim.x;

				if(docLocal < pivotDoc || position >= (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])){
					docLocal = NO_MORE_DOC;
					position = NO_VALID_POSITION;
				}

//				atomicMin(&(fingers[idTerm].docId) , docLocal);

				int docNeighbor, docAux = docLocal;
				for (int i = 16; i >= 1; i /= 2) {
					docNeighbor  = __shfl_down_sync(0xFFFFFFFF,docAux, i);

					if(docNeighbor < docAux)
						docAux = docNeighbor;
				}

				if( ((threadIdx.x & 0x1f) == 0)){
					atomicMin(&(fingers[idTerm].docId) , docAux);
				}


				__syncthreads();

				if(fingers[idTerm].docId == docLocal){
					fingers[idTerm].position = position;
				}
			}
		}

		__syncthreads();

		//Sort the terms in non decreasing order of DID
		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

		//Select term pivot
		if(THREAD_MASTER){
			selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
			score = 0.0;
		}
		__syncthreads();
	}

//	sortLocalTopkDocAndStoreInGlobal((float*)&(dTopkScoreListGlobal[blockIdx.x*iTopK]),(int*)&(iTopkDocListGlobal[blockIdx.x*iTopK]),iTopK,&documentTopk);

	int gIndex = blockIdx.x * iTopK + threadIdx.x;
	for (int localIndex = threadIdx.x; localIndex < iTopK; localIndex+=blockDim.x) {
		dTopkScoreListGlobal[gIndex] = documentTopk.score[localIndex];
		iTopkDocListGlobal[gIndex] = documentTopk.id[localIndex];

		gIndex+=blockDim.x;
	}
}

__global__ void matchWandParallel_VARIABLE_Batch_Block_2(const int* iDocIdList, const unsigned short int* iFreqList,
														const float *dUBlistGlobal, const float *dIdfListGlobal, const int *iDocLenghtList,
														const short int* iTermNumberByQuery, int *iTopkDocListGlobal,
														float *dTopkScoreListGlobal, const float dAverageDocumentLengthGlobal,
														const short int iTopK, const float iInitialThreshold,
														const int* iQueryTerms, const long long* ptrInitPostingList,
														int* ptrQueryPositions,int *iDocNumberByTermListGlobal){
//
//	if(blockIdx.x!=1)
//		return;


	__shared__ int queryPosition;
	__shared__ pivot sharedPivot;
	__shared__ finger fingers[TERM_NUMBER];

	__shared__ documentTopkList documentTopk;

	__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];
	__shared__ long long iSharedPositionInitialInList[TERM_NUMBER];
	__shared__ float dUBlist[TERM_NUMBER];
	__shared__ float dIdfList[TERM_NUMBER];
	__shared__ float dAverageDocumentLength;

	__shared__ int iDocNumberByTermList[TERM_NUMBER];

	__shared__ float score;
	__shared__ bool isValidCandidate;
	__shared__ int docCurrent;
	__shared__ limitDocId limitDoc;
	__shared__ short int iTermNumber;

	int padding;

 	float thresholdLocal;// = iInitialThreshold;
 	thresholdLocal = iInitialThreshold;

	int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
	int localIndex;

//	int count = 0;

	if(THREAD_MASTER){
		documentTopk.padding = iTopK;
		dAverageDocumentLength = dAverageDocumentLengthGlobal;
//	}else if(THREAD_MASTER_2){
//		limitDoc.minDocId = -1;
		limitDoc.secondMaxDocId = -1;

		iTermNumber = iTermNumberByQuery[blockIdx.x];
	}

	__syncthreads();

	if(threadIdx.x < iTermNumber){
		queryPosition = ptrQueryPositions[blockIdx.x];
		int idTerm = iQueryTerms[queryPosition + threadIdx.x];

		iDocNumberByTermList[threadIdx.x] = iDocNumberByTermListGlobal[idTerm];

		dUBlist[threadIdx.x] = dUBlistGlobal[idTerm];//[threadIdx.x];
		dIdfList[threadIdx.x] = dIdfListGlobal[idTerm];//[threadIdx.x];

//		printf(" %.2f ",dUBlist[threadIdx.x]);
		iSharedPositionInitialInList[threadIdx.x] = ptrInitPostingList[idTerm];
	}

	//Inicializa a lista de Score e Documentos dos Topk
	//Considero que o Top_K seja um número múltiplo do tamanho do bloco
	#pragma unroll 4
	for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
		documentTopk.id[localIndex] = -1;
		documentTopk.score[localIndex] = 0.0;
	}

	//Define o max e o min
	if(threadIdx.x < iTermNumber){
		int docAmount = iDocNumberByTermList[threadIdx.x];
//		fingers[threadIdx.x].final = 0;
//		limitDoc.extraPosition[threadIdx.x] = 0;

		globalIndex = 0;

		int maxDoc;

		if(THREAD_MASTER) limitDoc.minDocId = 0;

		globalIndex = docAmount-1;

		maxDoc =  iDocIdList[iSharedPositionInitialInList[threadIdx.x] +  globalIndex];

		atomicMax(&(limitDoc.secondMaxDocId), maxDoc);

		fingers[threadIdx.x].docId = iDocIdList[iSharedPositionInitialInList[threadIdx.x]];
		fingers[threadIdx.x].position = iSharedPositionInitialInList[threadIdx.x];
	}

	__syncthreads();


	sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

	if(THREAD_MASTER){
		selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
		docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
//	}else if(THREAD_MASTER_2){
		score = 0.0;
	}

	__syncthreads();

	while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

		isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
//		count++;
//
//		if(fingers[sharedPivot.idTerm].docId == 38182)
//			printf("Oi");

		if(isValidCandidate){
			if(threadIdx.x < iTermNumber){
		 		int termId = iOrderedTermSharedList[threadIdx.x];
		 		float scoreL = 0.0;
		 		if(fingers[termId].docId == fingers[sharedPivot.idTerm].docId){
		 			scoreL = scoreTf_Idf(iFreqList[fingers[termId].position],
										iDocLenghtList[fingers[termId].position],
										dIdfList[termId],dAverageDocumentLength,1.0);
		 		}

		 		float aux = 0;
		 		for (int i = 0; i < TERM_NUMBER; ++i) {
		 			aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
		 		}

		 		if(THREAD_MASTER) score = aux;
//		 		atomicAdd(&score,scoreL);
			}

			padding = documentTopk.padding;

			__syncthreads();

/*				If the heap is not full
			the candidate is inserted into the heap. If the heap is full
			and the new score is larger than the minimum score in the
			heap, the new document is inserted into the heap, replacing
			the one with the minimum score.

*/
			if(padding != 0 || thresholdLocal < score ){
				thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
			}

			if(threadIdx.x < iTermNumber ){
			 	int docPivot = fingers[sharedPivot.idTerm].docId;
			 	if(fingers[threadIdx.x].docId ==  docPivot){
			 		fingers[threadIdx.x].position++;
			 		if(fingers[threadIdx.x].position >= (iDocNumberByTermList[threadIdx.x]+iSharedPositionInitialInList[threadIdx.x])){//Não Válido
			 			fingers[threadIdx.x].docId = NO_MORE_DOC;
			 			fingers[threadIdx.x].position = NO_VALID_POSITION;
			 		}else{
			 			fingers[threadIdx.x].docId = iDocIdList[fingers[threadIdx.x].position];
//			 			if(fingers[threadIdx.x].docId > limitDoc.secondMaxDocId){
//			 				fingers[threadIdx.x].docId = NO_MORE_DOC;
//			 				fingers[threadIdx.x].position = NO_VALID_POSITION;
//			 			}
			 		}
			 	}
			}
		}
		else{
			int pivotDoc = docCurrent;
			long long position;
			int docLocal;
			int idTerm;
			for (int j = 0; j < sharedPivot.positionInOrderedList; ++j) {
				idTerm = iOrderedTermSharedList[j];

				if(fingers[idTerm].docId == fingers[sharedPivot.idTerm].docId)//Até alcançar um finger q aponte a um documento pivo
					break;

				fingers[idTerm].docId = NO_MORE_DOC;
				position = fingers[idTerm].position + 1 + threadIdx.x;
				docLocal = -1;
				while(position < (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])
						&& docLocal < pivotDoc){
					docLocal = iDocIdList[position];
					position += blockDim.x;

				}
				position -= blockDim.x;

				if(docLocal < pivotDoc || position >= (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])){
					docLocal = NO_MORE_DOC;
					position = NO_VALID_POSITION;
				}

//				atomicMin(&(fingers[idTerm].docId) , docLocal);

				int docNeighbor, docAux = docLocal;
				for (int i = 16; i >= 1; i /= 2) {
					docNeighbor  = __shfl_down_sync(0xFFFFFFFF,docAux, i);

					if(docNeighbor < docAux)
						docAux = docNeighbor;
				}

				if( ((threadIdx.x & 0x1f) == 0)){
					atomicMin(&(fingers[idTerm].docId) , docAux);
				}


				__syncthreads();

				if(fingers[idTerm].docId == docLocal){
					fingers[idTerm].position = position;
				}
			}
		}

		__syncthreads();

		//Sort the terms in non decreasing order of DID
		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

		//Select term pivot
		if(THREAD_MASTER){
			selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
			score = 0.0;
		}
		__syncthreads();
	}

//	sortLocalTopkDocAndStoreInGlobal((float*)&(dTopkScoreListGlobal[blockIdx.x*iTopK]),(int*)&(iTopkDocListGlobal[blockIdx.x*iTopK]),iTopK,&documentTopk);

	int gIndex = blockIdx.x * iTopK + threadIdx.x;
	for (int localIndex = threadIdx.x; localIndex < iTopK; localIndex+=blockDim.x) {
		dTopkScoreListGlobal[gIndex] = documentTopk.score[localIndex];
		iTopkDocListGlobal[gIndex] = documentTopk.id[localIndex];

		gIndex+=blockDim.x;
	}

//		if(THREAD_MASTER)
////			atomicAdd(&globalCount,count);
//
//		if(THREAD_MASTER)
//			printf("-----%d %d----", blockIdx.x, count);

}


__global__ void matchWandParallel_VARIABLE_Batch_Block(const int* iDocIdList, const unsigned short int* iFreqList,
										  const float *dUBlist, const float *dIdfList, const int *iDocLenghtList,
										  const short int *iTermNumberByQuery, int *iTopkDocListGlobal,
										  float *dTopkScoreListGlobal, const float dAverageDocumentLength,
										  const int iBlockRoundNumber, const int iGlobalRoundNumber,
										  const short int iTopK, const float iInitialThreshold,
										  const int* iQueryTerms, const long long* ptrPostingPositions,
										  int* ptrQueryPositions, int *iDocNumberByTermList){

	if(blockIdx.x != 4999)
		return;

	__shared__ short int iTermNumber;

	__shared__ pivot sharedPivot;
	__shared__ finger fingers[TERM_NUMBER];

	__shared__ postingList postingLists[TERM_NUMBER];
	__shared__ documentTopkList documentTopk;

	__shared__ long long ptrPostingPositionShared[TERM_NUMBER];
	__shared__ int iDocNumberByTermListShared[TERM_NUMBER];
	__shared__ int queryPosition;

	__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];

//	__shared__ int iGlobalInitialPositionInList;
	__shared__ unsigned short int iElementQuantityByBlock;

	__shared__ float score;
	__shared__ bool isValidCandidate;

//	__shared__ short int needSearchDocRange[TERM_NUMBER];
	__shared__ limitDocId limitDoc;

 	float thresholdLocal;// = iInitialThreshold;

 	thresholdLocal = iInitialThreshold;

	int globalIndex = 0;// = iInitialPositionGlobal + threadIdx.x;
	int localIndex;
//	long long positionInitialInTermPostingList;//int positionInitialInTermPostingList;

	if(thresholdLocal < globalThreshold)
		thresholdLocal = globalThreshold;

	if(THREAD_MASTER){
		documentTopk.padding = iTopK;
		iTermNumber = iTermNumberByQuery[blockDim.x];
	}else if(THREAD_MASTER_2){
		iElementQuantityByBlock = DOC_QUANTITY_IN_MEMORY;//iBlockRoundNumber * DOC_QUANTITY_IN_MEMORY;
//		iGlobalInitialPositionInList = 0;//iElementQuantityByBlock  * blockIdx.x * iGlobalRoundNumber;
	}

	//Inicializa a lista de Score e Documentos dos Topk
	//Considero que o Top_K seja um número múltiplo do tamanho do bloco
	for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
		documentTopk.id[localIndex] = -1;
		documentTopk.score[localIndex] = 0.0;
	}

//	if(THREAD_MASTER) documentTopk.padding = iTopK;

	__syncthreads();

	//Define o max e o min
	if(threadIdx.x < iTermNumber){
		limitDoc.extraPosition[threadIdx.x] = 0;
		queryPosition = ptrQueryPositions[blockDim.x];

		iDocNumberByTermListShared[threadIdx.x] = iDocNumberByTermList[iQueryTerms[queryPosition + threadIdx.x]];
		ptrPostingPositionShared[threadIdx.x] = ptrPostingPositions[iQueryTerms[queryPosition+threadIdx.x]];

		int docAmount = iDocNumberByTermListShared[threadIdx.x];//iDocNumberByTermList[threadIdx.x];
//		globalIndex = iGlobalInitialPositionInList;

		int aux, maxDoc;
		int maxNeighbor;

		if(THREAD_MASTER) limitDoc.minDocId = 0;

//		int isTail = globalIndex < docAmount;
//		globalIndex += iElementQuantityByBlock * iGlobalRoundNumber;
//		isTail &= globalIndex >= docAmount;

		globalIndex = docAmount - 1;
//		int isTail = iElementQuantityByBlock >= docAmount;
//
//		if(isTail){
//			globalIndex = iGlobalInitialPositionInList + (docAmount - iGlobalInitialPositionInList - 1);
//		}

//		maxDoc =  (isTail || globalIndex < docAmount) ? iDocIdList[ptrPostingPositionShared[threadIdx.x] +  globalIndex] - 1 :
//														-1;
		maxDoc =  iDocIdList[ptrPostingPositionShared[threadIdx.x] +  globalIndex];

		aux = maxDoc;
		for (int i = 1; i < iTermNumber; ++i) {
			maxNeighbor = __shfl_sync(0xFFFFFFFF,aux,i);
			if(maxNeighbor > maxDoc)
				maxDoc = maxNeighbor;
		}
		if(THREAD_MASTER) limitDoc.secondMaxDocId = maxDoc;
	}

//	__syncthreads();
//
//	//Busca faixa de documentos;
//	for (int internTermId = 0; internTermId < iTermNumber; ++internTermId) {
//		if(needSearchDocRange[internTermId])
//			searchRangeOfDocs_batch(iDocIdList,postingLists, internTermId, iGlobalInitialPositionInList,
//									&limitDoc,iElementQuantityByBlock,iGlobalRoundNumber,
//									iDocNumberByTermListShared[internTermId], ptrPostingPositionShared[internTermId]);
//	}

	__syncthreads();

	//Preenche a memória compartilhada
//	positionInitialInTermPostingList = 0;
	int docLocal, docAmount;
	for (int termId = 0; termId < iTermNumber; ++termId) {
//		globalIndex = iGlobalInitialPositionInList + limitDoc.extraPosition[termId] + threadIdx.x;
		globalIndex = threadIdx.x;

		docAmount = iDocNumberByTermListShared[termId];
		docLocal = -1;

		for (localIndex = threadIdx.x; localIndex < iElementQuantityByBlock; localIndex+=blockDim.x) {

			docLocal = (globalIndex < docAmount) ? iDocIdList[ptrPostingPositionShared[termId] + globalIndex]
			                                                  : NO_MORE_DOC;

			if(globalIndex > docAmount){
				postingLists[termId].docId[localIndex] = NO_MORE_DOC;
				fingers[termId].final = 1;
				break;
			}

			postingLists[termId].docId[localIndex] = docLocal;
			postingLists[termId].docLenght[localIndex] = iDocLenghtList[ptrPostingPositionShared[termId] + globalIndex];
			postingLists[termId].freq[localIndex] = iFreqList[ptrPostingPositionShared[termId] + globalIndex];

			globalIndex += blockDim.x;
		}

//		positionInitialInTermPostingList += iDocNumberByTermList[termId];
	}

//	__syncthreads();

	if(threadIdx.x < iTermNumber){
		fingers[threadIdx.x].docId = postingLists[threadIdx.x].docId[0];
		fingers[threadIdx.x].position = (fingers[threadIdx.x].docId == NO_MORE_DOC) ? NO_VALID_POSITION : 0;
		fingers[threadIdx.x].final = (fingers[threadIdx.x].final == 1) ? 1 : 0;
	}

	__syncthreads();

	__shared__ int docCurrent;

	sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

//	__syncthreads();

	if(THREAD_MASTER){
		selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
		docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
	}else if(THREAD_MASTER_2){
		score = 0.0;
	}

	int padding;
	int threadIdInWarp = (threadIdx.x & 0x1f);
	int idWarp = ((blockDim.x >> 5) == 1 ) ? 1 :  threadIdx.x >> 5;

	__syncthreads();

	while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

		if(THREAD_MASTER)
			isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);

		__syncthreads();

		if(isValidCandidate){

			if(threadIdx.x < iTermNumber){
				fullScore_3_1(&score, fingers[sharedPivot.idTerm].docId, iOrderedTermSharedList,
								fingers,postingLists, dIdfList, dAverageDocumentLength);
			}

			padding = documentTopk.padding;

			__syncthreads();

			if(thresholdLocal < score){
				thresholdLocal = managerMinValue_v5(&documentTopk, docCurrent, score,padding);
			}

			if(idWarp == 1 && threadIdInWarp < iTermNumber ){
				advancePivoTermFinger_4(sharedPivot,fingers, postingLists,iElementQuantityByBlock,threadIdInWarp);
			}
		}
		else{
			 advanceDocIdOfPredecessorTerm_4(postingLists,
										   	 iOrderedTermSharedList,
										   	 fingers,sharedPivot,fingers[sharedPivot.idTerm].docId,
										   	 iElementQuantityByBlock);
		}

		__syncthreads();

		for (int termId = 0; termId < iTermNumber; ++termId) {
			if(fingers[termId].docId == NO_MORE_DOC && 	fingers[termId].final == 0){

//				searchMoreDocs_batch(iDocIdList,iFreqList,iDocLenghtList,postingLists,
//							  	  	 termId,iGlobalInitialPositionInList,&limitDoc,
//							  	  	 iElementQuantityByBlock,&(fingers[termId]),docCurrent,
//							  	  	 iDocNumberByTermListShared[termId],ptrPostingPositionShared[termId]);

				searchMoreDocs_batch(iDocIdList,iFreqList,iDocLenghtList,postingLists,
									 termId, 0, &limitDoc,
									 iElementQuantityByBlock,&(fingers[termId]),docCurrent,
									 iDocNumberByTermListShared[termId],ptrPostingPositionShared[termId]);
//
//
//////
//					if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > thresholdGlobal){
////					if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > thresholdGlobal){
//						atomicMaxD(&thresholdGlobal,thresholdLocal);
//					}
//
//					if((documentTopk.padding < (iTopK >> 1)))
//					if(thresholdLocal < thresholdGlobal){
//						thresholdLocal = thresholdGlobal;
//					}

			}
		}


//		__syncthreads();//Talvez não precise
		//Sort the terms in non decreasing order of DID
		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

//		__syncthreads();//Talvez não precise

		//Select term pivot
		if(THREAD_MASTER){
			selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
			score = 0.0;
		}
		__syncthreads();
	}

//	if(THREAD_MASTER){
//	int max = iTopK - documentTopk.padding;
//	int i = 0;
//	while(i > max){
//		if(2*i+2 < max)
//			if(documentTopk.score[i] > documentTopk.score[2*i+2])
//				printf("ERRADO!!!\n");
//
//		if(2*i+1 < max)
//			if(documentTopk.score[i] > documentTopk.score[2*i+1])
//				printf("ERRADO!!!\n");
//
//		i++;
//	}}
//
	__syncthreads();

	sortLocalTopkDocAndStoreInGlobal_BLOCK(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);
//	globalIndex =  iTopK * blockIdx.x + threadIdx.x + documentTopk.padding;
//	for (localIndex = threadIdx.x; localIndex < (iTopK - documentTopk.padding) ; localIndex += blockDim.x) {
//		iTopkDocListGlobal[globalIndex]   = documentTopk.id[localIndex];
//		dTopkScoreListGlobal[globalIndex] = documentTopk.score[localIndex];
//		globalIndex += blockDim.x;
//	}
//	__syncthreads();



}


__global__ void matchWandParallel_BATCH(const int* iDocIdList, const unsigned short int* iFreqList,
									    const float *dUBlist, const float *dIdfList, const int *iDocLenghtList,
										const short int iTermNumber, int *iTopkDocListGlobal,
										float *dTopkScoreListGlobal, const float dAverageDocumentLength,
										const int iBlockRoundNumber, const int iGlobalRoundNumber,
										const short int iTopK, const float iInitialThreshold,
										const int* iQueryTerms, const long long* ptrInitPostingList,
										int* ptrQueryPositions, int idQuery,int *iDocNumberByTermList){


	__shared__ pivot sharedPivot;
	__shared__ finger fingers[TERM_NUMBER];

	__shared__ postingList postingLists[TERM_NUMBER];
	__shared__ documentTopkList documentTopk;

	__shared__ long long ptrInitPostingListShared[TERM_NUMBER];
	__shared__ int iDocNumberByTermListShared[TERM_NUMBER];
	__shared__ int queryPosition;

	__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];

	__shared__ int iGlobalInitialPositionInList;
	__shared__ unsigned short int iElementQuantityByBlock;

	__shared__ float score;
	__shared__ bool isValidCandidate;

	__shared__ short int needSearchDocRange[TERM_NUMBER];
	__shared__ limitDocId limitDoc;

 	float thresholdLocal;// = iInitialThreshold;

 	thresholdLocal = iInitialThreshold;

	int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
	int localIndex;
//	long long positionInitialInTermPostingList;//int positionInitialInTermPostingList;

	if(thresholdLocal < globalThreshold)
		thresholdLocal = globalThreshold;

	if(THREAD_MASTER){
		documentTopk.padding = iTopK;
	}else if(THREAD_MASTER_2){
		iElementQuantityByBlock = DOC_QUANTITY_IN_MEMORY;//iBlockRoundNumber * DOC_QUANTITY_IN_MEMORY;
		iGlobalInitialPositionInList = iElementQuantityByBlock  * blockIdx.x * iGlobalRoundNumber;
	}

	#ifdef DEBUG
		if(THREAD_MASTER_2)
			if(iGlobalInitialPositionInList < 0)
				printf("Opa!!!!");
	#endif

	//Inicializa a lista de Score e Documentos dos Topk
	//Considero que o Top_K seja um número múltiplo do tamanho do bloco
	for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
		documentTopk.id[localIndex] = -1;
		documentTopk.score[localIndex] = 0.0;
	}

//	if(THREAD_MASTER) documentTopk.padding = iTopK;

//	__syncthreads();

	//Define o max e o min
	if(threadIdx.x < iTermNumber){
		limitDoc.extraPosition[threadIdx.x] = 0;
		queryPosition = ptrQueryPositions[idQuery];
		iDocNumberByTermListShared[threadIdx.x] = iDocNumberByTermList[iQueryTerms[queryPosition + threadIdx.x]];
		ptrInitPostingListShared[threadIdx.x] = ptrInitPostingList[iQueryTerms[queryPosition + threadIdx.x]];

		int docAmount = iDocNumberByTermListShared[threadIdx.x];//iDocNumberByTermList[threadIdx.x];
		globalIndex = iGlobalInitialPositionInList;
//		positionInitialInTermPostingList = 0;
//
//		for (int i = 0; i < threadIdx.x; ++i) {
//			positionInitialInTermPostingList += iDocNumberByTermList[iQueryTerms[i]];//iDocNumberByTermList[i];
//		}

//		positionInitialInTermPostingList = ptrPostingPositionShared[threadIdx.x];

		int aux, maxDoc;
		int maxNeighbor;
		if(blockIdx.x != 0){
			int maxDoc = (globalIndex < docAmount) ? iDocIdList[ptrInitPostingListShared[threadIdx.x] + globalIndex - 1] : -1;
			maxDoc++;
			aux = maxDoc;

			atomicMax(&limitDoc.minDocId, maxDoc);

//			for (int i = 1; i < iTermNumber; ++i) {
//				maxNeighbor = __shfl(aux,i);
//				if(maxNeighbor > maxDoc)
//					maxDoc = maxNeighbor;
//			}
//
//			if(THREAD_MASTER) limitDoc.minDocId = maxDoc; //atomicExch(&(limitDoc.minDocId), maxDoc);

			if(aux < limitDoc.minDocId && aux != 0){
				needSearchDocRange[threadIdx.x] = 1;
				limitDoc.extraPosition[threadIdx.x] = NO_MORE_DOC;
			}
		}else
			if(THREAD_MASTER) limitDoc.minDocId = 0;

		int isTail = globalIndex < docAmount;
		globalIndex += iElementQuantityByBlock * iGlobalRoundNumber;
		isTail &= globalIndex >= docAmount;

		if(isTail){
			globalIndex = iGlobalInitialPositionInList + (docAmount - iGlobalInitialPositionInList - 1);
		}

		maxDoc =  (isTail || globalIndex < docAmount) ? iDocIdList[ptrInitPostingListShared[threadIdx.x] +  globalIndex] :
														-1;
		aux = maxDoc;
		for (int i = 1; i < iTermNumber; ++i) {
			maxNeighbor = __shfl_down_sync(0xFFFFFFFF,aux,i);
			if(maxNeighbor > maxDoc)
				maxDoc = maxNeighbor;
		}
		if(THREAD_MASTER) limitDoc.secondMaxDocId = maxDoc;
	}

	__syncthreads();

	//Busca faixa de documentos;
	for (int internTermId = 0; internTermId < iTermNumber; ++internTermId) {
		if(needSearchDocRange[internTermId])
			searchRangeOfDocs_batch(iDocIdList,postingLists, internTermId, iGlobalInitialPositionInList,
									&limitDoc,iElementQuantityByBlock,iGlobalRoundNumber,
									iDocNumberByTermListShared[internTermId],
									ptrInitPostingListShared[internTermId]);
	}

	__syncthreads();

	//Preenche a memória compartilhada
//	positionInitialInTermPostingList = 0;
	int docLocal, docAmount;
	for (int termId = 0; termId < iTermNumber; ++termId) {
		globalIndex = iGlobalInitialPositionInList + limitDoc.extraPosition[termId] + threadIdx.x;
		docAmount = iDocNumberByTermListShared[termId];
		docLocal = -1;

		for (localIndex = threadIdx.x; localIndex < iElementQuantityByBlock; localIndex+=blockDim.x) {

			docLocal = (globalIndex < docAmount) ? iDocIdList[ptrInitPostingListShared[termId] + globalIndex]
			                                     : NO_MORE_DOC;

			if(docLocal > limitDoc.secondMaxDocId || globalIndex > docAmount){
				postingLists[termId].docId[localIndex] = NO_MORE_DOC;
				fingers[termId].final = 1;
				break;
			}

			postingLists[termId].docId[localIndex] = docLocal;
			postingLists[termId].docLenght[localIndex] = iDocLenghtList[ptrInitPostingListShared[termId] + globalIndex];
			postingLists[termId].freq[localIndex] = iFreqList[ptrInitPostingListShared[termId] + globalIndex];

			globalIndex += blockDim.x;
		}

//		positionInitialInTermPostingList += iDocNumberByTermList[termId];
	}

//	__syncthreads();

	if(threadIdx.x < iTermNumber){
		fingers[threadIdx.x].docId = postingLists[threadIdx.x].docId[0];
		fingers[threadIdx.x].position = (fingers[threadIdx.x].docId == NO_MORE_DOC) ? NO_VALID_POSITION : 0;
		fingers[threadIdx.x].final = 0;
	}

	__syncthreads();

	__shared__ int docCurrent;

	sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

//	__syncthreads();

	if(THREAD_MASTER){
		selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
		docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
	}else if(THREAD_MASTER_2){
		score = 0.0;
	}

	int padding;
	int threadIdInWarp = (threadIdx.x & 0x1f);
	int idWarp = ((blockDim.x >> 5) == 1 ) ? 1 :  threadIdx.x >> 5;

	__syncthreads();

	while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

		if(THREAD_MASTER)
			isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);

		__syncthreads();

		if(isValidCandidate){

			if(threadIdx.x < iTermNumber){
				fullScore_3_1(&score, fingers[sharedPivot.idTerm].docId, iOrderedTermSharedList,
								fingers,postingLists, dIdfList, dAverageDocumentLength);
			}

			padding = documentTopk.padding;

			__syncthreads();

			if(thresholdLocal < score){
				thresholdLocal = managerMinValue_v5(&documentTopk, docCurrent, score,padding);
			}

			if(idWarp == 1 && threadIdInWarp < iTermNumber ){
				advancePivoTermFinger_4(sharedPivot,fingers, postingLists,iElementQuantityByBlock,threadIdInWarp);
			}
		}
		else{
			 advanceDocIdOfPredecessorTerm_4(postingLists,
										   iOrderedTermSharedList,
										   fingers,sharedPivot,fingers[sharedPivot.idTerm].docId,
										   iElementQuantityByBlock);
		}

		__syncthreads();

		for (int termId = 0; termId < iTermNumber; ++termId) {
			if(fingers[termId].docId == NO_MORE_DOC && 	fingers[termId].final == 0){

				searchMoreDocs_batch(iDocIdList,iFreqList,iDocLenghtList,postingLists,
							  	  	 termId,iGlobalInitialPositionInList,&limitDoc,
							  	  	 iElementQuantityByBlock,&(fingers[termId]),docCurrent,
							  	  	 iDocNumberByTermListShared[termId],ptrInitPostingListShared[termId]);

//
//
//////
//					if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > thresholdGlobal){
////					if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > thresholdGlobal){
//						atomicMaxD(&thresholdGlobal,thresholdLocal);
//					}
//
//					if((documentTopk.padding < (iTopK >> 1)))
//					if(thresholdLocal < thresholdGlobal){
//						thresholdLocal = thresholdGlobal;
//					}

			}
		}


//		__syncthreads();//Talvez não precise
		//Sort the terms in non decreasing order of DID
		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

//		__syncthreads();//Talvez não precise

		//Select term pivot
		if(THREAD_MASTER){
			selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
			score = 0.0;
		}
		__syncthreads();
	}

//	if(THREAD_MASTER){
//	int max = iTopK - documentTopk.padding;
//	int i = 0;
//	while(i > max){
//		if(2*i+2 < max)
//			if(documentTopk.score[i] > documentTopk.score[2*i+2])
//				printf("ERRADO!!!\n");
//
//		if(2*i+1 < max)
//			if(documentTopk.score[i] > documentTopk.score[2*i+1])
//				printf("ERRADO!!!\n");
//
//		i++;
//	}}
//
//	__syncthreads();

	sortLocalTopkDocAndStoreInGlobal(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);
//	globalIndex =  iTopK * blockIdx.x + threadIdx.x + documentTopk.padding;
//	for (localIndex = threadIdx.x; localIndex < (iTopK - documentTopk.padding) ; localIndex += blockDim.x) {
//		iTopkDocListGlobal[globalIndex]   = documentTopk.id[localIndex];
//		dTopkScoreListGlobal[globalIndex] = documentTopk.score[localIndex];
//		globalIndex += blockDim.x;
//	}
//	__syncthreads();

}

__global__ void preProcessingWand(const int* iDocIdList,
								  const short int iTermNumber,
								  const int* iDocNumberByTermList,
								  const int* iInitialPositionPostingList,
								  const int docIdNumberByBlock,
								  int* extraPositions, int* docMaxList){

	__shared__ int iGlobalInitialPositionInList;
	int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
//	int positionInitialInTermPostingList;

	__shared__ int sharedMinDoc;
	__shared__ int sharedMaxDoc;
	__shared__ int sharedExtraPositions[TERM_NUMBER];
	__shared__ int sharedInitialDocId[TERM_NUMBER];
	__shared__ int sharedDocNumberByList[TERM_NUMBER];

	if(THREAD_MASTER){
		iGlobalInitialPositionInList = docIdNumberByBlock  * blockIdx.x;
	}

//	__syncthreads();

	if(threadIdx.x < iTermNumber){
		sharedDocNumberByList[threadIdx.x] = iDocNumberByTermList[threadIdx.x];
		int docAmount = sharedDocNumberByList[threadIdx.x];
		globalIndex = iGlobalInitialPositionInList;
//		positionInitialInTermPostingList = iInitialPositionPostingList[threadIdx.x];

		int maxDoc;
//		int aux, maxDoc;
//		int maxNeighbor;
		if(blockIdx.x != 0){
			int maxDoc = (globalIndex < docAmount) ? iDocIdList[iInitialPositionPostingList[threadIdx.x] + globalIndex - 1] : -1;
			maxDoc++;
			sharedInitialDocId[threadIdx.x] = maxDoc;
//			aux = maxDoc;
//			for (int i = iTermNumber-1; i > 0; --i) {
//				maxNeighbor = __shfl(aux,i);
//				if(maxNeighbor > maxDoc) maxDoc = maxNeighbor;
//			}
			atomicMax(&sharedMinDoc, maxDoc);
//			if(THREAD_MASTER) sharedMinDoc = maxDoc;

//			if(aux < limitDoc.minDocId && aux != 0)
//				needSearchDocRange[threadIdx.x] = 1;
		}else
			sharedMinDoc = 0;

		int isTail = globalIndex < docAmount;
		globalIndex += docIdNumberByBlock;
		isTail &= globalIndex >= docAmount;

		if(isTail){
			globalIndex = iGlobalInitialPositionInList + (docAmount - iGlobalInitialPositionInList - 1);
		}

		maxDoc =  (isTail || globalIndex < docAmount) ? (iDocIdList[iInitialPositionPostingList[threadIdx.x] +  globalIndex]-1) : -1;
//		aux = maxDoc;
//		for (int i = 1; i < iTermNumber; ++i) {
//			maxNeighbor = __shfl(aux,i);
//			if(maxNeighbor > maxDoc)
//				maxDoc = maxNeighbor;
//		}
//		if(THREAD_MASTER) sharedMaxDoc = maxDoc;

		atomicMax(&sharedMaxDoc, maxDoc);
	}

	__syncthreads();

	for (int iTerm = 0; iTerm < iTermNumber; ++iTerm) {
		if(sharedInitialDocId[iTerm] < sharedMinDoc){
			globalIndex = iInitialPositionPostingList[iTerm] + iGlobalInitialPositionInList + threadIdx.x;

			int docLocal = -1;
			while (docLocal < sharedMinDoc && globalIndex < sharedDocNumberByList[iTerm]){
				docLocal = iDocIdList[globalIndex];
				globalIndex += blockDim.x;
			}
			globalIndex-= blockDim.x;

			long long int initialPosition;
			if(docLocal < sharedMinDoc)//Caso não encontre
				initialPosition = NO_VALID_POSITION;
			else
				initialPosition = globalIndex - iGlobalInitialPositionInList - iInitialPositionPostingList[iTerm];

			int positionNeighbor;
			for (int i = 16; i >= 1; i /= 2) {
				positionNeighbor  = __shfl_down_sync(0xFFFFFFFF,initialPosition, i);

				if(positionNeighbor < initialPosition)
					initialPosition = positionNeighbor;
			}

			if( ((threadIdx.x & 0x1f) == 0) && initialPosition != NO_MORE_DOC){
				atomicMin(&sharedExtraPositions[iTerm] , initialPosition);
			}
			//__syncthreads();


//			if(THREAD_MASTER){
//				globalIndex = iGlobalInitialPositionInList + iElementQuantityByBlock * roundGlobalNumber + threadIdx.x;
//				globalIndex += limitDoc->extraPosition[termId];
//			//
//				if(globalIndex < iDocNumberByTermList[termId]){
//					if(limitDoc->secondMaxDocId < iDocIdList[positionInListGlobal + globalIndex] -1)
//						limitDoc->secondMaxDocId = iDocIdList[positionInListGlobal + globalIndex]-1;
//				}
//			}
		}

		if(threadIdx.x < iTermNumber){
			extraPositions[iTermNumber*blockIdx.x + threadIdx.x] = sharedExtraPositions[threadIdx.x];

			if(THREAD_MASTER) docMaxList[blockIdx.x] = sharedMaxDoc;
		}
	}
}

__global__ void matchWandParallel_VARIABLE_3_Teste(const int* iDocIdList, const unsigned short int* iFreqList,
										  const float *dUBlist, const float *dIdfList,
										  const int *iDocLenghtList, const short int iTermNumber, int *iTopkDocListGlobal,
										  float *dTopkScoreListGlobal, const float dAverageDocumentLength,
										  const int iGlobalRoundNumber,
										  const short int iTopK, const float iInitialThreshold,
										  const int* d_iDocNumberByTermList,
										  const int* extraPositions, const int* docMaxList){

		__shared__ pivot sharedPivot;
		__shared__ finger fingers[TERM_NUMBER];

		__shared__ postingList postingLists[TERM_NUMBER];
		__shared__ documentTopkList documentTopk;

		__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];

		__shared__ int iGlobalInitialPositionInList;
		__shared__ unsigned short int iElementQuantityByBlock;

		__shared__ float score;
		__shared__ bool isValidCandidate;

//		__shared__ short int needSearchDocRange[TERM_NUMBER];
		__shared__ limitDocId limitDoc;

//		int count = 0;
		float thresholdLocal;// = iInitialThreshold;

		thresholdLocal = iInitialThreshold;

		int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
		int localIndex;
		int positionInitialInTermPostingList;

		if(thresholdLocal < globalThreshold)
			thresholdLocal = globalThreshold;

		if(threadIdx.x < iTermNumber){
			limitDoc.extraPosition[threadIdx.x] = extraPositions[blockIdx.x*iTermNumber + threadIdx.x];
			fingers[threadIdx.x].final = 0;
		}

		if(THREAD_MASTER){
			documentTopk.padding = iTopK;
			limitDoc.secondMaxDocId = docMaxList[blockIdx.x];
		}else if(THREAD_MASTER_2){
			iElementQuantityByBlock = DOC_QUANTITY_IN_MEMORY;//iBlockRoundNumber * DOC_QUANTITY_IN_MEMORY;
			iGlobalInitialPositionInList = iElementQuantityByBlock  * blockIdx.x * iGlobalRoundNumber;
		}

		//Inicializa a lista de Score e Documentos dos Topk
		//Considero que o Top_K seja um número múltiplo do tamanho do bloco
		for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
			documentTopk.id[localIndex] = -1;
			documentTopk.score[localIndex] = 0.0;
		}

		__syncthreads();

		//Preenche a memória compartilhada
		positionInitialInTermPostingList = 0;
		int docLocal, docAmount;
		for (int termId = 0; termId < iTermNumber; ++termId) {
			globalIndex = iGlobalInitialPositionInList + limitDoc.extraPosition[termId] + threadIdx.x;
			docAmount = d_iDocNumberByTermList[termId];
			docLocal = -1;

			for (localIndex = threadIdx.x; localIndex < iElementQuantityByBlock; localIndex+=blockDim.x) {

				docLocal = (globalIndex < docAmount) ? iDocIdList[positionInitialInTermPostingList + globalIndex]
																  : NO_MORE_DOC;

				if(docLocal > limitDoc.secondMaxDocId || globalIndex > docAmount){
					postingLists[termId].docId[localIndex] = NO_MORE_DOC;
					fingers[termId].final = 1;
					break;
				}

				postingLists[termId].docId[localIndex] = docLocal;
				postingLists[termId].docLenght[localIndex] = iDocLenghtList[positionInitialInTermPostingList + globalIndex];
				postingLists[termId].freq[localIndex] = iFreqList[positionInitialInTermPostingList + globalIndex];

				globalIndex += blockDim.x;
			}

			positionInitialInTermPostingList += d_iDocNumberByTermList[termId];
		}

	//	__syncthreads();

		if(threadIdx.x < iTermNumber){
			fingers[threadIdx.x].docId = postingLists[threadIdx.x].docId[0];
			fingers[threadIdx.x].position = (fingers[threadIdx.x].docId == NO_MORE_DOC) ? NO_VALID_POSITION : 0;
//			fingers[threadIdx.x].final = 0;
		}

		__syncthreads();

			__shared__ int docCurrent;

		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

	//	__syncthreads();

		if(THREAD_MASTER){
			selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
		}else if(THREAD_MASTER_2){
			score = 0.0;
		}

		int padding;
		int threadIdInWarp = (threadIdx.x & 0x1f);
		int idWarp = ((blockDim.x >> 5) == 1 ) ? 1 :  threadIdx.x >> 5;

		__syncthreads();

		while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){


			if(THREAD_MASTER){
				isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
			}

			__syncthreads();

			if(isValidCandidate){

				if(threadIdx.x < iTermNumber){
					fullScore_3_1(&score, fingers[sharedPivot.idTerm].docId, iOrderedTermSharedList,
								  fingers,postingLists, dIdfList, dAverageDocumentLength);
				}

				padding = documentTopk.padding;

				__syncthreads();

	/*				If the heap is not full
				the candidate is inserted into the heap. If the heap is full
				and the new score is larger than the minimum score in the
				heap, the new document is inserted into the heap, replacing
				the one with the minimum score.

	*/
				if(padding != 0 || thresholdLocal < score ){
					thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
				}

				if(idWarp == 1 && threadIdInWarp < iTermNumber){
					advancePivoTermFinger_4(sharedPivot,fingers, postingLists,iElementQuantityByBlock,threadIdInWarp);
				}
			}
			else{
				 advanceDocIdOfPredecessorTerm_4(postingLists,
											   iOrderedTermSharedList,
											   fingers,sharedPivot,fingers[sharedPivot.idTerm].docId,
											   iElementQuantityByBlock);
			}

			__syncthreads();

			for (int termId = 0; termId < iTermNumber; ++termId) {
				if(fingers[termId].docId == NO_MORE_DOC && 	fingers[termId].final == 0){

					searchMoreDocs(iDocIdList,iFreqList,iDocLenghtList,postingLists,
								  termId,iGlobalInitialPositionInList,
								  &limitDoc,iElementQuantityByBlock,
								  &(fingers[termId]),docCurrent,d_iDocNumberByTermList);

					if (SHAREDTHESHOLD == 1){//SHARED_READ
						if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > globalThreshold){
//							atomicMaxD(&globalThreshold,thresholdLocal);
							globalThreshold = thresholdLocal;
//							atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
						}

						if(thresholdLocal < globalThreshold){
							thresholdLocal = globalThreshold;
						}
					}else if (SHAREDTHESHOLD == 2){ //TSHARED_WRITEREAD
						if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > globalThreshold){
//							globalThreshold,thresholdLocal);
							globalThreshold = thresholdLocal;
//							atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
						}

						if((documentTopk.padding < (iTopK >> 1)))
							if(thresholdLocal < globalThreshold){
								thresholdLocal = globalThreshold;
							}
					}
				}
			}

			//Sort the terms in non decreasing order of DID
			sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

			//Select term pivot
			if(THREAD_MASTER){
				selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
				docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
				score = 0.0;
			}
			__syncthreads();
		}

		sortLocalTopkDocAndStoreInGlobal(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);

		if(THREAD_MASTER && thresholdLocal > globalThreshold){
			atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
		}
//		if(threadIdx.x == 0)
//			printf("---------%d----------",count);
}

__global__ void matchWandParallel_BATCH_2(const int* iDocIdList, const unsigned short int* iFreqList,
										const float *dUBlistGlobal, const float *dIdfListGlobal, const int *iDocLenghtList,
										const int iTermNumber, int *iTopkDocListGlobal,
										float *dTopkScoreListGlobal, const float dAverageDocumentLengthGlobal,
										const int iBlockRoundNumber, const int iGlobalRoundNumber,
										const short int iTopK, const float iInitialThreshold,
										const int* iQueryTerms, const long long* ptrInitPostingList,
										int* ptrQueryPositions, int idQuery,int *iDocNumberByTermListGlobal){

//
//		if(idQuery != 18 || blockIdx.x != 0)
//			return;

		__shared__ int queryPosition;
		__shared__ pivot sharedPivot;
		__shared__ finger fingers[TERM_NUMBER];

		__shared__ documentTopkList documentTopk;

		__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];
		__shared__ long long iSharedPositionInitialInList[TERM_NUMBER];
		__shared__ float dUBlist[TERM_NUMBER];
		__shared__ float dIdfList[TERM_NUMBER];
		__shared__ float dAverageDocumentLength;

		__shared__ int iDocNumberByTermList[TERM_NUMBER];
		__shared__ int iGlobalInitialPositionInList;

		__shared__ float score;
		__shared__ bool isValidCandidate;
		__shared__ int docCurrent;
		__shared__ limitDocId limitDoc;

		int padding;

	 	float thresholdLocal;// = iInitialThreshold;
	 	thresholdLocal = iInitialThreshold;

		int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
		int localIndex;
//		long long positionInitialInTermPostingList;

		if(thresholdLocal < globalThresholdBatch[idQuery])
			thresholdLocal = globalThresholdBatch[idQuery];

//		int count = 0;

		if(THREAD_MASTER){
			documentTopk.padding = iTopK;
			dAverageDocumentLength = dAverageDocumentLengthGlobal;
	//	}else if(THREAD_MASTER_2){
			limitDoc.minDocId = 0;
			limitDoc.secondMaxDocId = 0;
			iGlobalInitialPositionInList = DOC_QUANTITY_IN_MEMORY  * blockIdx.x * iGlobalRoundNumber;
		}

		if(threadIdx.x < iTermNumber){
			queryPosition = ptrQueryPositions[idQuery];
			int idTerm = iQueryTerms[queryPosition + threadIdx.x];

			fingers[threadIdx.x].docId = NO_MORE_DOC;
			fingers[threadIdx.x].position = NO_VALID_POSITION;

			iDocNumberByTermList[threadIdx.x] = iDocNumberByTermListGlobal[idTerm];

			dUBlist[threadIdx.x] = dUBlistGlobal[idTerm];//[threadIdx.x];
			dIdfList[threadIdx.x] = dIdfListGlobal[idTerm];//[threadIdx.x];

			iSharedPositionInitialInList[threadIdx.x] = ptrInitPostingList[idTerm];
		}

		//Inicializa a lista de Score e Documentos dos Topk
		//Considero que o Top_K seja um número múltiplo do tamanho do bloco
		for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
			documentTopk.id[localIndex] = -1;
			documentTopk.score[localIndex] = 0.0;
		}

		//Define o max e o min
		if(threadIdx.x < iTermNumber){
			int docAmount = iDocNumberByTermList[threadIdx.x];
			fingers[threadIdx.x].final = 0;
			limitDoc.extraPosition[threadIdx.x] = 0;

			globalIndex = iGlobalInitialPositionInList;

			int maxDoc;
			if(blockIdx.x != 0){
				maxDoc = (globalIndex < docAmount) ? iDocIdList[iSharedPositionInitialInList[threadIdx.x] + globalIndex - 1] : -1;
				maxDoc++;

				atomicMax(&(limitDoc.minDocId), maxDoc);
			}else{
				if(THREAD_MASTER) limitDoc.minDocId = 0;
			}

			int isTail = globalIndex < docAmount;
			globalIndex = globalIndex + DOC_QUANTITY_IN_MEMORY * iGlobalRoundNumber;
			isTail = isTail && globalIndex >= docAmount;

			if(isTail){
				globalIndex = iGlobalInitialPositionInList + (docAmount - iGlobalInitialPositionInList - 1);
			}

			maxDoc =  ( (isTail || (globalIndex < docAmount))
							? (iDocIdList[iSharedPositionInitialInList[threadIdx.x] +  globalIndex-1]) : -1);
			atomicMax(&(limitDoc.secondMaxDocId), maxDoc);
		}

		__syncthreads();

		long long pos;
		int docLocal;
		for (int idTerm = 0; idTerm < iTermNumber; ++idTerm) {
			pos = iSharedPositionInitialInList[idTerm] + iGlobalInitialPositionInList + threadIdx.x;
			docLocal = -1;
			while(pos < (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])
					&& docLocal < limitDoc.minDocId && docLocal <= limitDoc.secondMaxDocId ){
				docLocal = iDocIdList[pos];
				pos += blockDim.x;
			}
			docLocal = ((docLocal != -1)
					&& (docLocal >= limitDoc.minDocId && docLocal <= limitDoc.secondMaxDocId)) ? docLocal : NO_MORE_DOC;
			pos = (docLocal != NO_MORE_DOC) ? pos-blockDim.x : NO_VALID_POSITION;

			atomicMin(&(fingers[idTerm].docId) , docLocal);

			__syncthreads();

			if(fingers[idTerm].docId == docLocal){
				fingers[idTerm].position = pos;
			}
		}

		__syncthreads();

		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

		if(THREAD_MASTER){
			selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
	//	}else if(THREAD_MASTER_2){
			score = 0.0;
		}

		__syncthreads();

		while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

			isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
//			count++;

			if(isValidCandidate){
				  if(threadIdx.x < iTermNumber){
			 		int termId = iOrderedTermSharedList[threadIdx.x];
			 		float scoreL = 0.0;
			 		if(fingers[termId].docId == fingers[sharedPivot.idTerm].docId){
			 			scoreL = scoreTf_Idf(iFreqList[fingers[termId].position],
											iDocLenghtList[fingers[termId].position],
											dIdfList[termId],dAverageDocumentLength,1.0);
			 		}

			 		float aux = 0;
			 		for (int i = 0; i < TERM_NUMBER; ++i) {
			 			aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
			 		}

			 		if(THREAD_MASTER) score = aux;
	//		 		atomicAdd(&score,scoreL);
				}

				padding = documentTopk.padding;

				__syncthreads();

	/*				If the heap is not full
				the candidate is inserted into the heap. If the heap is full
				and the new score is larger than the minimum score in the
				heap, the new document is inserted into the heap, replacing
				the one with the minimum score.

	*/
				if(padding != 0 || thresholdLocal < score ){
					thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
				}

				if(threadIdx.x < iTermNumber ){

				 	int docPivot = fingers[sharedPivot.idTerm].docId;
				 	if(fingers[threadIdx.x].docId ==  docPivot){
				 		fingers[threadIdx.x].position++;
				 		if(fingers[threadIdx.x].position >= (iDocNumberByTermList[threadIdx.x]+iSharedPositionInitialInList[threadIdx.x])){//Não Válido
				 			fingers[threadIdx.x].docId = NO_MORE_DOC;
				 			fingers[threadIdx.x].position = NO_VALID_POSITION;
				 		}else{
				 			fingers[threadIdx.x].docId = iDocIdList[fingers[threadIdx.x].position];
				 			if(fingers[threadIdx.x].docId > limitDoc.secondMaxDocId){
				 				fingers[threadIdx.x].docId = NO_MORE_DOC;
				 				fingers[threadIdx.x].position = NO_VALID_POSITION;
				 			}
				 		}
				 	}
				}
			}
			else{
				int pivotDoc = docCurrent;
				int position;
				int docLocal;
				int idTerm;
				for (int j = 0; j < sharedPivot.positionInOrderedList; ++j) {
					idTerm = iOrderedTermSharedList[j];

					if(fingers[idTerm].docId == fingers[sharedPivot.idTerm].docId)//Até alcançar um finger q aponte a um documento pivo
						break;

					fingers[idTerm].docId = NO_MORE_DOC;
					position = fingers[idTerm].position + 1 + threadIdx.x;
					docLocal = -1;
					while(position < (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])
							&& docLocal < pivotDoc && docLocal <= limitDoc.secondMaxDocId){
						docLocal = iDocIdList[position];
						position += blockDim.x;
					}
					docLocal = (docLocal >= pivotDoc && docLocal <= limitDoc.secondMaxDocId) ? docLocal : NO_MORE_DOC;
					position = (docLocal != NO_MORE_DOC) ? position-blockDim.x : NO_VALID_POSITION;

					__syncthreads();

					atomicMin(&(fingers[idTerm].docId) , docLocal);

					__syncthreads();

					if(fingers[idTerm].docId == docLocal){
						fingers[idTerm].position = position;
					}
				}
			}


			__syncthreads();

			//Sort the terms in non decreasing order of DID
			sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

			//Select term pivot
			if(THREAD_MASTER){
				selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
				docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
				score = 0.0;
			}

			if (SHAREDTHESHOLD == 1){//SHARED_READ
				if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > globalThresholdBatch[idQuery]){
	//							atomicMaxD(&globalThreshold,thresholdLocal);
//					atomicMax((unsigned long long int*)&(globalThresholdBatch[idQuery]),(unsigned long long int)thresholdLocal);
//					atomicMaxD((volatile double*)&(globalThresholdBatch[idQuery]),thresholdLocal);
					globalThresholdBatch[idQuery] = thresholdLocal;
				}

				if(thresholdLocal < globalThresholdBatch[idQuery]){
					thresholdLocal = globalThresholdBatch[idQuery];
				}
			}else if (SHAREDTHESHOLD == 2){ //TSHARED_WRITEREAD
				if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > globalThresholdBatch[idQuery]){
//					atomicMax((unsigned long long int*)&(globalThresholdBatch[idQuery]),(unsigned long long int)thresholdLocal);
//					atomicMaxD(((volatile double*)&(globalThresholdBatch[idQuery])),thresholdLocal);
					 globalThresholdBatch[idQuery] = thresholdLocal;
				}

				if((documentTopk.padding < (iTopK >> 1)))
					if(thresholdLocal < globalThresholdBatch[idQuery]){
						thresholdLocal = globalThresholdBatch[idQuery];
					}
			}
			__syncthreads();
		}

		sortLocalTopkDocAndStoreInGlobal(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);

//		if(thresholdLocal > globalThreshold)
//			globalThreshold = thresholdLocal;

//		if(THREAD_MASTER)
////			atomicAdd(&globalCount,count);
//
//		if(THREAD_MASTER && idQuery == 0)
//			printf("-----%d %d----", blockIdx.x, count);

}


__global__ void matchWandParallel_VARIABLE_4_2(const int* iDocIdList, const unsigned short int* iFreqList,
										  const float *dUBlistGlobal, const float *dIdfListGlobal,
										  const int *iDocLenghtList, const short int iTermNumber, int *iTopkDocListGlobal,
										  float *dTopkScoreListGlobal, const float dAverageDocumentLengthGlobal,
										  const int iGlobalRoundNumber,
										  const short int iTopK, const float iInitialThreshold,
										  const int* iDocNumberByTermListGlobal){

//	if(blockIdx.x != 720)
//		return;
	__shared__ pivot sharedPivot;
	__shared__ finger fingers[TERM_NUMBER];

	__shared__ documentTopkList documentTopk;

	__shared__ postingList2 postings[TERM_NUMBER];

	__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];
	__shared__ long long iSharedPositionInitialInList[TERM_NUMBER];
	__shared__ float dUBlist[TERM_NUMBER];
	__shared__ float dIdfList[TERM_NUMBER];
	__shared__ float dAverageDocumentLength;

	__shared__ int iDocNumberByTermList[TERM_NUMBER];
	__shared__ int iGlobalInitialPositionInList;

	__shared__ float score;
	__shared__ bool isValidCandidate;
	__shared__ int docCurrent;
	__shared__ limitDocId limitDoc;

//	int count = iTopK;

//	__shared__ int paddingInShared;
	int padding;

 	float thresholdLocal = iInitialThreshold;
 	thresholdLocal = iInitialThreshold;

	int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
	int localIndex;
	int positionInitialInTermPostingList;


	if(thresholdLocal < globalThreshold)
		thresholdLocal = globalThreshold;

	if(THREAD_MASTER){
		documentTopk.padding = iTopK;
		dAverageDocumentLength = dAverageDocumentLengthGlobal;
//	}else if(THREAD_MASTER_2){
		iGlobalInitialPositionInList = DOC_QUANTITY_IN_MEMORY  * blockIdx.x * iGlobalRoundNumber;
		limitDoc.minDocId = 0;
		limitDoc.secondMaxDocId = 0;
	}

	if(threadIdx.x < iTermNumber){
//		paddingInShared=0;
		fingers[threadIdx.x].docId = NO_MORE_DOC;
		fingers[threadIdx.x].position = NO_VALID_POSITION;
		iDocNumberByTermList[threadIdx.x] = iDocNumberByTermListGlobal[threadIdx.x];
		dUBlist[threadIdx.x] = dUBlistGlobal[threadIdx.x];
		dIdfList[threadIdx.x] = dIdfListGlobal[threadIdx.x];
	}

	//Inicializa a lista de Score e Documentos dos Topk
	//Considero que o Top_K seja um número múltiplo do tamanho do bloco
	for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
		documentTopk.id[localIndex] = -1;
		documentTopk.score[localIndex] = 0.0;
	}

	//Define o max e o min
	if(threadIdx.x < iTermNumber){
		int docAmount = iDocNumberByTermList[threadIdx.x];
		fingers[threadIdx.x].final = 0;
		limitDoc.extraPosition[threadIdx.x] = 0;

		globalIndex = iGlobalInitialPositionInList;
		positionInitialInTermPostingList = 0;

		for (int i = 0; i < threadIdx.x; ++i) {
			positionInitialInTermPostingList += iDocNumberByTermList[i];
		}
		iSharedPositionInitialInList[threadIdx.x] = positionInitialInTermPostingList;

		int maxDoc;
		if(blockIdx.x != 0){
			maxDoc = (globalIndex < docAmount) ? iDocIdList[positionInitialInTermPostingList + globalIndex - 1] : -1;
			maxDoc++;
			atomicMax(&(limitDoc.minDocId), maxDoc);
		}else{
			if(THREAD_MASTER) limitDoc.minDocId = 0;
		}

		int isTail = globalIndex < docAmount;
		globalIndex += DOC_QUANTITY_IN_MEMORY * iGlobalRoundNumber - 1;
		isTail = (isTail && globalIndex >= docAmount);

		if(isTail){
			globalIndex = iGlobalInitialPositionInList + (docAmount - iGlobalInitialPositionInList - 1);
		}

		maxDoc =  (isTail || globalIndex < docAmount) ? iDocIdList[positionInitialInTermPostingList +  globalIndex] : -1;
		atomicMax(&(limitDoc.secondMaxDocId), maxDoc);
	}

	__syncthreads();

	long long pos;
//	int docLocal;
	for (int idTerm = 0; idTerm < iTermNumber; ++idTerm) {
		pos = iSharedPositionInitialInList[idTerm] + iGlobalInitialPositionInList + threadIdx.x;
		int docLocal = -1;
		while(pos < (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])
				&& docLocal < limitDoc.minDocId && docLocal <= limitDoc.secondMaxDocId ){
			docLocal = iDocIdList[pos];
			pos += blockDim.x;
		}
		docLocal = ( (docLocal != -1)
				&& (docLocal >= limitDoc.minDocId && docLocal <= limitDoc.secondMaxDocId)) ? docLocal : NO_MORE_DOC;

		pos = (docLocal != NO_MORE_DOC) ? pos-blockDim.x : NO_VALID_POSITION;

//		atomicMin(&(fingers[idTerm].docId) , docLocal);

		int docNeighbor, docAux = docLocal;
		for (int i = 16; i >= 1; i /= 2) {
			docNeighbor  = __shfl_down_sync(0xFFFFFFFF,docAux, i);

			if(docNeighbor < docAux)
				docAux = docNeighbor;
		}

		if( ((threadIdx.x & 0x1f) == 0)){
			atomicMin(&(fingers[idTerm].docId) , docAux);
		}


		__syncthreads();

		if(fingers[idTerm].docId == docLocal){
			fingers[idTerm].position = pos;
		}
	}

	__syncthreads();

	for (int termId = 0; termId < iTermNumber; ++termId) {
		if(fingers[termId].position != NO_VALID_POSITION){
			long long gIndex = fingers[termId].position + threadIdx.x;
			for (int localIndex = threadIdx.x; localIndex < DOCS_TEST; localIndex+=blockDim.x) {
				if(gIndex < (iSharedPositionInitialInList[termId]+iDocNumberByTermList[termId])
					&& (iDocIdList[gIndex] <= limitDoc.secondMaxDocId) ){
					postings[termId].docId[localIndex] =  iDocIdList[gIndex];
					postings[termId].freq[localIndex] =   iFreqList[gIndex];
					postings[termId].docLenght[localIndex] =  iDocLenghtList[gIndex];
					if(localIndex == 0) postings[termId].positionInShared = 0;
				}
				else{
					postings[termId].docId[localIndex] =  NO_MORE_DOC;
					if(localIndex == 0) postings[termId].positionInShared = NO_VALID_POSITION;
				}
				gIndex += blockDim.x;
			}
		}
		else{
			postings[termId].positionInShared = NO_VALID_POSITION;
		}
	}


	sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

	if(THREAD_MASTER){
		selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
		docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
		score = 0.0;
	}

	__syncthreads();

	while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

		isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
//		count++;

//		if(fingers[sharedPivot.idTerm].docId==33769946 && THREAD_MASTER)
//			printf("blockId.x %d!!!\n",blockIdx.x);

		if(isValidCandidate){
			if(threadIdx.x < iTermNumber){
		 		int termId = iOrderedTermSharedList[threadIdx.x];
		 		float scoreL = 0.0;
		 		if(fingers[termId].docId == fingers[sharedPivot.idTerm].docId){
		 			scoreL = scoreTf_Idf(postings[termId].freq[postings[termId].positionInShared],
		 					postings[termId].docLenght[postings[termId].positionInShared],
										dIdfList[termId],dAverageDocumentLength,1.0);
		 		}
		 		float aux = 0;
		 		for (int i = 0; i < TERM_NUMBER; ++i) {
		 			aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
		 		}

		 		if(THREAD_MASTER) score = aux;
//		 		atomicAdd(&score,scoreL);
			}

			padding = documentTopk.padding;

			__syncthreads();

/*				If the heap is not full
			the candidate is inserted into the heap. If the heap is full
			and the new score is larger than the minimum score in the
			heap, the new document is inserted into the heap, replacing
			the one with the minimum score.

*/
			if(padding != 0 || thresholdLocal < score ){
//				if(THREAD_MASTER && fingers[sharedPivot.idTerm].docId==6364669)//&& score == 3.53512168))//40920063
//					printf("blockIdx.x %d\n",blockIdx.x);

				thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
//				if(count != 0) count--;
			}

//			float test = checkMinHeapProperty(documentTopk,score,fingers[sharedPivot.idTerm].docId,iTopK);

//			if(count != documentTopk.padding){
//				printf("Padding error! count %d | padding %d |  blockIdx %d | docId %d\n",count, documentTopk.padding, blockIdx.x, fingers[sharedPivot.idTerm].docId);
//			}
//
//			int result = __syncthreads_or(test != 0.0);
//			if(THREAD_MASTER && result != 0){
//				printf("Oi\n");
//				return;
//			}

			if(threadIdx.x < iTermNumber ){
			 	int docPivot = fingers[sharedPivot.idTerm].docId;
			 	int posInShared;
			 	if(fingers[threadIdx.x].docId ==  docPivot){
			 		fingers[threadIdx.x].position++;
			 		postings[threadIdx.x].positionInShared++;
			 		posInShared = postings[threadIdx.x].positionInShared;

			 		if(posInShared >= DOCS_TEST || postings[threadIdx.x].docId[posInShared]  == NO_MORE_DOC){
			 			fingers[threadIdx.x].docId = NO_MORE_DOC;
			 			if(docPivot == docCurrent)
			 				atomicInc((unsigned int*)(&docCurrent),docCurrent);
			 		}else{
			 			fingers[threadIdx.x].docId = postings[threadIdx.x].docId[posInShared];
			 		}
			 	}
			}
		}
		else{
			int pivotDoc = docCurrent;
			int position;
			int docLocal;
			int idTerm;
//			__syncthreads();
			for (int j = 0; j < sharedPivot.positionInOrderedList; ++j) {
				idTerm = iOrderedTermSharedList[j];

				if(fingers[idTerm].docId == pivotDoc)//Até alcançar um finger q aponte a um documento pivo
					break;

				position = postings[idTerm].positionInShared + 1 + threadIdx.x;
				fingers[idTerm].docId = NO_MORE_DOC;

				if(position < DOCS_TEST)
					docLocal = postings[idTerm].docId[position];
				else
					docLocal = NO_MORE_DOC;

				while( (position < DOCS_TEST) && (docLocal < pivotDoc) ){
					docLocal = postings[idTerm].docId[position];
					position += blockDim.x;
				}

				docLocal = (docLocal > pivotDoc) ? docLocal : NO_MORE_DOC;
				position = (docLocal != NO_MORE_DOC) ? position-blockDim.x : DOCS_TEST;

//				__syncthreads();

				int docNeighbor, docAux = docLocal;
				for (int i = 16; i >= 1; i /= 2) {
					docNeighbor  = __shfl_down_sync(0xFFFFFFFF,docAux, i);

					if(docNeighbor < docAux)
						docAux = docNeighbor;
				}

				if( ((threadIdx.x & 0x1f) == 0)){
					atomicMin(&(fingers[idTerm].docId) , docAux);
				}

				__syncthreads();

				if(fingers[idTerm].docId == docLocal){
					if(position != DOCS_TEST){
						fingers[idTerm].position += (position-postings[idTerm].positionInShared);
						postings[idTerm].positionInShared += threadIdx.x + 1;
					}
					else {
						postings[idTerm].positionInShared = DOCS_TEST;
//						fingers[idTerm].position = NO_VALID_POSITION;
					}
				}
			}
		}

		for (int termId = 0; termId < iTermNumber; ++termId) {
			long long gIndex;
			int count=0,isValid=0, docLocal, isOutRange=0;
			if(postings[termId].positionInShared >= DOCS_TEST && postings[termId].positionInShared != NO_VALID_POSITION){
				gIndex = fingers[termId].position + threadIdx.x;
				for (int localIndex = threadIdx.x; localIndex < DOCS_TEST; localIndex+=blockDim.x) {

					count=0;isValid=0;isOutRange=0;
					do{
						isOutRange = gIndex >= (iSharedPositionInitialInList[termId]+iDocNumberByTermList[termId]);
						docLocal = (!isOutRange) ? iDocIdList[gIndex] : NO_MORE_DOC;
						isOutRange = isOutRange || (docLocal > limitDoc.secondMaxDocId);
						isValid =  isOutRange || (docLocal >= docCurrent);

//						count = __syncthreads_count(!isValid);
						count = __ballot_sync(0xFFFFFFFF,!isValid);
						count = __popc(count);

//						if((threadIdx.x & 0x1f) == 0){
//							atomicAdd(&paddingInShared,count);
//						}
//						__syncthreads();
//						count = paddingInShared;
						gIndex += count;
						if(localIndex == 0) fingers[termId].position += count;
					}while(count != 0);

					if(!isOutRange){
						postings[termId].docId[localIndex] = docLocal;
						postings[termId].freq[localIndex] =   iFreqList[gIndex];
						postings[termId].docLenght[localIndex] =  iDocLenghtList[gIndex];
						if(localIndex == 0) postings[termId].positionInShared = 0;
					}
					else{
						postings[termId].docId[localIndex] =  NO_MORE_DOC;
						if(localIndex == 0) postings[termId].positionInShared = NO_VALID_POSITION;
					}
					gIndex += blockDim.x;
				}

				if(threadIdx.x == 0){
					fingers[termId].docId = postings[termId].docId[0];
				}
//				paddingInShared=0;
//				__syncthreads();
			}
		}

		__syncthreads();

		//Sort the terms in non decreasing order of DID
		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

		//Select term pivot
		if(THREAD_MASTER){
			selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
			score = 0.0;
		}

		__syncthreads();


		if (SHAREDTHESHOLD == 1){//SHARED_READ
			if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > globalThreshold){
//				atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
				globalThreshold=thresholdLocal;
			}

			if(thresholdLocal < globalThreshold){
				thresholdLocal = globalThreshold;
			}

		}else if (SHAREDTHESHOLD == 2){ //TSHARED_WRITEREAD
			if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > globalThreshold){
//				atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
				globalThreshold=thresholdLocal;
			}

			if((documentTopk.padding < (iTopK >> 1)))
				if(thresholdLocal < globalThreshold){
					thresholdLocal = globalThreshold;
				}
		}

	}//Fim do WAND - pivot = NO_MORE_DOC


//	for (int i = blockIdx.x*iTopK+threadIdx.x; i < blockIdx.x*iTopK; i+= blockDim.x) {
//		printf("---%d %d---",blockIdx.x,iTopkDocListGlobal[i]);
//	}

	sortLocalTopkDocAndStoreInGlobal(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);


//	__syncthreads();



//	float test = checkSorting(documentTopk, dTopkScoreListGlobal, iTopkDocListGlobal, iTopK);
//
//	int result = __syncthreads_or(test != 0.0);
//	if(THREAD_MASTER && result != 0){
//		printf("Oi no Sorting!\n");
//		return;
//	}

//	if(thresholdLocal > globalThreshold)
//		thresholdGlobal = thresholdLocal;

//	if(THREAD_MASTER)
//		atomicAdd(&globalCount,count);
//////
//	if(THREAD_MASTER)
//		printf("-----%d----", globalCount);
}

__global__ void matchWandParallel_VARIABLE_4(const int* iDocIdList, const unsigned short int* iFreqList,
										  const float *dUBlistGlobal, const float *dIdfListGlobal,
										  const int *iDocLenghtList, const short int iTermNumber, int *iTopkDocListGlobal,
										  float *dTopkScoreListGlobal, const float dAverageDocumentLengthGlobal,
										  const int iGlobalRoundNumber,
										  const short int iTopK, const float iInitialThreshold,
										  const int* iDocNumberByTermListGlobal){

//	if(blockIdx.x != 1104)
//		return;

	__shared__ pivot sharedPivot;
	__shared__ finger fingers[TERM_NUMBER];

	__shared__ documentTopkList documentTopk;

	__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];
	__shared__ long long iSharedPositionInitialInList[TERM_NUMBER];
	__shared__ float dUBlist[TERM_NUMBER];
	__shared__ float dIdfList[TERM_NUMBER];
	__shared__ float dAverageDocumentLength;

	__shared__ int iDocNumberByTermList[TERM_NUMBER];
	__shared__ int iGlobalInitialPositionInList;

	__shared__ float score;
	__shared__ bool isValidCandidate;
	__shared__ int docCurrent;
	__shared__ limitDocId limitDoc;

//	int count = iTopK;

	int padding;

 	float thresholdLocal = iInitialThreshold;
 	thresholdLocal = iInitialThreshold;

	int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
	int localIndex;
	int positionInitialInTermPostingList;

	if(thresholdLocal < globalThreshold)
		thresholdLocal = globalThreshold;

	if(THREAD_MASTER){
		documentTopk.padding = iTopK;
		dAverageDocumentLength = dAverageDocumentLengthGlobal;
//	}else if(THREAD_MASTER_2){
		iGlobalInitialPositionInList = DOC_QUANTITY_IN_MEMORY  * blockIdx.x * iGlobalRoundNumber;
		limitDoc.minDocId = 0;
		limitDoc.secondMaxDocId = 0;
	}

	if(threadIdx.x < iTermNumber){
		fingers[threadIdx.x].docId = NO_MORE_DOC;
		fingers[threadIdx.x].position = NO_VALID_POSITION;
		iDocNumberByTermList[threadIdx.x] = iDocNumberByTermListGlobal[threadIdx.x];
		dUBlist[threadIdx.x] = dUBlistGlobal[threadIdx.x];
		dIdfList[threadIdx.x] = dIdfListGlobal[threadIdx.x];
	}

	//Inicializa a lista de Score e Documentos dos Topk
	//Considero que o Top_K seja um número múltiplo do tamanho do bloco
	for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
		documentTopk.id[localIndex] = -1;
		documentTopk.score[localIndex] = 0.0;
	}

//	__syncthreads();

//	if(blockIdx.x == 83 && THREAD_MASTER)
//		printf("Oi! \n");

	//Define o max e o min
	if(threadIdx.x < iTermNumber){
		int docAmount = iDocNumberByTermList[threadIdx.x];
		fingers[threadIdx.x].final = 0;
		limitDoc.extraPosition[threadIdx.x] = 0;

		globalIndex = iGlobalInitialPositionInList;
		positionInitialInTermPostingList = 0;

		for (int i = 0; i < threadIdx.x; ++i) {
			positionInitialInTermPostingList += iDocNumberByTermList[i];
		}
		iSharedPositionInitialInList[threadIdx.x] = positionInitialInTermPostingList;

		int maxDoc;
		if(blockIdx.x != 0){
			maxDoc = (globalIndex < docAmount) ? iDocIdList[positionInitialInTermPostingList + globalIndex - 1] : -1;
			maxDoc++;
			atomicMax(&(limitDoc.minDocId), maxDoc);
		}else{
			if(THREAD_MASTER) limitDoc.minDocId = 0;
		}

		int isTail = globalIndex < docAmount;
		globalIndex += DOC_QUANTITY_IN_MEMORY * iGlobalRoundNumber - 1;
		isTail = (isTail && globalIndex >= docAmount);

		if(isTail){
			globalIndex = iGlobalInitialPositionInList + (docAmount - iGlobalInitialPositionInList - 1);
		}

		maxDoc =  (isTail || globalIndex < docAmount) ? iDocIdList[positionInitialInTermPostingList +  globalIndex] : -1;
		atomicMax(&(limitDoc.secondMaxDocId), maxDoc);
	}

	__syncthreads();

	long long pos;
	int docLocal;
	for (int idTerm = 0; idTerm < iTermNumber; ++idTerm) {
		pos = iSharedPositionInitialInList[idTerm] + iGlobalInitialPositionInList + threadIdx.x;
		docLocal = -1;
		while(pos < (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])
				&& docLocal < limitDoc.minDocId && docLocal <= limitDoc.secondMaxDocId ){
			docLocal = iDocIdList[pos];
			pos += blockDim.x;
		}
		docLocal = ( (docLocal != -1)
				&& (docLocal >= limitDoc.minDocId && docLocal <= limitDoc.secondMaxDocId)) ? docLocal : NO_MORE_DOC;
		pos = (docLocal != NO_MORE_DOC) ? pos-blockDim.x : NO_VALID_POSITION;

		atomicMin(&(fingers[idTerm].docId) , docLocal);

		__syncthreads();

		if(fingers[idTerm].docId == docLocal){
			fingers[idTerm].position = pos;
		}
	}

	sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

	if(THREAD_MASTER){
		selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
		docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
//	}else if(THREAD_MASTER_2){
		score = 0.0;
	}

	__syncthreads();

	while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

		isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
//		count++;

		if(isValidCandidate){
			if(threadIdx.x < iTermNumber){
		 		int termId = iOrderedTermSharedList[threadIdx.x];
		 		float scoreL = 0.0;
		 		if(fingers[termId].docId == fingers[sharedPivot.idTerm].docId){
		 			scoreL = scoreTf_Idf(iFreqList[fingers[termId].position],
										iDocLenghtList[fingers[termId].position],
										dIdfList[termId],dAverageDocumentLength,1.0);
		 		}

		 		float aux = 0;
		 		for (int i = 0; i < TERM_NUMBER; ++i) {
		 			aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
		 		}

		 		if(THREAD_MASTER) score = aux;
//		 		atomicAdd(&score,scoreL);
			}

			padding = documentTopk.padding;

			__syncthreads();

/*				If the heap is not full
			the candidate is inserted into the heap. If the heap is full
			and the new score is larger than the minimum score in the
			heap, the new document is inserted into the heap, replacing
			the one with the minimum score.

*/
			if(padding != 0 || thresholdLocal < score ){
				if(THREAD_MASTER && fingers[sharedPivot.idTerm].docId==46517642)//&& score == 3.53512168))//40920063
					printf("blockIdx.x %d\n",blockIdx.x);

				thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
//				if(count != 0) count--;
			}

//			float test = checkMinHeapProperty(documentTopk,score,fingers[sharedPivot.idTerm].docId,iTopK);

//			if(count != documentTopk.padding){
//				printf("Padding error! count %d | padding %d |  blockIdx %d | docId %d\n",count, documentTopk.padding, blockIdx.x, fingers[sharedPivot.idTerm].docId);
//			}
//
//			int result = __syncthreads_or(test != 0.0);
//			if(THREAD_MASTER && result != 0){
//				printf("Oi\n");
//				return;
//			}

			if(threadIdx.x < iTermNumber ){

			 	int docPivot = fingers[sharedPivot.idTerm].docId;
			 	if(fingers[threadIdx.x].docId ==  docPivot){
			 		fingers[threadIdx.x].position++;
			 		if(fingers[threadIdx.x].position >= (iDocNumberByTermList[threadIdx.x]+iSharedPositionInitialInList[threadIdx.x])){//Não Válido
			 			fingers[threadIdx.x].docId = NO_MORE_DOC;
			 			fingers[threadIdx.x].position = NO_VALID_POSITION;
			 		}else{
			 			fingers[threadIdx.x].docId = iDocIdList[fingers[threadIdx.x].position];
			 			if(fingers[threadIdx.x].docId > limitDoc.secondMaxDocId){
			 				fingers[threadIdx.x].docId = NO_MORE_DOC;
			 				fingers[threadIdx.x].position = NO_VALID_POSITION;
			 			}
			 		}
			 	}
			}
		}
		else{
			int pivotDoc = docCurrent;
			long long position;
			int docLocal;
			int idTerm;
			for (int j = 0; j < sharedPivot.positionInOrderedList; ++j) {
				idTerm = iOrderedTermSharedList[j];

				if(fingers[idTerm].docId == fingers[sharedPivot.idTerm].docId)//Até alcançar um finger q aponte a um documento pivo
					break;

				fingers[idTerm].docId = NO_MORE_DOC;
				position = fingers[idTerm].position + 1 + threadIdx.x;
				docLocal = -1;
				while(position < (iSharedPositionInitialInList[idTerm]+iDocNumberByTermList[idTerm])
						&& docLocal < pivotDoc && docLocal <= limitDoc.secondMaxDocId){
					docLocal = iDocIdList[position];
					position += blockDim.x;
				}
				docLocal = (docLocal >= pivotDoc && docLocal <= limitDoc.secondMaxDocId) ? docLocal : NO_MORE_DOC;
				position = (docLocal != NO_MORE_DOC) ? position-blockDim.x : NO_VALID_POSITION;

				__syncthreads();

				atomicMin(&(fingers[idTerm].docId) , docLocal);

				__syncthreads();

				if(fingers[idTerm].docId == docLocal){
					fingers[idTerm].position = position;
				}
			}
		}

		__syncthreads();

		//Sort the terms in non decreasing order of DID
		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

		//Select term pivot
		if(THREAD_MASTER){
			selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
			score = 0.0;
		}

		__syncthreads();

		if (SHAREDTHESHOLD == 1){//SHARED_READ
			if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > globalThreshold){
	//							atomicMaxD(&globalThreshold,thresholdLocal);
//				atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
				globalThreshold = thresholdLocal;
			}

			if(thresholdLocal < globalThreshold){
				thresholdLocal = globalThreshold;
			}
		}else if (SHAREDTHESHOLD == 2){ //TSHARED_WRITEREAD
			if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > globalThreshold){
//				atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
//								atomicMaxD(&globalThreshold,thresholdLocal);
				globalThreshold = thresholdLocal;
			}

			if((documentTopk.padding < (iTopK >> 1)))
				if(thresholdLocal < globalThreshold){
					thresholdLocal = globalThreshold;
				}
		}

	}

//	for (int i = blockIdx.x*iTopK+threadIdx.x; i < blockIdx.x*iTopK; i+= blockDim.x) {
//		printf("---%d %d---",blockIdx.x,iTopkDocListGlobal[i]);
//	}

	sortLocalTopkDocAndStoreInGlobal(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);

//	float test = checkSorting(documentTopk, dTopkScoreListGlobal, iTopkDocListGlobal, iTopK);
//
//	int result = __syncthreads_or(test != 0.0);
//	if(THREAD_MASTER && result != 0){
//		printf("Oi no Sorting!\n");
//		return;
//	}

//	if(thresholdLocal > globalThreshold)
//		thresholdGlobal = thresholdLocal;

//	if(THREAD_MASTER)
//		atomicAdd(&globalCount,count);
//////
//	if(THREAD_MASTER)
//		printf("-----%d----", globalCount);
}



__global__ void matchWandParallel_FIXED_3(const int* iDocIdList, const unsigned short int* iFreqList,
										  const float *dUBlistGlobal, const float *dIdfListGlobal,
										  const int *iDocLenghtList, const short int iTermNumber, int *iTopkDocListGlobal,
										  float *dTopkScoreListGlobal, const float dAverageDocumentLengthGlobal,
										  const int iGlobalRoundNumber,
										  const short int iTopK, const float iInitialThreshold,
										  const int* iDocNumberByTermListGlobal){
	//	if(blockIdx.x != 0)
	//		return;

		__shared__ pivot sharedPivot;
		__shared__ finger fingers[TERM_NUMBER];

		__shared__ documentTopkList documentTopk;

		__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];
//		__shared__ long long int iSharedPositionInitialInList[TERM_NUMBER];
		__shared__ float dUBlist[TERM_NUMBER];
		__shared__ float dIdfList[TERM_NUMBER];
		__shared__ float dAverageDocumentLength;

		__shared__ int iDocNumberByTermList[TERM_NUMBER];
		__shared__ int iGlobalInitialPositionInList;

		__shared__ float score;
		__shared__ bool isValidCandidate;
		__shared__ int docCurrent;


		__shared__ long long finalPositions[TERM_NUMBER];
	//	int count =0;

		int padding;

	 	float thresholdLocal = iInitialThreshold;
	 	thresholdLocal = iInitialThreshold;

		int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
		int localIndex;
		long long int positionInitialInTermPostingList;

		if(thresholdLocal < globalThreshold)
			thresholdLocal = globalThreshold;

		if(THREAD_MASTER){
			documentTopk.padding = iTopK;
			dAverageDocumentLength = dAverageDocumentLengthGlobal;
			iGlobalInitialPositionInList = DOC_QUANTITY_IN_MEMORY  * blockIdx.x * iGlobalRoundNumber;
		}

		if(threadIdx.x < iTermNumber){
			iDocNumberByTermList[threadIdx.x] = iDocNumberByTermListGlobal[threadIdx.x];

			dUBlist[threadIdx.x] = dUBlistGlobal[threadIdx.x];
			dIdfList[threadIdx.x] = dIdfListGlobal[threadIdx.x];

			globalIndex = iGlobalInitialPositionInList;
			positionInitialInTermPostingList = 0;

			for (int i = 0; i < threadIdx.x; ++i) {
				positionInitialInTermPostingList += iDocNumberByTermList[i];
			}
//			iSharedPositionInitialInList[threadIdx.x] = positionInitialInTermPostingList;

			fingers[threadIdx.x].position = positionInitialInTermPostingList + globalIndex;

			if(fingers[threadIdx.x].position < (positionInitialInTermPostingList+iDocNumberByTermList[threadIdx.x])){
				fingers[threadIdx.x].docId = iDocIdList[fingers[threadIdx.x].position];
			}else{
				fingers[threadIdx.x].position = NO_VALID_POSITION;
				fingers[threadIdx.x].docId = NO_PIVOT_TERM;
			}

			finalPositions[threadIdx.x] = positionInitialInTermPostingList + globalIndex + DOC_QUANTITY_IN_MEMORY * iGlobalRoundNumber;

			if(finalPositions[threadIdx.x] >= (positionInitialInTermPostingList+iDocNumberByTermList[threadIdx.x]))
				finalPositions[threadIdx.x] = positionInitialInTermPostingList+iDocNumberByTermList[threadIdx.x];
		}

		//Inicializa a lista de Score e Documentos dos Topk
		//Considero que o Top_K seja um número múltiplo do tamanho do bloco
		for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
			documentTopk.id[localIndex] = -1;
			documentTopk.score[localIndex] = 0.0;
		}

		__syncthreads();

		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

		if(THREAD_MASTER){
			selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
	//	}else if(THREAD_MASTER_2){
			score = 0.0;
		}

		__syncthreads();

		while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){

			isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
	//		count++;

			if(isValidCandidate){
				if(threadIdx.x < iTermNumber){
			 		int termId = iOrderedTermSharedList[threadIdx.x];
			 		float scoreL = 0.0;
			 		if(fingers[termId].docId == fingers[sharedPivot.idTerm].docId){
			 			scoreL = scoreTf_Idf(iFreqList[fingers[termId].position],
											iDocLenghtList[fingers[termId].position],
											dIdfList[termId],dAverageDocumentLength,1.1);
			 		}

			 		float aux = 0;
			 		for (int i = 0; i < TERM_NUMBER; ++i) {
			 			aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
			 		}

			 		if(THREAD_MASTER) score = aux;
	//		 		atomicAdd(&score,scoreL);
				}

				padding = documentTopk.padding;

				__syncthreads();

	/*				If the heap is not full
				the candidate is inserted into the heap. If the heap is full
				and the new score is larger than the minimum score in the
				heap, the new document is inserted into the heap, replacing
				the one with the minimum score.

	*/
				if(padding != 0 || thresholdLocal < score ){
					thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
				}

				if(threadIdx.x < iTermNumber ){

				 	int docPivot = fingers[sharedPivot.idTerm].docId;
				 	if(fingers[threadIdx.x].docId ==  docPivot){
				 		fingers[threadIdx.x].position++;
				 		if(fingers[threadIdx.x].position >= finalPositions[threadIdx.x] ){//Não Válido
				 			fingers[threadIdx.x].docId = NO_MORE_DOC;
				 			fingers[threadIdx.x].position = NO_VALID_POSITION;
				 		}else{
				 			fingers[threadIdx.x].docId = iDocIdList[fingers[threadIdx.x].position];
				 		}
				 	}
				}
			}
			else{
				int pivotDoc = docCurrent;
				long long int position;
				int docLocal;
				int idTerm;
				for (int j = 0; j < sharedPivot.positionInOrderedList; ++j) {
					idTerm = iOrderedTermSharedList[j];

					if(fingers[idTerm].docId == fingers[sharedPivot.idTerm].docId)//Até alcançar um finger q aponte a um documento pivo
						break;

					fingers[idTerm].docId = NO_MORE_DOC;
					position = fingers[idTerm].position + 1 + threadIdx.x;
					docLocal = -1;
					while(position < finalPositions[idTerm] && docLocal < pivotDoc){
						docLocal = iDocIdList[position];
						position += blockDim.x;
					}
					position -= blockDim.x;
					if((docLocal < pivotDoc ||  position >= finalPositions[idTerm])){
						docLocal = NO_MORE_DOC;
						position = NO_VALID_POSITION;
					}

					__syncthreads();

					atomicMin(&(fingers[idTerm].docId) , docLocal);

					__syncthreads();

					if(fingers[idTerm].docId == docLocal){
						fingers[idTerm].position = position;
					}
				}
			}

			__syncthreads();

			//Sort the terms in non decreasing order of DID
			sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

			//Select term pivot
			if(THREAD_MASTER){
				selectTermPivot_No_SharedMemory(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
				docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
				score = 0.0;
			}

			__syncthreads();

			if (SHAREDTHESHOLD == 1){//SHARED_READ
				if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > globalThreshold){
		//							atomicMaxD(&globalThreshold,thresholdLocal);
//					atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
					globalThreshold = thresholdLocal;
				}

				if(thresholdLocal < globalThreshold){
					thresholdLocal = globalThreshold;
				}
			}else if (SHAREDTHESHOLD == 2){ //TSHARED_WRITEREAD
				if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > globalThreshold){
//					atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
		//							atomicMaxD(&globalThreshold,thresholdLocal);
					globalThreshold = thresholdLocal;
				}

				if((documentTopk.padding < (iTopK >> 1)))
					if(thresholdLocal < globalThreshold){
						thresholdLocal = globalThreshold;
					}
			}
		}

		sortLocalTopkDocAndStoreInGlobal(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);



//		if(thresholdLocal > globalThreshold)
//			globalThreshold = thresholdLocal;

	//	if(THREAD_MASTER)
	//		atomicAdd(&globalCount,count);
	////
	//	if(THREAD_MASTER)
	//		printf("-----%d----", globalCount);
}

__global__ void matchWandParallel_VARIABLE_3(const int* iDocIdList, const unsigned short int* iFreqList,
										  const float *dUBlist, const float *dIdfList,
										  const int *iDocLenghtList, const short int iTermNumber, int *iTopkDocListGlobal,
										  float *dTopkScoreListGlobal, const float dAverageDocumentLength,
										  const int iGlobalRoundNumber,
										  const short int iTopK, const float iInitialThreshold,
										  const int* iDocNumberByTermList){
//		if(blockIdx.x != 1687)
//			return;

//		int count = 0;
		__shared__ pivot sharedPivot;
		__shared__ finger fingers[TERM_NUMBER];

		__shared__ postingList postingLists[TERM_NUMBER];
		__shared__ documentTopkList documentTopk;

		__shared__ unsigned int iOrderedTermSharedList[TERM_NUMBER];
//		__shared__ float dUBlist[TERM_NUMBER];

//		__shared__ int iDocNumberByTermList[TERM_NUMBER];


		__shared__ int iGlobalInitialPositionInList;
		__shared__ unsigned short int iElementQuantityByBlock;

		__shared__ float score;
		__shared__ bool isValidCandidate;
		__shared__ int docCurrent;

		__shared__ short int needSearchDocRange[TERM_NUMBER];
		__shared__ limitDocId limitDoc;

	 	float thresholdLocal;// = iInitialThreshold;

//	 	int count = 0;

	 	thresholdLocal = iInitialThreshold;

		int globalIndex;// = iInitialPositionGlobal + threadIdx.x;
		int localIndex;
		int positionInitialInTermPostingList;

		if(thresholdLocal < globalThreshold)
			thresholdLocal = globalThreshold;

		if(THREAD_MASTER){
			documentTopk.padding = iTopK;
		}else if(THREAD_MASTER_2){
			iElementQuantityByBlock = DOC_QUANTITY_IN_MEMORY;//iBlockRoundNumber * DOC_QUANTITY_IN_MEMORY;
			iGlobalInitialPositionInList = iElementQuantityByBlock  * blockIdx.x * iGlobalRoundNumber;
		}

		//Inicializa a lista de Score e Documentos dos Topk
		//Considero que o Top_K seja um número múltiplo do tamanho do bloco
		for (localIndex = threadIdx.x; localIndex < iTopK; localIndex += blockDim.x) {
			documentTopk.id[localIndex] = -1;
			documentTopk.score[localIndex] = 0.0;
		}

	//	if(THREAD_MASTER) documentTopk.padding = iTopK;

		__syncthreads();

		//Define o max e o min
		if(threadIdx.x < iTermNumber){
//			iDocNumberByTermList[threadIdx.x] = globalDocNumberByTermList[threadIdx.x];
			fingers[threadIdx.x].final = 0;
			limitDoc.extraPosition[threadIdx.x] = 0;
//			dUBlist[threadIdx.x] = dUBlistGlobal[blockIdx.x * iTermNumber + threadIdx.x];
			int docAmount = iDocNumberByTermList[threadIdx.x];
			globalIndex = iGlobalInitialPositionInList;
			positionInitialInTermPostingList = 0;

			for (int i = 0; i < threadIdx.x; ++i) {
				positionInitialInTermPostingList += iDocNumberByTermList[i];
			}
//			if(threadIdx.x == 0 && blockIdx.x == 1687){
//				printf("Oi");
//			}
			int aux, maxDoc;
			int maxNeighbor;
			if(blockIdx.x != 0){
				int maxDoc = (globalIndex < docAmount) ? iDocIdList[positionInitialInTermPostingList + globalIndex - 1] : -1;
				maxDoc++;
				aux = maxDoc;

//				atomicMax(&limitDoc.minDocId, maxDoc);
//				__syncwarp(0xFFFFFFFF);
				for (int i = 1; i < iTermNumber; ++i) {
					maxNeighbor = __shfl_sync(0xFFFFFFFF,aux,i);
					if(maxNeighbor > maxDoc)
						maxDoc = maxNeighbor;
				}
	//
				if(THREAD_MASTER) {
					limitDoc.minDocId = maxDoc; //atomicExch(&(limitDoc.minDocId), maxDoc);
				}
				__syncwarp(0xFFFFFFFF);

				if(aux < limitDoc.minDocId && aux != 0)
					needSearchDocRange[threadIdx.x] = 1;
			}else
				if(THREAD_MASTER) limitDoc.minDocId = 0;

			int isTail = globalIndex < docAmount;
			globalIndex += iElementQuantityByBlock * iGlobalRoundNumber;
			isTail &= globalIndex >= docAmount;

			if(isTail){
				globalIndex = iGlobalInitialPositionInList + (docAmount - iGlobalInitialPositionInList - 1);
			}

			maxDoc =  (isTail || globalIndex < docAmount) ? iDocIdList[positionInitialInTermPostingList +  globalIndex] - 1 :
															-1;
			aux = maxDoc;
			for (int i = 1; i < iTermNumber; ++i) {
				maxNeighbor = __shfl_sync(0xFFFFFFFF,aux,i);
				if(maxNeighbor > maxDoc)
					maxDoc = maxNeighbor;
			}
			if(THREAD_MASTER) limitDoc.secondMaxDocId = maxDoc;
		}

		__syncthreads();

		//Busca faixa de documentos;
		for (int termId = 0; termId < iTermNumber; ++termId) {
			if(needSearchDocRange[termId])
				searchRangeOfDocs(iDocIdList,postingLists, termId,
								  iGlobalInitialPositionInList, &limitDoc,
								  iElementQuantityByBlock,iGlobalRoundNumber,iDocNumberByTermList);
		}

		__syncthreads();

		//Preenche a memória compartilhada
		positionInitialInTermPostingList = 0;
		int docLocal, docAmount;
		for (int termId = 0; termId < iTermNumber; ++termId) {
			globalIndex = iGlobalInitialPositionInList + limitDoc.extraPosition[termId] + threadIdx.x;
			docAmount = iDocNumberByTermList[termId];
			docLocal = -1;

			for (localIndex = threadIdx.x; localIndex < iElementQuantityByBlock; localIndex+=blockDim.x) {

				docLocal = (globalIndex < docAmount) ? iDocIdList[positionInitialInTermPostingList + globalIndex]
				                                                  : NO_MORE_DOC;

				if(docLocal > limitDoc.secondMaxDocId || globalIndex > docAmount){
					postingLists[termId].docId[localIndex] = NO_MORE_DOC;
					fingers[termId].final = 1;
					break;
				}

				postingLists[termId].docId[localIndex] = docLocal;
				postingLists[termId].docLenght[localIndex] = iDocLenghtList[positionInitialInTermPostingList + globalIndex];
				postingLists[termId].freq[localIndex] = iFreqList[positionInitialInTermPostingList + globalIndex];

				globalIndex += blockDim.x;
			}

			positionInitialInTermPostingList += iDocNumberByTermList[termId];
		}

		if(threadIdx.x < iTermNumber){
			fingers[threadIdx.x].docId = postingLists[threadIdx.x].docId[0];
			fingers[threadIdx.x].position = (fingers[threadIdx.x].docId == NO_MORE_DOC) ? NO_VALID_POSITION : 0;
//			fingers[threadIdx.x].final = 0 | fingers[threadIdx.x].final;
		}

//		if(threadIdx.x == 0 && blockIdx.x == 3430){
//			printf("Oi");
//		}
		__syncthreads();

//		__shared__ int docCurrent;

		sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

	//	__syncthreads();

		if(THREAD_MASTER){
			selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
			docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
		}else if(THREAD_MASTER_2){
			score = 0.0;
		}

		int padding;
		int threadIdInWarp = (threadIdx.x & 0x1f);
		int idWarp = ((blockDim.x >> 5) == 1 ) ? 1 :  threadIdx.x >> 5;

		__syncthreads();

		while((sharedPivot.positionInOrderedList < iTermNumber) && (sharedPivot.idTerm < iTermNumber)){


			if(THREAD_MASTER){
				isValidCandidate = (fingers[sharedPivot.idTerm].docId == fingers[iOrderedTermSharedList[0]].docId);
			}
			__syncthreads();

			if(isValidCandidate){

				if(threadIdx.x < iTermNumber){
					fullScore_3_1(&score, fingers[sharedPivot.idTerm].docId, iOrderedTermSharedList,
								  fingers,postingLists, dIdfList, dAverageDocumentLength);
				}

				padding = documentTopk.padding;

				__syncthreads();

/*				If the heap is not full
				the candidate is inserted into the heap. If the heap is full
				and the new score is larger than the minimum score in the
				heap, the new document is inserted into the heap, replacing
				the one with the minimum score.

 */
				if(padding != 0 || thresholdLocal < score ){
					thresholdLocal = managerMinValue_v5(&documentTopk, fingers[sharedPivot.idTerm].docId, score,padding);
				}

				if(idWarp == 1 && threadIdInWarp < iTermNumber ){
					advancePivoTermFinger_4(sharedPivot,fingers, postingLists,iElementQuantityByBlock,threadIdInWarp);
				}
			}
			else{
				 advanceDocIdOfPredecessorTerm_4(postingLists,
											   iOrderedTermSharedList,
											   fingers,sharedPivot,fingers[sharedPivot.idTerm].docId,
											   iElementQuantityByBlock);
			}

			__syncthreads();

			for (int termId = 0; termId < iTermNumber; ++termId) {
				if(fingers[termId].docId == NO_MORE_DOC && 	fingers[termId].final == 0){
//					if(termId == 0) count++;
//
//					if(blockIdx.x == 27 && count == 48 && THREAD_MASTER)
//						printf("Oi!");

					searchMoreDocs(iDocIdList,iFreqList,iDocLenghtList,postingLists,
								  termId,iGlobalInitialPositionInList,
								  &limitDoc,iElementQuantityByBlock,
								  &(fingers[termId]),docCurrent,iDocNumberByTermList);


					//#endif

					if (SHAREDTHESHOLD == 1){//SHARED_READ
						if(THREAD_MASTER && documentTopk.padding == 0 && thresholdLocal > globalThreshold){
//							atomicMaxD(&globalThreshold,thresholdLocal);
							atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
						}

						if(thresholdLocal < globalThreshold){
							thresholdLocal = globalThreshold;
						}
					}else if (SHAREDTHESHOLD == 2){ //TSHARED_WRITEREAD
						if(THREAD_MASTER && (documentTopk.padding < (iTopK >> 1)) && thresholdLocal > globalThreshold){
							atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
//							atomicMaxD(&globalThreshold,thresholdLocal);
						}

						if((documentTopk.padding < (iTopK >> 1)))
							if(thresholdLocal < globalThreshold){
								thresholdLocal = globalThreshold;
							}
					}
					//#endif

				}
			}


	//		__syncthreads();//Talvez não precise
			//Sort the terms in non decreasing order of DID
			sortingTerms_2(fingers, iOrderedTermSharedList, iTermNumber);

	//		__syncthreads();//Talvez não precise

			//Select term pivot
			if(THREAD_MASTER){
				selectTermPivot_2(&sharedPivot,iOrderedTermSharedList,fingers,dUBlist,iTermNumber,thresholdLocal);
				docCurrent = (sharedPivot.idTerm != NO_PIVOT_TERM) ? fingers[sharedPivot.idTerm].docId : NO_MORE_DOC;
				score = 0.0;
			}
			__syncthreads();
		}

//		if(threadIdx.x == 0){// && blockIdx.x == 1687){
//			printf("----%d %d----",blockIdx.x,count);
//		}

		sortLocalTopkDocAndStoreInGlobal(dTopkScoreListGlobal,iTopkDocListGlobal,iTopK,&documentTopk);
//		globalIndex =  iTopK * blockIdx.x + threadIdx.x + documentTopk.padding;
//		for (localIndex = threadIdx.x; localIndex < (iTopK - documentTopk.padding) ; localIndex += blockDim.x) {
//			iTopkDocListGlobal[globalIndex]   = documentTopk.id[localIndex];
//			dTopkScoreListGlobal[globalIndex] = documentTopk.score[localIndex];
//			globalIndex += blockDim.x;
//		}
//		__syncthreads();

		if(THREAD_MASTER && thresholdLocal > globalThreshold){
			atomicMax((unsigned long long int*)&globalThreshold,(unsigned long long int)thresholdLocal);
		}
}
