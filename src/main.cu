/*
 * main.c
 *
 *  Created on: 06/12/2017
 *      Author: roussian
 */
#include "HostManager.cuh"
#include <stdio.h>

int main(int argc, char *argv[])
{
//	hipSetDevice(0);
	//Argumentos
	if( argc < 5 ) {
		printf( "\n Parametros incorretos.\n Uso: <top_K>, <blockSize>, <BlockRoundNumber>, <iGlobalNumberRound>,"
				" <MergeNumberByBlock> <QueryType> onde: \n" );
		printf( "\t <top_K> - quantidade de documentos retornados (precisa ser multiplo do blockSize).\n" );
		printf( "\t <blockSize> - tamanho do bloco.\n" );
		printf( "\t <BlockRoundNumber> - numero de partes continuas que cada bloco ira processar.\n" );
		printf( "\t <GlobalNumberRound> - numero de partes nao continuas das listas invertidas que cada bloco ira processar.\n" );
		printf( "\t <MergeNumberByBlock> - numero de merge que cada bloco irá executar.\n" );
		printf( "\t <QueryType> (Optional) - [0] OR Query  --- [1] AND Query.\n" );
        return 0;
	}

	//Quantidade de postings em cada lista em função do tamanho do bloco que cada bloco irá processar
	int iTopk = atoi( argv[1] );
	int iBlockSize = atoi( argv[2] );
	int iBlockNumberRound = atoi( argv[3] );
	int iGlobalNumberRound = atoi( argv[4] );
	int iMergeNumberByBlock = atoi( argv[5] );
	int iQueryType = 0;

	if(argc == 7)
	   iQueryType = atoi( argv[6] );


//	#ifdef BATCH
//		queryBatchProcessingHost_Mix(iTopk, iBlockSize, iBlockNumberRound, iGlobalNumberRound, iMergeNumberByBlock, iQueryType);
//	#else
	querySingleProcessingHost(iTopk, iBlockSize, iBlockNumberRound, iGlobalNumberRound, iMergeNumberByBlock,  iQueryType, 1);
//	#endif

//	queryBatchProcessingHost_ByBlock(iTopk, iBlockSize, iBlockNumberRound,
//							 iGlobalNumberRound, iMergeNumberByBlock, iQueryType,500);


	exit(EXIT_SUCCESS);
}

