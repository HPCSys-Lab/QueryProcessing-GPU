#include "hip/hip_runtime.h"
/*
 * DeviceFunctions.cu
 *
 *  Created on: 08/12/2017
 *      Author: roussian
 */
#include "DeviceFunctions.cuh"
#include "Structs.cuh"
#include <stdio.h>


 __device__  float scoreTf_Idf(int tf, int dDocLength, float idf,
							  float averageDocumentLength, float keyFrequency){
	float k_1 = 1.2;
	float b = 0.75;
	float robertsonTf = (k_1 * tf) / ( tf + (k_1 * ((1 - b) + (b * dDocLength) / averageDocumentLength)));

	return keyFrequency * robertsonTf * idf;
}


__device__  float atomicAddD(double* address, double val){

	unsigned long long int* address_as_ull = (unsigned long long int*) address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
		// Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}


__device__  double atomicExchD(double volatile *address, double volatile val){

	unsigned long long int* address_as_ull = (unsigned long long int*) address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;

		old = atomicExch(address_as_ull, __double_as_longlong(val));
		// Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}


__device__  double atomicMaxD(double volatile *address, double volatile val){

	unsigned long long int* address_as_ull = (unsigned long long int*) address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;

		old = atomicMax(address_as_ull, __double_as_longlong(val));
		// Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}


 __device__  void sortingTerms_2(finger* fingers,
								unsigned int *iOrderedTermListShared,
								const short int iTermNumber){

		if(threadIdx.x < iTermNumber){
			int ownValue = fingers[threadIdx.x].docId;
			int otherThreadValue = 0;
			int position = 0;
			for (int round = 0; round < iTermNumber; ++round) {
				otherThreadValue = __shfl_sync(0xFFFFFFFF,ownValue, round);
				if((otherThreadValue < ownValue) ||
						(otherThreadValue == ownValue && round > threadIdx.x)){
					position++;
				}
			}
			iOrderedTermListShared[position] = threadIdx.x;//O id da thread representa o termo que será apontado.
		}
}

__device__  void selectTermPivot_No_SharedMemory(pivot* pivot,
								   unsigned int *iOrderedTermListShared,
								   finger* fingers,
								   const float *dUBlist,
								   int iTermNumber,
								   float threshold){
	float sumUB = 0.0;
	int iPivotPosition = 0;

	do {
		sumUB += dUBlist[iOrderedTermListShared[iPivotPosition]];
		iPivotPosition++;
	} while ( (iPivotPosition < iTermNumber)
				&& (sumUB < threshold)
				&& (fingers[iOrderedTermListShared[iPivotPosition]].docId != NO_MORE_DOC));
	iPivotPosition--;

// 	if(fingers[0].position == NO_VALID_POSITION && fingers[1].position == NO_VALID_POSITION)
// 		printf("Oi");

	if( (sumUB < threshold) || iPivotPosition >= iTermNumber || (fingers[iOrderedTermListShared[iPivotPosition]].docId == NO_MORE_DOC)){
		pivot->positionInOrderedList = NO_PIVOT_TERM;
		pivot->idTerm = NO_PIVOT_TERM;
// 		if(sumUB > threshold)
// 			printf("--%d---",blockIdx.x);
	}
	else{
// 	 	if(fingers[iOrderedTermListShared[iPivotPosition]].position >= DOC_QUANTITY_IN_MEMORY)
// 	 		printf("Eita");

		pivot->positionInOrderedList = iPivotPosition;
		pivot->idTerm = iOrderedTermListShared[iPivotPosition];
	}
}

__device__  void selectTermPivot_2(pivot* pivot,
 								   unsigned int *iOrderedTermListShared,
 								   finger* fingers,
 								   const float *dUBlist,
 								   int iTermNumber,
 								   float threshold){
 	float sumUB = 0.0;
 	int iPivotPosition = 0;

 	do {
 		sumUB += dUBlist[iOrderedTermListShared[iPivotPosition]];
 		iPivotPosition++;
 	} while ( (iPivotPosition < iTermNumber) && (sumUB < threshold) && (fingers[iOrderedTermListShared[iPivotPosition]].docId != NO_MORE_DOC));
		iPivotPosition--;
// 	if(fingers[0].position == NO_VALID_POSITION && fingers[1].position == NO_VALID_POSITION)
// 		printf("Oi");

 	if( (sumUB < threshold) || iPivotPosition >= iTermNumber || (fingers[iOrderedTermListShared[iPivotPosition]].position >= DOC_QUANTITY_IN_MEMORY)
 			|| (fingers[iOrderedTermListShared[iPivotPosition]].docId == NO_MORE_DOC)){
 		pivot->positionInOrderedList = NO_PIVOT_TERM;
 		pivot->idTerm = NO_PIVOT_TERM;
// 		if(sumUB > threshold)
// 			printf("--%d---",blockIdx.x);
 	}
 	else{
// 	 	if(fingers[iOrderedTermListShared[iPivotPosition]].position >= DOC_QUANTITY_IN_MEMORY)
// 	 		printf("Eita");

 		pivot->positionInOrderedList = iPivotPosition;
 		pivot->idTerm = iOrderedTermListShared[iPivotPosition];
 	}
}

  __device__ void advancePivoTermFinger_2(pivot pivot,
 									    finger *fingers,
 									    postingList *postingLists){

 	int docPivot =  fingers[pivot.idTerm].docId;

 	if(fingers[threadIdx.x].docId ==  docPivot){

 		fingers[threadIdx.x].position++;

 		if(fingers[threadIdx.x].position >= postingLists[threadIdx.x].maxIndex ||
 				fingers[threadIdx.x].position >= NO_VALID_POSITION){//Não Válido
 			fingers[threadIdx.x].docId = NO_MORE_DOC;
 			fingers[threadIdx.x].position = NO_VALID_POSITION;
 		}else{
 			fingers[threadIdx.x].docId = postingLists[threadIdx.x].docId[fingers[threadIdx.x].position];
 		}
 	}
}

/*
 * Avança em paralelo os ponteiros das listas invertidas anterior ao termo pivô;
 * Cada thread obtem o seu elemento de acordo com a posição do atual docId
 * e verifica se o elemento que lhe pertence é o primeiro maior docId.
 */
__device__ void advanceDocIdOfPredecessorTerm_3(postingList *postingList,
											    unsigned int *iOrderedTermListShared,
											    finger *fingers, pivot pivot,
											    const int iBlockRoundNumber){
	int threadPosition;
	int idTerm;

	int docId;
	int docFingerOld;
//	int warpNumber = blockDim.x >> 5;

	int warpId = threadIdx.x >> 5;
	int idThreadInWarp = threadIdx.x - (warpId << 5);

	while(warpId < pivot.positionInOrderedList && warpId < TERM_NUMBER){

		idTerm = iOrderedTermListShared[warpId];
		docFingerOld = fingers[idTerm].docId;

		if(fingers[idTerm].docId == fingers[pivot.idTerm].docId){
//			if(warpId == 0)//Se entrar, está errado algo!
//				printf("Erro: Primeiro elemento igual ao pivo idTerm %d doc %d\n",idTerm,fingers[pivot.idTerm].docId);
			break;
		}

		threadPosition = idThreadInWarp + fingers[idTerm].position + 1;
		docId = 0;//(iPositionThread < iBlockRoundNumber * blockDim.x) ? postingList[iTermId].docId[iPositionThread] : 0;

		while(threadPosition < postingList[warpId].maxIndex
				&& docId < fingers[pivot.idTerm].docId){

			docId = postingList[idTerm].docId[threadPosition];
			threadPosition += warpSize;
		}
		threadPosition -= warpSize;

		if((docId >= fingers[pivot.idTerm].docId)
					&& (postingList[idTerm].docId[threadPosition - 1] < fingers[pivot.idTerm].docId)){
			fingers[idTerm].docId = docId;
			fingers[idTerm].position = threadPosition;
		}

		if(docFingerOld == fingers[idTerm].docId && ((threadIdx.x & 0x1f) == 0) ){
			fingers[idTerm].docId = NO_MORE_DOC;
			fingers[idTerm].position = NO_VALID_POSITION;
		}
		warpId += (blockDim.x >> 5);
	}
}

 __device__ void advancePivoTermFinger_4(pivot pivot,
 									    finger *fingers,
 									    postingList *postingLists,
 									    unsigned short int iElementQuantityByBlock,
 									    int threadIdInWarp){

 	int docPivot =  fingers[pivot.idTerm].docId;

 	if(fingers[threadIdInWarp].docId ==  docPivot){

 		fingers[threadIdInWarp].position++;

 		if(fingers[threadIdInWarp].position >= iElementQuantityByBlock){//Não Válido
 			fingers[threadIdInWarp].docId = NO_MORE_DOC;
 			fingers[threadIdInWarp].position = NO_VALID_POSITION;
 		}else{
 			fingers[threadIdInWarp].docId = postingLists[threadIdInWarp].docId[fingers[threadIdInWarp].position];

 			if(fingers[threadIdInWarp].docId == NO_MORE_DOC)
 				fingers[threadIdInWarp].position = NO_VALID_POSITION;
 		}
 	}
 }

  __device__ void advanceDocIdOfPredecessorTerm_4(postingList *postingList,
 											    unsigned int *iOrderedTermListShared,
 											    finger *fingers, pivot pivot, int docPivot,
 											    const unsigned short int iElementQuantityByBlock){
 	int threadPosition;
 	int idTerm;

 	int docId;
 	int docFingerOld;

 	int warpId = threadIdx.x >> 5;
 	int idThreadInWarp = threadIdx.x - (warpId << 5);

 	//Race
 	int positionFinger;

 	while(warpId < pivot.positionInOrderedList && warpId < TERM_NUMBER){

 		idTerm = iOrderedTermListShared[warpId];
 		positionFinger = fingers[idTerm].position;
 		docFingerOld = fingers[idTerm].docId;

 		if(fingers[idTerm].docId == docPivot){
 			break;
 		}

 		threadPosition = idThreadInWarp + positionFinger + 1;
 		docId = 0;//(iPositionThread < iBlockRoundNumber * blockDim.x) ? postingList[iTermId].docId[iPositionThread] : 0;

 		while(threadPosition < iElementQuantityByBlock
 				&& docId < docPivot){

 			docId = postingList[idTerm].docId[threadPosition];
 			threadPosition += warpSize;
 		}
 		threadPosition -= warpSize;

 		if((docId >= docPivot)
 					&& (postingList[idTerm].docId[threadPosition - 1] < docPivot)){
 			fingers[idTerm].docId = docId;
 			fingers[idTerm].position = threadPosition;
 		}

 		warpId += (blockDim.x >> 5);
 	}

 	__syncthreads();


 	if( ((threadIdx.x & 0x1f) == 0) && (threadIdx.x >> 5) < pivot.positionInOrderedList){

 		idTerm = iOrderedTermListShared[threadIdx.x >> 5];
 		if((threadIdx.x >> 5) < pivot.positionInOrderedList &&
 				docFingerOld == fingers[idTerm].docId ){

 			fingers[idTerm].docId = NO_MORE_DOC;
 			fingers[idTerm].position = NO_VALID_POSITION;
 		}
 	}
}

__device__ void fullScore_3(float *score, pivot pivot,
							unsigned int *iOrderedTermSharedList,
							finger *fingers,
							postingList *postingList,
							const float *dIdfList,
							const float dAverageDocLength){

	if(threadIdx.x < TERM_NUMBER){
		int termId = iOrderedTermSharedList[threadIdx.x];
		float scoreL = 0.0;

		if(fingers[termId].docId == fingers[pivot.idTerm].docId){

			scoreL = scoreTf_Idf(postingList[termId].freq[fingers[termId].position],
								postingList[termId].docLenght[fingers[termId].position],
								dIdfList[termId],dAverageDocLength,1.1);

		}

		float aux = 0;
		#pragma unroll 2
		for (int i = 0; i < TERM_NUMBER; ++i) {
			aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
		}

		if(THREAD_MASTER)
			*score = aux;
	}
	__syncthreads();
}

 __device__ void fullScore_3_1(float *score, int docPivot,
 						  	 unsigned int *iOrderedTermSharedList,
 						  	 finger *fingers,
 						  	 postingList *postingList,
 						  	 const float *dIdfList,
 						  	 const float dAverageDocLength){

 		int termId = iOrderedTermSharedList[threadIdx.x];
 		float scoreL = 0.0;

 		if(fingers[termId].docId == docPivot){

 			scoreL = scoreTf_Idf(postingList[termId].freq[fingers[termId].position],
 								postingList[termId].docLenght[fingers[termId].position],
 								dIdfList[termId],dAverageDocLength,1.1);

 		}

 		float aux = 0;
 		for (int i = 0; i < TERM_NUMBER; ++i) {
 			aux += __shfl_sync(0xFFFFFFFF,scoreL,i);
 		}

 		if(THREAD_MASTER)
 			*score = aux;
}


 __device__ void insertValueInEmptyList_2(documentTopkList *documentTokList,
		 	 	 	 	 	 	 	 	 int newDocId, float newScore){
	documentTokList->id[0] = newDocId;
	documentTokList->score[0] = newScore;
	documentTokList->padding--;
}

 __device__  void insertValueInFullList_2(documentTopkList *topkList, int newDocId, float newScore){
//												documentTopkList* documentTemp){

	int position = (threadIdx.x < TOP_K) ? threadIdx.x : NO_VALID_POSITION;
	int nextPosition = position + 1;

	float nextScore, ownScore;
	int nextDocId;

//	#pragma unroll 4
	for (int i = 0; i < TOP_K/blockDim.x; ++i) {
		ownScore = (position == NO_VALID_POSITION) ? 0.0 : topkList->score[position];
		if(position != NO_VALID_POSITION){
			nextScore = (nextPosition < TOP_K) ? topkList->score[nextPosition] : ownScore;
			nextDocId = (nextPosition < TOP_K) ? topkList->id[nextPosition] : topkList->id[position];
		}

//		__syncthreads();

		if( (ownScore <= newScore) || ( (ownScore >= newScore) && (nextScore == 0.0)) ){
			if(nextScore <= newScore && nextPosition != TOP_K){
				topkList->score[position] = nextScore;
				topkList->id[position]  = nextDocId;
			}else{
				topkList->score[position] = newScore;
				topkList->id[position]  = newDocId;
			}
		}
		position += blockDim.x;
		nextPosition += blockDim.x;
	}
}


 __device__ void insertValueInNotFullList_2(documentTopkList *topkList,
													int newDocId, float newScore){
	int position = (threadIdx.x < TOP_K) ? (TOP_K - threadIdx.x - 1) : NO_VALID_POSITION;
	int previousPosition = (position == 0) ? position : position - 1;

	float previousScore, ownScore;
	int previousDocId;

//	if(THREAD_MASTER && topkList->score[1] >  newScore)
//		topkList->score[0] = newScore;

	#pragma unroll 4
	for (int i = 0; i < TOP_K/blockDim.x; ++i) {

		ownScore = (position == NO_VALID_POSITION) ? 0.0 : topkList->score[position];
		if(position != NO_VALID_POSITION){
			previousScore = (previousPosition >= 0) ? topkList->score[previousPosition] : ownScore;
			previousDocId = (previousPosition >= 0) ? topkList->id[previousPosition] : topkList->id[position];
		}

//		__syncthreads();

		if(ownScore >= newScore || (previousScore != 0.0 && ownScore == 0.0)){
			if(previousScore < newScore || position == 0){
				topkList->score[position] = newScore;
				topkList->id[position]  = newDocId;
			}else{
				topkList->score[position] = previousScore;
				topkList->id[position]  = previousDocId;
			}
		}

		position -= blockDim.x;
		previousPosition -= blockDim.x;
	}

	if(THREAD_MASTER)
		(topkList->padding)--;

//	__syncthreads();
}

 __device__ void insertValueInNotFullList_Heap(documentTopkList *topkList,
													int newDocId, float newScore){


	int insertPosition = TOP_K - topkList->padding;

	if(THREAD_MASTER)
		(topkList->padding)--;

//	if(insertPosition <= 2){//Inserção básica: os filhos da raiz não estão preenchidos
//		topkList->id[insertPosition] = newDocId;
//		topkList->score[insertPosition] = newScore;
//		return;
//	}

	int height =  log2f(insertPosition+1);

	//Uma thread para cada ancestral do nó positionInsert, com exceção da raiz
	if(threadIdx.x <= height){ //&& threadIdx.x != 0){
		int parentPosition,initialPosition = insertPosition;
		int elementNumberInLevel = (1 << height);

		int doc;
		float score;

		insertPosition = ( insertPosition - (elementNumberInLevel - 1) ) / ( elementNumberInLevel >> threadIdx.x );
		insertPosition += (1 << threadIdx.x) - 1;

		if(topkList->score[insertPosition] > newScore || initialPosition == insertPosition){

			parentPosition = (insertPosition - 1) >> 1;
			if(topkList->score[parentPosition] > newScore){
				doc = topkList->id[parentPosition];
				score = topkList->score[parentPosition];
			}else{//No level 1, sempre o pai é menor, pois é a raiz
				doc = newDocId;
				score = newScore;
			}

			topkList->id[insertPosition] = doc;
			topkList->score[insertPosition] =  score;
		}
	}
//	__syncthreads();
}


__device__ inline float insertValueInFullList_Heap(documentTopkList *topkList,
													int newDocId, float newScore){

//	if(blockIdx.x != 0) return 2;

	int height = HEIGHT_HEAP;
	//3º Passo = Se a propriedade do heap-min for desfeito, encontrar o caminho para inserir o novo elemento
	if(threadIdx.x <= height){

		//1º Passo = Extrai o menor valor (raiz) e aumenta a chave (substitui a raiz com o novo valor)
		topkList->id[0]    = newDocId;
		topkList->score[0] = newScore;

		//2º Passo = Identificar a raiz do subHeap que será alterado
		int subHeapRootIndex = 0;// = (topkList->score[1] < topkList->score[2]) ? 1 : 2;

//		if(topkList->score[subHeapRootIndex] < newScore){
			int level = 0;
			int doc;
			float score;

			int positionChild;

//			if(threadIdx.x == 0) subHeapRootIndex = 0;

			while (level < threadIdx.x  && topkList->score[subHeapRootIndex] <= newScore){
				subHeapRootIndex <<= 1;
				subHeapRootIndex++;
				if(subHeapRootIndex + 1 < TOP_K )
					if(topkList->score[subHeapRootIndex + 1] <= topkList->score[subHeapRootIndex])//Obtém o index do filho que tem o menor  valor
						subHeapRootIndex++;
				level++;
			}

			if(subHeapRootIndex >= TOP_K || topkList->score[subHeapRootIndex] > newScore){
				level--;
				subHeapRootIndex = -1;
			}

//			while(level < threadIdx.x && topkList->score[subHeapRootIndex] <= newScore){
//				subHeapRootIndex <<= 1;
//				subHeapRootIndex++;
////				if(subHeapRootIndex + 1 < TOP_K )
//					if(topkList->score[subHeapRootIndex + 1] <= topkList->score[subHeapRootIndex])//Obtém o index do filho que tem o menor  valor
//						subHeapRootIndex++;
//
//				if(topkList->score[subHeapRootIndex] < newScore)
//					level++;
//			}

//			if(level != threadIdx.x)
//				subHeapRootIndex = -1;

			positionChild = __shfl_down_sync(0xFFFFFFFF,subHeapRootIndex, 1);

			if(threadIdx.x ==  height)  //thread que está na folha
				positionChild = -1;

			if(level == threadIdx.x){

				if(positionChild != -1){// && ((threadIdx.x <  height)) {
					doc = topkList->id[positionChild];
					score = topkList->score[positionChild];
				}
				else{
					doc = newDocId;
					score = newScore;
				}

//				__syncwarp();

				topkList->id[subHeapRootIndex] = doc;
				topkList->score[subHeapRootIndex] = score;
			}
//		}
	}
//	else{
//		int subHeapRootIndex = (topkList->score[1] < topkList->score[2]) ? 1 : 2;
//
//		if(topkList->score[subHeapRootIndex] < topkList->score[0])
//			return topkList->score[subHeapRootIndex];
//	}

//	__syncthreads();
	return topkList->score[0];
}

 __device__ float managerMinValue_v4(documentTopkList *documentTopk,int newDocId, float newScore, int padding){

	if(padding == 0){
		insertValueInFullList_2(documentTopk,newDocId,newScore);
	}else if(padding == TOP_K){
		if(THREAD_MASTER) insertValueInEmptyList_2(documentTopk,newDocId, newScore);
	}else{
		insertValueInNotFullList_2(documentTopk, newDocId, newScore);
	}

	__syncthreads();

	return documentTopk->score[0];;
}


 __device__ float managerMinValue_v5(documentTopkList *documentTopk, int newDocId, float newScore, int padding){

	float min;

	if(padding == 0){
		insertValueInFullList_Heap(documentTopk,newDocId,newScore);
	}else if(padding == TOP_K){
		if(THREAD_MASTER) insertValueInEmptyList_2(documentTopk,newDocId, newScore);
		min = newScore;
	}else{
		insertValueInNotFullList_Heap(documentTopk, newDocId, newScore);
		min = documentTopk->score[0];
	}

	min = documentTopk->score[0];
	return min;
}

 __device__ void searchRangeOfDocs(const int* iDocIdList,
		  	  	  	  	  	  	  postingList *postingLists, int termId,
		  	  	  	  	  	  	  int iGlobalInitialPositionInList, limitDocId *limitDoc,
		  	  	  	  	  	  	  unsigned short int iElementQuantityByBlock,
		  	  	  	  	  	  	  int roundGlobalNumber,const int *iDocNumberByTermList){



	//	__shared__ int newPosition;
		int positionInListGlobal = 0;
		int globalIndex = iGlobalInitialPositionInList + threadIdx.x;

		if(THREAD_MASTER)//Race
		limitDoc->extraPosition[termId] = NO_MORE_DOC;

		__syncthreads();

		#pragma unroll 2
		for (int i = 0; i < termId; ++i) {
			positionInListGlobal += iDocNumberByTermList[i];
		}

		int docLocal = -1;
		while (docLocal < limitDoc->minDocId && globalIndex < iDocNumberByTermList[termId]){
			docLocal = iDocIdList[positionInListGlobal + globalIndex];
			globalIndex += blockDim.x;
		}
		globalIndex-= blockDim.x;
		long long int initialPosition;

		if(docLocal < limitDoc->minDocId)//Caso não encontre
			initialPosition = NO_VALID_POSITION;
		else
			initialPosition = globalIndex - iGlobalInitialPositionInList;

		int positionNeighbor;
		for (int i = 16; i >= 1; i /= 2) {
			positionNeighbor  = __shfl_down_sync(0xFFFFFFFF,initialPosition, i);

			if(positionNeighbor < initialPosition)
				initialPosition = positionNeighbor;
		}

	//	__syncthreads();

		if( ((threadIdx.x & 0x1f) == 0) && initialPosition != NO_MORE_DOC){
			atomicMin(&(limitDoc->extraPosition[termId]) , initialPosition);
		}
		__syncthreads();

		if( THREAD_MASTER && limitDoc->extraPosition[termId] == NO_MORE_DOC){
			limitDoc->extraPosition[termId] = 0;
		}

		if(THREAD_MASTER){
			globalIndex = iGlobalInitialPositionInList + iElementQuantityByBlock * roundGlobalNumber + threadIdx.x;
			globalIndex += limitDoc->extraPosition[termId];
	//
			if(globalIndex < iDocNumberByTermList[termId]){
				if(limitDoc->secondMaxDocId < iDocIdList[positionInListGlobal + globalIndex] -1)
					limitDoc->secondMaxDocId = iDocIdList[positionInListGlobal + globalIndex]-1;
			}
		}
	//
	//	if(THREAD_MASTER) limitDoc->extraPosition[termId] = newPosition;
}


__device__ void searchRangeOfDocs_batch(const int* iDocIdList,
		  	  	  	  	  	  	  	  	postingList *postingLists, int termId,
		  	  	  	  	  	  	  	  	int iGlobalInitialPositionInList, limitDocId *limitDoc,
		  	  	  	  	  	  	  	  	unsigned short int iElementQuantityByBlock,
		  	  	  	  	  	  	  	  	int roundGlobalNumber,int iPostingListSize,
		  	  	  	  	  	  	  	  	long long positionInListGlobal){

//	__shared__ int newPosition;
//		long long positionInListGlobal = ptrPostingPositions[termId];
	int globalIndex = iGlobalInitialPositionInList + threadIdx.x;

//	if(((threadIdx.x & 0x1f) == 0) && limitDoc->extraPosition[termId] == 0)//if(THREAD_MASTER)//Race
//		limitDoc->extraPosition[termId] = NO_MORE_DOC;

//	__syncthreads();

//		for (int i = 0; i < termId; ++i) {
//			positionInListGlobal += iDocNumberByTermList[i];
//		}

	int docLocal = -1;
	while (docLocal < limitDoc->minDocId && globalIndex < iPostingListSize){
		docLocal = iDocIdList[positionInListGlobal + globalIndex];
		globalIndex += blockDim.x;
	}
	globalIndex -= blockDim.x;

	long long int initialPosition;
	if(docLocal < limitDoc->minDocId)//Caso não encontre
		initialPosition = NO_VALID_POSITION;
	else
		initialPosition = globalIndex - iGlobalInitialPositionInList;

	int positionNeighbor;
	for (int i = 16; i >= 1; i /= 2) {
		positionNeighbor  = __shfl_down_sync(0xFFFFFFFF,initialPosition, i);

		if(positionNeighbor < initialPosition)
			initialPosition = positionNeighbor;
	}

//	__syncthreads();

	if( ((threadIdx.x & 0x1f) == 0) && initialPosition != NO_VALID_POSITION){
//		atomicMax(&(limitDoc->extraPosition[termId]) , initialPosition);
		atomicMin(&(limitDoc->extraPosition[termId]) , initialPosition);
	}

	__syncthreads();

	if( THREAD_MASTER ){//1066

		if (limitDoc->extraPosition[termId] == NO_MORE_DOC)
			limitDoc->extraPosition[termId] = 0;

		globalIndex = iGlobalInitialPositionInList + iElementQuantityByBlock * roundGlobalNumber + threadIdx.x;
		globalIndex += limitDoc->extraPosition[termId];

		if(globalIndex < iPostingListSize){
			if(limitDoc->secondMaxDocId < iDocIdList[positionInListGlobal + globalIndex] - 1)
				limitDoc->secondMaxDocId = iDocIdList[positionInListGlobal + globalIndex] - 1;
		}
	}

//	if(THREAD_MASTER){
//		globalIndex = iGlobalInitialPositionInList + iElementQuantityByBlock * roundGlobalNumber + threadIdx.x;
//		globalIndex += limitDoc->extraPosition[termId];
//
//		if(globalIndex < iPostingListSize){
//			if(limitDoc->secondMaxDocId < iDocIdList[positionInListGlobal + globalIndex] -1)
//				limitDoc->secondMaxDocId = iDocIdList[positionInListGlobal + globalIndex]-1;
//		}
//	}
//
//	if(THREAD_MASTER) limitDoc->extraPosition[termId] = newPosition;
}

 __device__ void searchMoreDocs(const int* iDocIdList,
							   const unsigned short int* iFreqList,  const int *iDocLengthList,
		  	  	  	  	  	   postingList *postingLists, int termId,
		  	  	  	  	  	   int iGlobalInitialPositionInList, limitDocId *limitDoc,
		  	  	  	  	  	   unsigned short int iElementQuantityByBlock,
		  	  	  	  	  	   finger *fingerT, int docCurrent,const int *iDocNumberByTermList){

	int globalIndex = iGlobalInitialPositionInList + limitDoc->extraPosition[termId] + iElementQuantityByBlock + threadIdx.x;
	int docLocal, positionInListGlobal=0;
	int localIndex;
	int docAmount = iDocNumberByTermList[termId];

	__shared__ int lowerAmount;
	lowerAmount =0;
//	#pragma unroll 2
	for (int i = 0; i < termId; ++i) {
		positionInListGlobal += iDocNumberByTermList[i];
	}
	docLocal = (globalIndex < docAmount) ? iDocIdList[positionInListGlobal + globalIndex] : NO_MORE_DOC;

//	if(blockIdx.x == 1230 && THREAD_MASTER)
//			printf("Atualizando Termo %d \n", termId);
//
//	if(docLocal < docCurrent){
//		if(THREAD_FINAL || ( (globalIndex + 1) == docAmount) || iDocIdList[positionInListGlobal + globalIndex + 1] > docCurrent){
//			lowerAmount = threadIdx.x;
//		}
//	}
//
	int isLowerLocal = (docLocal < docCurrent);
	#pragma unroll 16
	for (int i = 16; i >= 1; i /= 2) {
		isLowerLocal += __shfl_down_sync(0xFFFFFFFF,isLowerLocal, i);
	}

	if( ((threadIdx.x & 0x1f) == 0) && isLowerLocal != 0){
		atomicAdd(&lowerAmount, isLowerLocal);
	}

	__syncthreads();
//
//	isLowerLocal = lowerAmount;
	globalIndex += lowerAmount;
	for (localIndex = threadIdx.x; localIndex < iElementQuantityByBlock; localIndex += blockDim.x) {

		docLocal = (globalIndex < docAmount) ? iDocIdList[positionInListGlobal + globalIndex] : NO_MORE_DOC;

		if(docLocal > limitDoc->secondMaxDocId){
				postingLists[termId].docId[localIndex] = NO_MORE_DOC;
				fingerT->final = 1;
				break;
		}

		postingLists[termId].docId[localIndex] = docLocal;
		postingLists[termId].docLenght[localIndex] = iDocLengthList[positionInListGlobal + globalIndex];
		postingLists[termId].freq[localIndex] = iFreqList[positionInListGlobal + globalIndex];

		globalIndex += blockDim.x;
	}

//	__syncthreads();

	if(THREAD_MASTER){
		fingerT->docId = postingLists[termId].docId[0];
		fingerT->position = (fingerT->docId == NO_MORE_DOC) ? NO_VALID_POSITION : 0;
		if (fingerT->docId == NO_MORE_DOC) fingerT->position = 1;
		limitDoc->extraPosition[termId] += iElementQuantityByBlock + lowerAmount;//isLowerLocal + iElementQuantityByBlock;
	}
}


__device__ void searchMoreDocs_batch(const int* iDocIdList, const unsigned short int* iFreqList,
									 const int *iDocLengthList, postingList *postingLists, int termId,
									 int iGlobalInitialPositionInList, limitDocId *limitDoc,
									 unsigned short int iElementQuantityByBlock, finger *fingerT,
									 int docCurrent, int iPostingListSize, long long positionInGlobalList){

	int globalIndex = iGlobalInitialPositionInList + limitDoc->extraPosition[termId] + iElementQuantityByBlock + threadIdx.x;
	int docLocal;//, positionInListGlobal=0;
	int localIndex;
//	int docAmount = iPostingListSize;

	__shared__ int lowerAmount;
	lowerAmount =0;

//	for (int i = 0; i < termId; ++i) {
//		positionInListGlobal += iDocNumberByTermList[i];
//	}
	docLocal = (globalIndex < iPostingListSize) ? iDocIdList[positionInGlobalList + globalIndex] : NO_MORE_DOC;

//	if(blockIdx.x == 1230 && THREAD_MASTER)
//			printf("Atualizando Termo %d \n", termId);
//
//	if(docLocal < docCurrent){
//		if(THREAD_FINAL || ( (globalIndex + 1) == docAmount) || iDocIdList[positionInListGlobal + globalIndex + 1] > docCurrent){
//			lowerAmount = threadIdx.x;
//		}
//	}
//
	int isLowerLocal = (docLocal < docCurrent);
	#pragma unroll 16
	for (int i = 16; i >= 1; i /= 2) {
		isLowerLocal += __shfl_down_sync(0xFFFFFFFF,isLowerLocal, i);
	}

	if( ((threadIdx.x & 0x1f) == 0) && isLowerLocal != 0){
		atomicAdd(&lowerAmount, isLowerLocal);
	}

	__syncthreads();

//	isLowerLocal = lowerAmount;
	globalIndex += lowerAmount;
	for (localIndex = threadIdx.x; localIndex < iElementQuantityByBlock; localIndex += blockDim.x) {

		docLocal = (globalIndex < iPostingListSize) ? iDocIdList[positionInGlobalList + globalIndex] : NO_MORE_DOC;

		if(docLocal > limitDoc->secondMaxDocId){
			postingLists[termId].docId[localIndex] = NO_MORE_DOC;
			fingerT->final = 1;
			break;
		}

		postingLists[termId].docId[localIndex] = docLocal;
		postingLists[termId].docLenght[localIndex] = iDocLengthList[positionInGlobalList + globalIndex];
		postingLists[termId].freq[localIndex] = iFreqList[positionInGlobalList + globalIndex];

		globalIndex += blockDim.x;
	}

//	__syncthreads();

	if(THREAD_MASTER){
		fingerT->docId = postingLists[termId].docId[0];
		fingerT->position = (fingerT->docId == NO_MORE_DOC) ? NO_VALID_POSITION : 0;
		if (fingerT->docId == NO_MORE_DOC) fingerT->position = 1;
		limitDoc->extraPosition[termId] += iElementQuantityByBlock + lowerAmount;//isLowerLocal + iElementQuantityByBlock;
	}
}


__device__ void sortLocalTopkDocAndStoreInGlobal_BLOCK( float *dTopkScoreListGlobal,int *iTopkDocListGlobal, int iTopk,
												documentTopkList *localTopkDoc){

//	if(blockIdx.x == 4999 && threadIdx.x == 32)
//		printf("okay");

	float score_1, scoreAux, scoreNeighborAux; //score_3
	int position_1;//, position_2;//position_3;
	int threadIdInWarp = (threadIdx.x & 0x1f);
	int maxIndex = iTopk - localTopkDoc->padding;
	int globalIndex = iTopk * blockIdx.x;// + localTopkDoc->padding;

	for (int localIndex = threadIdx.x; localIndex < maxIndex; localIndex += blockDim.x) {
		position_1 = 0;
//		position_2 = 0;

		score_1 = localTopkDoc->score[localIndex];
//		score_2 = (localIndex + blockDim.x < maxIndex) ? localTopkDoc->score[localIndex + blockDim.x] : 0.0;

		for (int i = threadIdInWarp; i < maxIndex; i+=32) {
			scoreAux = localTopkDoc->score[i];

			for (int t = 0; t < 32; ++t) {
				scoreNeighborAux = 0.0;
				scoreNeighborAux = __shfl_sync(0xFFFFFFFF,scoreAux, t);

				if(threadIdInWarp == maxIndex - 1)
					scoreNeighborAux = 0.0;

				if(threadIdInWarp == 31 && i + 1 < maxIndex){
					scoreNeighborAux = localTopkDoc->score[i + 1];
				}

				if(score_1 < scoreNeighborAux || (score_1 == scoreNeighborAux && (i+t) < localIndex)){
					position_1++;
				}


//
//				if(scoreNeighborAux != 0.0){
//					if(score_1 < scoreNeighborAux || (score_1 == scoreNeighborAux && (i+t) < localIndex)){
//						position_1++;
//					}
//				}
			}
		}

		dTopkScoreListGlobal[globalIndex + position_1] = score_1;
		iTopkDocListGlobal[globalIndex + position_1] = localTopkDoc->id[localIndex];

		globalIndex += blockDim.x;

//		if(localIndex+blockDim.x < maxIndex){
//			globalIndex += blockDim.x;
//			dTopkScoreListGlobal[globalIndex + position_2] = score_2;
//			iTopkDocListGlobal[globalIndex + position_2] = localTopkDoc->id[localIndex + blockDim.x];
//		}

//		if(localIndex + localIndex + (blockDim.x << 1) < maxIndex){
//			globalIndex += blockDim.x;
//			dTopkScoreListGlobal[globalIndex + position_3] = score_3;
//			iTopkDocListGlobal[globalIndex + position_3] = localTopkDoc->id[localIndex + (blockDim.x << 1)];
//		}
	}

//	int length = TOP_K - localTopkDoc->padding;
//	int height = log2f(length);
//
//	if(threadIdx.x < height){
//		int maxIndex = length - 1;
//		int subHeapRootIndex, positionChild, level, doc;
//		float score;
//
//		for (int i = (length - 1); i > 0; i--) {
//
//			doc = localTopkDoc->id[i];
//			score = localTopkDoc->score[i];
//
//			localTopkDoc->id[i] = localTopkDoc->id[0];
//			localTopkDoc->score[i] = localTopkDoc->score[0];
//
//			maxIndex--;
//			if((1 << height) - 1 > maxIndex)
//				height--;
//
//			if(threadIdx.x > height) break;
//
//			level = 0;
//			subHeapRootIndex = 0;
//			while(level < threadIdx.x && subHeapRootIndex <= maxIndex && localTopkDoc->score[subHeapRootIndex] <= score){
//				subHeapRootIndex <<= 1;
//				subHeapRootIndex++;
//				if(subHeapRootIndex + 1 <= maxIndex )
//					if(localTopkDoc->score[subHeapRootIndex + 1] < localTopkDoc->score[subHeapRootIndex])//Obtém o index do filho que tem o menor  valor
//						subHeapRootIndex++;
//
//				level++;
//			}
//
//			if(level != threadIdx.x ||  subHeapRootIndex > maxIndex)
//				subHeapRootIndex = -1;
//
//			positionChild = __shfl_down(subHeapRootIndex, 1);
//
//			if(level == threadIdx.x && subHeapRootIndex != -1){
//
//				if(positionChild != -1 && (threadIdx.x != height)) {
//					doc = localTopkDoc->id[positionChild];
//					score = localTopkDoc->score[positionChild];
//				}
//
//				localTopkDoc->id[subHeapRootIndex] = doc;
//				localTopkDoc->score[subHeapRootIndex] = score;
//			}
//		}
//	}
//
//	length--;
//	int globalIndex =  iTopk * blockIdx.x + threadIdx.x - localTopkDoc->padding;
//	int localIndex = blockDim.x - 1 - threadIdx.x - length;
//
//	__syncthreads();
//
//	for (; localIndex >= 0; localIndex -= blockDim.x) {
//		iTopkDocListGlobal[globalIndex]   = localTopkDoc->id[localIndex];
//		dTopkScoreListGlobal[globalIndex] = localTopkDoc->score[localIndex];
//		globalIndex += blockDim.x;
//	}
}




__device__ void sortLocalTopkDocAndStoreInGlobal(float *dTopkScoreListGlobal,int *iTopkDocListGlobal, int iTopk,
												documentTopkList *localTopkDoc){
//	float score_1=0.0, scoreAux=0.0, scoreNeighborAux=0.0;//score_3 score_2=0.0,
//	int position_1=0;//, position_2=0;//,position_3;
//	int threadIdInWarp = (threadIdx.x & 0x1f);
//	int maxIndex = iTopk - localTopkDoc->padding;
//	int maxIndexThreads = ((maxIndex >> 5) <<5);
//	maxIndexThreads = (maxIndex > maxIndexThreads) ?  (maxIndexThreads + 32) : maxIndexThreads;
//	int globalIndex = iTopk * blockIdx.x + localTopkDoc->padding; //threadIdx.x + localTopkDoc->padding;
//	int activeThreads;

//	if(maxIndex >= 32)
//		activeThreads = 32;
//	else
//		activeThreads = maxIndex;
//
//
//	for (int localIndex = threadIdx.x; localIndex < maxIndexThreads; localIndex+=blockDim.x) {
//
//		if(localIndex < maxIndex){
//			score_1 = localTopkDoc->score[localIndex];
//			position_1 = 0;
//		}
//
//		for (int i = threadIdInWarp; i < maxIndexThreads; i+=32) {
//			if(i < maxIndex)
//				scoreAux = localTopkDoc->score[i];
//
//			for (int t = 0; t < activeThreads; ++t) {
//				scoreNeighborAux = __shfl_sync(0xFFFFFFFF,scoreAux, t);
//
//				if(score_1 > scoreNeighborAux || (score_1 == scoreNeighborAux &&  ((i >> 5) << 5) + t < localIndex )){// i > localIndex ) ){ //(i+t) < localIndex)){
//					position_1++;
//				}
//			}
//			activeThreads = maxIndex - (((i+32) >> 5) << 5); //Gato: valor de i precisa estar atualizado
//		}
//
//		if(localIndex < maxIndex){
//			dTopkScoreListGlobal[globalIndex + position_1] = score_1;
//			iTopkDocListGlobal[globalIndex + position_1] = localTopkDoc->id[localIndex];
//			if(maxIndex >= 32)
//					activeThreads = 32;
//				else
//					activeThreads = maxIndex;
//		}
//	}

	int globalIndex = iTopk * blockIdx.x + localTopkDoc->padding; //threadIdx.x + localTopkDoc->padding;
	float ownScore=0.0, scoreNeighborAux=0.0, scoreInList=0.0;
	int threadIdInWarp =  (threadIdx.x & 0x1f);
	int maxIndex =  iTopk - localTopkDoc->padding;
	long long position;
	for (int localIndex = threadIdx.x; localIndex < iTopk; localIndex+=blockDim.x) {
		position = 0;
		if(localIndex < maxIndex){
			ownScore = localTopkDoc->score[localIndex];
		}
		else {
			ownScore = NO_MORE_DOC;
		}

		for (int indexInlist = threadIdInWarp; indexInlist < iTopk; indexInlist+=32) {
			if(indexInlist < maxIndex){
				scoreInList = localTopkDoc->score[indexInlist];
			}else{
				scoreInList= NO_MORE_DOC;
			}

			for (int threadId = 0; threadId < 32; ++threadId) {

				if(threadId >= (maxIndex - ((indexInlist >> 5) << 5)))
					break;

				scoreNeighborAux = __shfl_sync(0xFFFFFFFF,scoreInList, threadId);

				if(ownScore > scoreNeighborAux || (ownScore == scoreNeighborAux && (((indexInlist >> 5) << 5) + threadId) < localIndex )){// i > localIndex ) ){ //(i+t) < localIndex)){
					position++;
				}
			}
		}
		if(localIndex < maxIndex){
			dTopkScoreListGlobal[globalIndex + position] = ownScore;
			iTopkDocListGlobal[globalIndex + position] = localTopkDoc->id[localIndex];
		}

	}

	__syncthreads();
}

