#include "hip/hip_runtime.h"
/*
 * UnityTest.c
 *
 *  Created on: 04/08/2018
 *      Author: roussian
 */


#include <stdio.h>
#include <stdlib.h>
#include "UnityTest.cuh"

__device__ float checkMinHeapProperty(documentTopkList heap, float newScore, int docCurrent,
									  int topk){

	int index = (topk - 1) - heap.padding - threadIdx.x;
	int parent = index;
	float result=0.0; //0 - Min heap is correct  1 - Min heap is not correct

	for (; index > 0; index-=blockDim.x ) {
		do{
			parent = (parent-1)/2;
			if(heap.score[index] < heap.score[parent]){
				result = heap.score[index];
				printf("[HEAP] position - %d | parent's position - %d | newScore - %.2f | docCurrent - %d | padding -%d \n", index, parent, newScore, docCurrent,heap.padding);
				break;
			}
		}while(parent > 0);

		if(result != 0.0)
			break;
	}

	if(result == 0.0){
		index = threadIdx.x;
		int max = (topk - 1) - heap.padding;
		for (; index < max; index+=blockDim.x ) {
			for (int i=index+1; i < max; i++ ) {
				if(heap.id[index] == heap.id[i]){
					printf("[HEAP] Duplicated document in heap %d | score %.2f | initial position %d | duplicated position %d | blockId %d | docCurrent %d\n",
										 heap.id[index], heap.score[index], index, i, blockIdx.x, docCurrent);
					result = heap.id[index];
				}
			}
		}
	}

	return result;
}

__device__ float checkSorting(documentTopkList heap, float *dTopkScoreListGlobal, int *iTopkDocListGlobal, int topk){

	int globalIndex = blockIdx.x * topk + heap.padding;
	int result=0;
	int index,maxIndex;

	if(THREAD_MASTER){
		if(heap.id[0] != iTopkDocListGlobal[globalIndex]){
			printf("[SORTING] First doc is not correct in topk: global index %d | blockIdx.x %d | doc %d | global doc %d !\n",
					blockIdx.x * topk, blockIdx.x,heap.id[0],iTopkDocListGlobal[globalIndex]);
			result = 1;
			return result;
		}
	}

	int gindex = blockIdx.x * topk + threadIdx.x;
	maxIndex = blockIdx.x * topk + heap.padding;
	for (int i = gindex; i < maxIndex; i+=blockDim.x) {
		if(iTopkDocListGlobal[i] != -1 ){
			printf("[SORTING] Error in padding: blockId %d | index %d | document in position %d\n", blockIdx.x, i,iTopkDocListGlobal[i]);
			result = iTopkDocListGlobal[i];
			return result;
		}
	}

    gindex = blockIdx.x * topk + heap.padding + threadIdx.x;
	maxIndex = (blockIdx.x+1) * topk;
	for (int i = gindex; i < maxIndex; i+=blockDim.x) {
		if(iTopkDocListGlobal[i] == -1 ){
			printf("[SORTING] Docs is empty: blockId %d | index %d | document in position %d\n", blockIdx.x, i,iTopkDocListGlobal[i]);
			result = iTopkDocListGlobal[i];
			return result;
		}
	}

	maxIndex = topk * (blockIdx.x + 1);
	index = topk * blockIdx.x + heap.padding + threadIdx.x;
	for (; index < maxIndex; index+=blockDim.x ) {
		for (int i=index+1; i < maxIndex; i++ ) {
			if(iTopkDocListGlobal[index] == iTopkDocListGlobal[i]){
				printf("[SORTING] Duplicate document in global topk list %d - score %.2f \n", iTopkDocListGlobal[index], dTopkScoreListGlobal[index]);
				result = iTopkDocListGlobal[index];
				return result;
			}
		}
	}

	index=threadIdx.x;
	globalIndex = blockIdx.x * topk;
	maxIndex = topk * (blockIdx.x + 1);
	for (; index < topk-heap.padding; index += blockDim.x) {
		int count = 0;
		for (int i = globalIndex; i < maxIndex; ++i) {
			if(heap.id[index] == iTopkDocListGlobal[i]){
				count++;
				if(heap.score[index] != dTopkScoreListGlobal[i]){
					printf("[SORTING] document's score is wrong! doc %d | score %.2f | local index %d | global index %d\n",
							heap.id[index], heap.score[index], index, i);
					result = heap.id[index];
					return result;
				}
				break;
			}
		}

		if(count == 0){
			printf("[SORTING] Document disappeared: doc %d | score %.2f | local index %d\n",
					heap.id[index], heap.score[index], index);
			result = heap.id[index];
			return result;
		}
	}

	index = blockIdx.x * topk + heap.padding + threadIdx.x;
	maxIndex = topk * (blockIdx.x + 1);
	for (; index < maxIndex; index+=blockDim.x ) {
		for (int i=index+1; i < maxIndex; i++ ) {
			if(dTopkScoreListGlobal[index] > dTopkScoreListGlobal[i]){
				printf("[SORTING] BlockIdx %d | Documents are not sorting!!! doc %d (%.2f) is greater than doc %d (%.2f)\n",
						blockIdx.x,iTopkDocListGlobal[index],dTopkScoreListGlobal[index],iTopkDocListGlobal[i],dTopkScoreListGlobal[i]);
				result = iTopkDocListGlobal[index] ;
				return result;
			}
		}
	}

	return result;
}

__device__ float checkMerge_Sorting_Documents(documentTopkList sortingList,int iSkipMerges,
	  	  	  	  	  	  	  	  	  	  	  int iSkipBlocks, int topk){

	float result = 0.0;
	float score;

	for (int i = threadIdx.x; i < topk; i+=blockDim.x) {
		int doc = sortingList.id[i];

		if((doc != -1 && sortingList.score[i] == 0.0) || (doc == -1 && sortingList.score[i] != 0.0) )
			printf("[MERGE] BlockId %d | SkipBlocks %d | SkipMerges %d | Document is inconsistent: doc %d (%.2f - %d)",iSkipBlocks,
					blockIdx.x, iSkipMerges, doc, sortingList.score[i], i);

		if(doc != -1)
			for (int j = i+1; j < topk; ++j) {
				if(sortingList.id[j] != -1 && sortingList.id[j] == doc){
					printf("[MERGE] BlockId %d | SkipBlocks %d | SkipMerges %d | Duplicated Document: doc %d (%.2f - %d) - doc %d (%.2f - %d)\n",
							blockIdx.x, iSkipBlocks, iSkipMerges, sortingList.id[i], sortingList.score[i], i, sortingList.id[j], sortingList.score[j], j);
					result = sortingList.id[i];
					return result;
				}
			}
	}

	for (int i = threadIdx.x; i < topk; i+=blockDim.x) {
		score = sortingList.score[i];
		if(score != 0.0)
			for (int j = i+1; j < topk; ++j) {
				if(sortingList.score[j] != 0.0 && score > sortingList.score[j]){
					printf("[MERGE] BlockId %d | SkipBlocks %d | SkipMerges %d | Documents are not sorting!!! doc %d (%.2f - %d) is greater than doc %d (%.2f - %d)\n",
							blockIdx.x, iSkipBlocks, iSkipMerges, sortingList.id[i], sortingList.score[i], i, sortingList.id[j], sortingList.score[j], j);
					result = sortingList.id[i];
					return result;
				}
			}
	}

	return result;
}
