#include "hip/hip_runtime.h"
/*
 * HostManager.c
 *
 *  Created on: 07/12/2017
 *      Author: roussian
 */

#include "HostManager.cuh"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#include "../include/hip/hip_runtime_api.h"
#include "FileManager.cuh"
#include "ParallelPrunningDaat.cuh"
#include "Structs.cuh"

//#define DOC_QUANTITY_IN_MEMORY 64

void setThresholdForANDQuery(float *dInitialThreshold, float* h_dUBList, int iTermNumber){
	printf("Verify if the query is AND... ");
	for (int i = 0; i < iTermNumber; ++i) {
		*dInitialThreshold += h_dUBList[i];
	}

	*dInitialThreshold = *dInitialThreshold * 0.9 ;
	printf("OK\n");
}

void setThresholdForANDQueryOnInvertedList(float *dInitialThreshold, float* h_dUBList,
										   int *query, int iTermNumberByQuery){
	printf("Verify if the query is AND... ");

	for (int i = 0; i < iTermNumberByQuery; ++i) {
		*dInitialThreshold += h_dUBList[ query[i] ];
	}

	*dInitialThreshold = *dInitialThreshold * 0.70;

	printf("OK\n");

}

void initializeSingleExperimentQuery(int* iTermNumber, float* dAverageDocLength,  int* iTopK,
						  	  	  	 int*** h_iDocIdList, int*** h_iLenghtList,  unsigned short int*** h_iFreqList,
						  	  	  	 float** h_dIdfList, float** h_dUBList, int** h_iDocNumberList, int* iPostingTotalNumber,
						  	  	  	 int* iMaxNumberInList, int iExperimentNumber){


	readQuery(iTermNumber, dAverageDocLength, iTopK, h_iDocIdList, h_iLenghtList,
			  h_iFreqList, h_dIdfList, h_dUBList, h_iDocNumberList,iExperimentNumber);

	for (int i = 0; i < *iTermNumber; ++i) {

		*iPostingTotalNumber +=  (*h_iDocNumberList)[i];

		if(*iMaxNumberInList < (*h_iDocNumberList)[i]){
			*iMaxNumberInList = (*h_iDocNumberList)[i];
		}
	}

	printf("#Terms: %hu\n",*iTermNumber);
	printf("Max: %i\n",*iMaxNumberInList);
	printf("Total: %i\n",*iPostingTotalNumber);

}

__host__ void initializeInvertedIndex(int* iTermNumberInVocabulary, float* dAverageDocLength, int*** h_iDocIdList,
							          int*** h_iLenghtList,  unsigned short int*** h_iFreqList,
							          float** h_dIdfList, float** h_dUBList, int** h_iDocNumberList,
							          unsigned long long* docTotalNumber){

	#ifdef DEBUG
		printf("Initializing inverted list in host memory... ");
	#endif

	readInvertedList(iTermNumberInVocabulary, dAverageDocLength, h_iDocIdList, h_iLenghtList,h_iFreqList,
			  	  	 h_dIdfList, h_dUBList, h_iDocNumberList, docTotalNumber);

	#ifdef DEBUG
		printf("OK!\n");
	#endif
//	for (int i = 0; i < *iTermNumber; ++i) {
//
//		*iPostingTotalNumber +=  (*h_iDocNumberList)[i];
//
//		if(*iMaxNumberInList < (*h_iDocNumberList)[i]){
//			*iMaxNumberInList = (*h_iDocNumberList)[i];
//		}
//	}
	#ifdef DEBUG
		printf("#Terms in Inverted List: %hu\n",*iTermNumberInVocabulary);
	#endif
//	printf("Max: %i\n",*iMaxNumberInList);
//	printf("Total: %i\n",*iPostingTotalNumber);

}

__host__ void initializeQueryBatch(int ***h_iQueryBatches, int *h_iQueryNumber, int **h_iTermNumberList){

	#ifdef DEBUG
		printf("Get query batch to host memory... ");
	#endif

	readQueryBatch(h_iQueryBatches, h_iQueryNumber, h_iTermNumberList);

	#ifdef DEBUG
		printf("OK!\n");
	#endif
}

__host__ void memoryAllocationOfInvertedIndexInGPU(unsigned long long iDocTotalNumber, int iTermNumber,
												   int **d_iDocIdList, int **d_iDocLenghtList,
												   unsigned short int **d_iFreqDocList,
												   float **d_dUBlist, float **d_dIdfList,
												   int **d_iDocNumberByTermList){

	#ifdef DEBUG
		printf("Allocating Pointer to Pointer and coping yours values in/to Device Memory... ");
	#endif

	unsigned long long int nbytes_short = iDocTotalNumber * sizeof(unsigned short int);
	unsigned long long int nbytes = iDocTotalNumber * sizeof(int);

	checkCudaErrors(hipMalloc((void**) &(*d_iDocIdList), nbytes));
	checkCudaErrors(hipMalloc((void**) &(*d_iDocLenghtList), nbytes));
	checkCudaErrors(hipMalloc((void**) &(*d_iFreqDocList), nbytes_short));

	nbytes = iTermNumber * sizeof(float);
	checkCudaErrors(hipMalloc((void **) &(*d_dUBlist), nbytes));
	checkCudaErrors(hipMalloc((void **) &(*d_dIdfList), nbytes));

	nbytes = iTermNumber * sizeof(int);
	checkCudaErrors(hipMalloc((void **) &(*d_iDocNumberByTermList), nbytes));

//	nbytes = iTopk * sizeof(int) * topkListNumber;
//	checkCudaErrors(hipMalloc((void **) &(*d_iTopkDocList), nbytes));
//	checkCudaErrors(hipMemset(*d_iTopkDocList, -1, nbytes));

//	nbytes = iTopk * sizeof(float) * topkListNumber;
//	checkCudaErrors(hipMalloc((void **)&(*d_dTopkScoreList), nbytes));
//	checkCudaErrors(hipMemset(*d_dTopkScoreList, 0.0, nbytes));
	#ifdef DEBUG
		printf("OK!\n");
	#endif
}


__host__ void memoryAllocationOfSingleQueryInGPU(int iDocTotalNumber, int iTermNumber,
												 int topkListNumber, int iTopk,
												 int **d_iDocIdList, int **d_iDocLenghtList,
												 unsigned short int **d_iFreqDocList,
												 float **d_dUBlist, float **d_dIdfList,
												 int **d_iDocNumberByTermList, int **d_iTopkDocList,
												 float **d_dTopkScoreList){
	#ifdef DEBUG
		printf("Allocating Pointer to Pointer and coping yours values in/to Device Memory... ");
	#endif

	int nbytes_short = iDocTotalNumber * sizeof(unsigned short int);
	long long nbytes = iDocTotalNumber * sizeof(int);

	checkCudaErrors(hipMalloc((void**) &(*d_iDocIdList), nbytes));
	checkCudaErrors(hipMalloc((void**) &(*d_iDocLenghtList), nbytes));
	checkCudaErrors(hipMalloc((void**) &(*d_iFreqDocList), nbytes_short));

	nbytes = iTermNumber * sizeof(float);
	checkCudaErrors(hipMalloc((void **) &(*d_dUBlist), nbytes));
	checkCudaErrors(hipMalloc((void **) &(*d_dIdfList), nbytes));

	nbytes = iTermNumber * sizeof(int);
	checkCudaErrors(hipMalloc((void **) &(*d_iDocNumberByTermList), nbytes));

	nbytes = iTopk * sizeof(int) * topkListNumber;
	checkCudaErrors(hipMalloc((void **) &(*d_iTopkDocList), nbytes));
	checkCudaErrors(hipMemset(*d_iTopkDocList, -1, nbytes));

	nbytes = iTopk * sizeof(float) * topkListNumber;
	checkCudaErrors(hipMalloc((void **)&(*d_dTopkScoreList), nbytes));
	checkCudaErrors(hipMemset(*d_dTopkScoreList, 0.0, nbytes));

	#ifdef DEBUG
		printf("OK!\n");
	#endif
}

__host__ void freeAllocationOfInvertedListInGPU(int *d_iDocIdList, int *d_iDocLenghtList,
											   unsigned short int *d_iFreqDocList,
											   float *d_dUBlist, float *d_dIdfList,
											   int *d_iDocNumberByTermList){
	#ifdef DEBUG
		printf("Free Memory in GPU... ");
	#endif
	checkCudaErrors(hipFree(d_iDocIdList));
	checkCudaErrors(hipFree(d_iFreqDocList));
	checkCudaErrors(hipFree(d_iDocLenghtList));
	checkCudaErrors(hipFree(d_dUBlist));
	checkCudaErrors(hipFree(d_dIdfList));
	checkCudaErrors(hipFree(d_iDocNumberByTermList));
	#ifdef DEBUG
		printf("OK!\n");
	#endif
}


__host__ void freeAllocationOfSingleQueryInGPU(int *d_iDocIdList, int *d_iDocLenghtList,
											   unsigned short int *d_iFreqDocList,
											   float *d_dUBlist, float *d_dIdfList,
											   int *d_iDocNumberByTermList, int *d_iTopkDocList,
											   float *d_dTopkScoreList){
	#ifdef DEBUG
	   printf("Free Memory in GPU... ");
	#endif
	checkCudaErrors(hipFree(d_iDocIdList));
	checkCudaErrors(hipFree(d_iFreqDocList));
	checkCudaErrors(hipFree(d_iDocLenghtList));
	checkCudaErrors(hipFree(d_dUBlist));
	checkCudaErrors(hipFree(d_dIdfList));
	checkCudaErrors(hipFree(d_iDocNumberByTermList));
	checkCudaErrors(hipFree(d_iTopkDocList));
	checkCudaErrors(hipFree(d_dTopkScoreList));
	#ifdef DEBUG
		printf("OK!\n");
	#endif
}

__host__ void memoryCopyOfSingleQuery_To_Device(unsigned long long iDocTotalNumber, int iTermNumber,
											    int* h_iDocNumberByTermList,
											    int *d_iDocIdList, int *d_iDocLenghtList,
											    unsigned short int *d_iFreqDocList,float *d_dUBlist, float *d_dIdfList,
											    int *d_iDocNumberByTermList, int **h_iDocIdList,
											    int **h_iDocLenghtList, unsigned short int **h_iFreqList,
												float *h_dUBlist, float *h_dIdfList){
	#ifdef DEBUG
		printf("Memory Copy of Host Memory To Device Memory... ");
	#endif

	unsigned long long int nbytes_short;// = iDocTotalNumber * sizeof(unsigned short int);
	unsigned long long int nbytes;// = iDocTotalNumber * sizeof(int);
	int position = 0;

	for (int i = 0; i < iTermNumber; ++i) {
		nbytes = h_iDocNumberByTermList[i] * sizeof(int);
		checkCudaErrors(hipMemcpyAsync(&d_iDocIdList[position], h_iDocIdList[i], nbytes, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(&d_iDocLenghtList[position], h_iDocLenghtList[i], nbytes, hipMemcpyHostToDevice));

		nbytes_short = h_iDocNumberByTermList[i] *sizeof(unsigned short int);
		checkCudaErrors(hipMemcpyAsync(&d_iFreqDocList[position], h_iFreqList[i], nbytes_short, hipMemcpyHostToDevice));

		position += h_iDocNumberByTermList[i];
	}

	nbytes = iTermNumber * sizeof(float);
	checkCudaErrors(hipMemcpyAsync(d_dUBlist, h_dUBlist, nbytes, hipMemcpyHostToDevice, 0));
	checkCudaErrors(hipMemcpyAsync(d_dIdfList, h_dIdfList, nbytes, hipMemcpyHostToDevice, 0));

	nbytes = iTermNumber * sizeof(int);
	checkCudaErrors(hipMemcpyAsync(d_iDocNumberByTermList, h_iDocNumberByTermList, nbytes, hipMemcpyHostToDevice, 0));
//	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(iDocNumberByTermListConstant), h_iDocNumberByTermList, nbytes));
//	checkCudaErrors(hipDeviceSynchronize());

	#ifdef DEBUG
		printf("OK!\n");
	#endif
}

__host__ void callToBatchKernels_byBlock(dim3 blocksByGrid, dim3 threadsByBlock, int iMergeNumberByBlock, int iTopk,
								float dAverageDocumentLength, float dInitialThreshold,
								short int* iTermNumberInQuery,
								float *d_dUBList, float *d_dIdfList, int *d_iDocIdList,
								unsigned short int *d_iFreqDocList, int *d_iDocLenghtList,
								int *d_iTopkDocList, float *d_dTopkScoreList,
								int *iQueryTerms,long long* d_ptrPostingPositions,
								int* d_ptrQueryPositions, int *d_iDocNumberByTermList){
	#ifdef DEBUG
		printf("Launching Kernels by Block: WAND... ");
	#endif
//    int nbytes;
    //--------------------Creation CUDA Event Handles------------------------
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	float gpu_time = 0.0f;
    //-------------------------------------------------------------------------

   checkCudaErrors(hipEventRecord(start, 0));

//	matchWandParallel_VARIABLE_Batch_Block<<<blocksByGrid, threadsByBlock>>>(d_iDocIdList, d_iFreqDocList,
//	        				  	  	  	  	  	  	  	  	  	  	  	  d_dUBList, d_dIdfList, d_iDocLenghtList,
//	        				  	  	  	  	  	  	  	  	  	  	  	  iTermNumberInQuery, d_iTopkDocList, d_dTopkScoreList,
//	        				  	  	  	  	  	  	  	  	  	  	  	  dAverageDocumentLength,iBlockNumberRound,
//	        				  	  	  	  	  	  	  	  	  	  	  	  iGlobalNumberRound,iTopk,dInitialThreshold,
//	        				  	  	  	  	  	  	  	  	  	  	  	  iQueryTerms, d_ptrPostingPositions,
//	        				  	  	  	  	  	  	  	  	  	  	  	  d_ptrQueryPositions, d_iDocNumberByTermList);

   matchWandParallel_VARIABLE_Batch_Block_2<<<blocksByGrid, threadsByBlock>>>(d_iDocIdList, d_iFreqDocList,
																			   d_dUBList, d_dIdfList, d_iDocLenghtList,
																			   iTermNumberInQuery, d_iTopkDocList, d_dTopkScoreList,
																			   dAverageDocumentLength,
																			   iTopk,dInitialThreshold,
																			   iQueryTerms, d_ptrPostingPositions,
																			   d_ptrQueryPositions, d_iDocNumberByTermList);



	#ifdef DEBUG
		printf("Ok \n ");
	#endif

//
//    printf("OK!\n");

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));


	#ifdef DEBUG
		printf("OK!\n");
	#endif

	checkCudaErrors(hipDeviceSynchronize());
	printf("Coping results from GPU to CPU... ");
	int * h_iTopkDocList = (int*) malloc(iTopk * sizeof(int)*500);
	float * h_dTopkScoreList = (float*) malloc(iTopk * sizeof(float)*500);

	int nbytes = iTopk * sizeof(int)*500;
	checkCudaErrors(hipMemcpy(h_iTopkDocList,d_iTopkDocList, nbytes, hipMemcpyDeviceToHost));

	nbytes = iTopk * sizeof(float)*500;
	checkCudaErrors(hipMemcpy(h_dTopkScoreList,d_dTopkScoreList, nbytes, hipMemcpyDeviceToHost));
	checkCudaErrors(hipDeviceSynchronize());

	for (int j = 0; j < 500; ++j) {
		printf("\n--- %d Query -----\n",j);
		for (int i = 0; i < iTopk; ++i) {
			printf("--- %d %.2f----",h_iTopkDocList[j*iTopk + i],h_dTopkScoreList[j*iTopk + i]);
		}
	}

    printf("Time spent executing by the GPU: %.2f ms\n", gpu_time);
    printf("Batch %d, %d, %d, %.4f \n",TOP_K,SHAREDTHESHOLD,threadsByBlock.x, gpu_time);
}

__host__ void __inline__ callToBatchKernels(dim3 blocksByGrid, dim3 threadsByBlock, int iMergeNumberByBlock, int iTopk,
								float dAverageDocumentLength, float dInitialThreshold,
								int iBlockNumberRound, int iGlobalNumberRound, int iTermNumberInQuery,
								float *d_dUBList, float *d_dIdfList,
								int *d_iDocIdList, unsigned short int *d_iFreqDocList, int *d_iDocLenghtList,
								int *d_iTopkDocList, float *d_dTopkScoreList,
								int idQuery,
								hipStream_t stream, int *iQueryTerms, long long* d_ptrPostingPositions,
								int* d_ptrQueryPositions, int *d_iDocNumberByTermList){

	#ifdef DEBUG
		printf("Launching Kernels: WAND... ");
	    //--------------------Creation CUDA Event Handles------------------------
		hipEvent_t start, stop;
		checkCudaErrors(hipEventCreate(&start));
		checkCudaErrors(hipEventCreate(&stop));

		float gpu_time = 0.0f;
		    checkCudaErrors(hipEventRecord(start, stream));
	    //-------------------------------------------------------------------------

	#endif




	matchWandParallel_BATCH_2<<<blocksByGrid, threadsByBlock, 0, stream>>>(d_iDocIdList, d_iFreqDocList,
	        				  	  	  	  	  	  	  	  	  	  	  	  d_dUBList, d_dIdfList, d_iDocLenghtList,
	        				  	  	  	  	  	  	  	  	  	  	  	  iTermNumberInQuery, d_iTopkDocList, d_dTopkScoreList,
	        				  	  	  	  	  	  	  	  	  	  	  	  dAverageDocumentLength,iBlockNumberRound,
	        				  	  	  	  	  	  	  	  	  	  	  	  iGlobalNumberRound,iTopk,dInitialThreshold,
	        				  	  	  	  	  	  	  	  	  	  	  	  iQueryTerms, d_ptrPostingPositions,
	        				  	  	  	  	  	  	  	  	  	  	  	  d_ptrQueryPositions, idQuery, d_iDocNumberByTermList);


	int iTotalElementos = blocksByGrid.x * iTopk;
	int iProcessedListNumberbyBlock = iMergeNumberByBlock + 1;
	int iSkipTopKBetweenThreadBlocks = iProcessedListNumberbyBlock;
	int iSkipTopKBetweenMerges = 1;
	int exponent = 0;
	int exponent_b = 1;

	blocksByGrid.x = ceilf( blocksByGrid.x/(iProcessedListNumberbyBlock) );
//    checkCudaErrors(hipDeviceSynchronize());
//    evaluateAccuracyInGPU_Test(iTopk, copy_block , d_iTopkDocList,round);


	#ifdef DEBUG
		checkCudaErrors(hipDeviceSynchronize());
		printf("Ok\n");
		printf("Launching Kernels: Merge... ");
	#endif

    while(blocksByGrid.x >= 1){
//    	checkCudaErrors(hipDeviceSynchronize());
        mergeTopkLists_v3<<<blocksByGrid, iTopk, 0,stream>>>(d_dTopkScoreList, d_iTopkDocList,
      													 iTopk, iMergeNumberByBlock,
      													 iSkipTopKBetweenMerges,
      													 iSkipTopKBetweenThreadBlocks,iTotalElementos);

    	blocksByGrid.x = ceilf (blocksByGrid.x/(iProcessedListNumberbyBlock) );
    	exponent ++;
    	exponent_b ++;
    	iSkipTopKBetweenMerges = pow(iProcessedListNumberbyBlock, exponent);
    	iSkipTopKBetweenThreadBlocks = pow(iProcessedListNumberbyBlock, exponent_b);
    }


	#ifdef DEBUG
    	checkCudaErrors(hipDeviceSynchronize());
		printf("Ok \n ");
	#endif

//	printf("Coping results from GPU to CPU... ");
//
//	int nbytes = iTopk * sizeof(int);
//	int *h_iTopkDocList = (int*) malloc(sizeof(int) * iTopk);
//	checkCudaErrors(hipMemcpy(h_iTopkDocList, d_iTopkDocList, nbytes, hipMemcpyDeviceToHost));
//
//	float *h_dTopkScoreList = (float*) malloc(sizeof(float) * iTopk);
//	nbytes = iTopk * sizeof(float);
//    checkCudaErrors(hipMemcpy(h_dTopkScoreList, d_dTopkScoreList, nbytes, hipMemcpyDeviceToHost));
//
//    checkCudaErrors(hipDeviceSynchronize());
//
////    for (int j = 0; j < 500; ++j) {
//    	printf("\n--- %d Query -----\n",idQuery);
//		for (int i = 0; i < iTopk; ++i) {
//			printf("--- %d %.2f----",h_iTopkDocList[i],h_dTopkScoreList[i]);
//		}
////	}
//
//    free(h_iTopkDocList);
//    free(h_dTopkScoreList);

	#ifdef DEBUG
	    checkCudaErrors(hipEventRecord(stop));
	    checkCudaErrors(hipEventSynchronize(stop));
	    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
	    checkCudaErrors(hipEventDestroy(start));
	    checkCudaErrors(hipEventDestroy(stop));

		printf("Time spent executing by the GPU: %.2f ms\n", gpu_time);
	#endif
}

__host__ void callToKernels_Teste(dim3 blocksByGrid, dim3 threadsByBlock, int iMergeNumberByBlock, int iTopk,
							float dAverageDocumentLength, float dInitialThreshold,
							int iBlockNumberRound, int iGlobalNumberRound, int iTermNumber,
							float *d_dUBList, float *d_dIdfList,
							int *d_iDocIdList, unsigned short int *d_iFreqDocList, int *d_iDocLenghtList,
							int *d_iTopkDocList, float *d_dTopkScoreList,
							int *h_iTopkDocList, float *h_dTopkScoreList, int *d_iDocNumberByTermList,
							int* d_iInitialPositionPostingList, int* d_extraPositions, int* d_docMaxList,
							int docIdNumberByBlock){

		#ifdef DEBUG
			printf("Launching Kernels: WAND Teste... ");
		#endif
		int nbytes;
		//--------------------Creation CUDA Event Handles------------------------
		hipEvent_t start, stop;
		checkCudaErrors(hipEventCreate(&start));
		checkCudaErrors(hipEventCreate(&stop));

		float gpu_time = 0.0f;
		//-------------------------------------------------------------------------

		checkCudaErrors(hipEventRecord(start, 0));

		preProcessingWand<<<blocksByGrid, 1024, 0, 0>>>(d_iDocIdList, iTermNumber,
												      d_iDocNumberByTermList, d_iInitialPositionPostingList,
												      docIdNumberByBlock, d_extraPositions, d_docMaxList);

//		printf("Launching Kernels: WAND Teste... ");
//	    checkCudaErrors(hipDeviceSynchronize());
		matchWandParallel_VARIABLE_3_Teste<<<blocksByGrid, threadsByBlock, 0, 0>>>(d_iDocIdList, d_iFreqDocList,
																			  d_dUBList, d_dIdfList,
																			  d_iDocLenghtList,
																			  iTermNumber, d_iTopkDocList, d_dTopkScoreList,
																			  dAverageDocumentLength, //iBlockNumberRound,
																			  iGlobalNumberRound,iTopk,dInitialThreshold,d_iDocNumberByTermList,
																			  d_extraPositions, d_docMaxList);


		int iTotalElementos = blocksByGrid.x * iTopk;
		int iProcessedListNumberbyBlock = iMergeNumberByBlock + 1;
		int iSkipTopKBetweenThreadBlocks = iProcessedListNumberbyBlock;
		int iSkipTopKBetweenMerges = 1;
		int exponent = 0;
		int exponent_b = 1;

		blocksByGrid.x = ceilf( blocksByGrid.x/(iProcessedListNumberbyBlock) );
		//    checkCudaErrors(hipDeviceSynchronize());
		//    evaluateAccuracyInGPU_Test(iTopk, copy_block , d_iTopkDocList,round);

		#ifdef DEBUG
			checkCudaErrors(hipDeviceSynchronize());
			printf("Ok\n");
			printf("Launching Kernels: Merge... ");
		#endif

		while(blocksByGrid.x >= 1){
		//    	checkCudaErrors(hipDeviceSynchronize());
			mergeTopkLists_v3<<<blocksByGrid, iTopk, 0,0>>>(d_dTopkScoreList, d_iTopkDocList,
															 iTopk, iMergeNumberByBlock,
															 iSkipTopKBetweenMerges,
															 iSkipTopKBetweenThreadBlocks,iTotalElementos);

			blocksByGrid.x = ceilf (blocksByGrid.x/(iProcessedListNumberbyBlock) );
			exponent ++;
			exponent_b ++;
			iSkipTopKBetweenMerges = pow(iProcessedListNumberbyBlock, exponent);
			iSkipTopKBetweenThreadBlocks = pow(iProcessedListNumberbyBlock, exponent_b);
		}

		#ifdef DEBUG
			checkCudaErrors(hipDeviceSynchronize());
		#endif

		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
		checkCudaErrors(hipDeviceSynchronize());


		#ifdef DEBUG
			printf("Ok \n ");
			printf("Coping results from GPU to CPU... ");
		#endif

		nbytes = iTopk * sizeof(int);
		checkCudaErrors(hipMemcpy(h_iTopkDocList, d_iTopkDocList, nbytes, hipMemcpyDeviceToHost));

		nbytes = iTopk * sizeof(float);
		checkCudaErrors(hipMemcpy(h_dTopkScoreList, d_dTopkScoreList, nbytes, hipMemcpyDeviceToHost));
		//    checkCudaErrors(hipDeviceSynchronize());

		#ifdef DEBUG
			printf("OK!\n");
		#endif

		checkCudaErrors(hipEventDestroy(start));
		checkCudaErrors(hipEventDestroy(stop));

		printf("Result %d, %d, %d, %d, %d, %.4f \n",TOP_K,SHAREDTHESHOLD,DOC_QUANTITY_IN_MEMORY,threadsByBlock.x, iGlobalNumberRound, gpu_time);

		//	#ifdef DEBUG
		printf("Time spent executing by the GPU: %.4f ms\n", gpu_time);
		//	#endif


}


__host__ void callToKernels(dim3 blocksByGrid, dim3 threadsByBlock, int iMergeNumberByBlock, int iTopk,
							float dAverageDocumentLength, float dInitialThreshold,
							int iBlockNumberRound, int iGlobalNumberRound, int iTermNumber,
							float *d_dUBList, float *d_dIdfList,
							int *d_iDocIdList, unsigned short int *d_iFreqDocList, int *d_iDocLenghtList,
							int *d_iTopkDocList, float *d_dTopkScoreList,
							int *h_iTopkDocList, float *h_dTopkScoreList, int *d_iDocNumberByTermList){
	#ifdef DEBUG
		printf("Launching Kernels: WAND... ");
	#endif
    int nbytes;
    //--------------------Creation CUDA Event Handles------------------------
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	float gpu_time = 0.0f;
    //-------------------------------------------------------------------------

	int iTotalElementos = blocksByGrid.x * iTopk;
	int iProcessedListNumberbyBlock = iMergeNumberByBlock;// + 1;
	iMergeNumberByBlock = (1<<iMergeNumberByBlock)-1;
//	iProcessedListNumberbyBlock = iProcessedListNumberbyBlock / 2;
	int iSkipTopKBetweenThreadBlocks = 1 << iProcessedListNumberbyBlock;
	int iSkipTopKBetweenMerges = 1;
//	int exponent = 0;
//	int exponent_b = 1;


    checkCudaErrors(hipEventRecord(start, 0));

    matchWandParallel_VARIABLE_4_2<<<blocksByGrid, threadsByBlock, 0, 0>>>(d_iDocIdList, d_iFreqDocList,
	        				  	  	  	  	  	  	  	  	  	  	  	  d_dUBList, d_dIdfList,
	        				  	  	  	  	  	  	  	  	  	  	  	  d_iDocLenghtList,
	        				  	  	  	  	  	  	  	  	  	  	  	  iTermNumber, d_iTopkDocList, d_dTopkScoreList,
	        				  	  	  	  	  	  	  	  	  	  	  	  dAverageDocumentLength, //iBlockNumberRound,
	        				  	  	  	  	  	  	  	  	  	  	  	  iGlobalNumberRound,iTopk,dInitialThreshold,d_iDocNumberByTermList);


//	iTotalElementos = blocksByGrid.x * iTopk;
//	iProcessedListNumberbyBlock = iMergeNumberByBlock + 1;
//	iSkipTopKBetweenThreadBlocks = iProcessedListNumberbyBlock;
//	iSkipTopKBetweenMerges = 1;
//	exponent = 0;
//	exponent_b = 1;

    blocksByGrid.x = (((blocksByGrid.x & 1) == 1) && (blocksByGrid.x != 1)) ? blocksByGrid.x+1 :  blocksByGrid.x;
	blocksByGrid.x = blocksByGrid.x >> iProcessedListNumberbyBlock; //ceilf( blocksByGrid.x/(iProcessedListNumberbyBlock) );

	#ifdef DEBUG
		checkCudaErrors(hipDeviceSynchronize());
		printf("Ok\n");
		printf("Launching Kernels: Merge... ");
	#endif

    while(blocksByGrid.x >= 1){
//    	checkCudaErrors(hipDeviceSynchronize());
//    	printf("========>Blocks %d iSkipMerges %d iSkipeBlocks %d \n", blocksByGrid.x,iSkipTopKBetweenMerges,iSkipTopKBetweenThreadBlocks);
        mergeTopkLists_v3<<<blocksByGrid, iTopk, 0,0>>>(d_dTopkScoreList, d_iTopkDocList,
      													 iTopk, iMergeNumberByBlock,
      													 iSkipTopKBetweenMerges,
      													 iSkipTopKBetweenThreadBlocks,iTotalElementos);
//        if(blocksByGrid.x == 1) blocksByGrid.x = 0;
        blocksByGrid.x = (((blocksByGrid.x & 1) == 1) && (blocksByGrid.x != 1)) ? blocksByGrid.x+1 :  blocksByGrid.x;
        blocksByGrid.x =  blocksByGrid.x >> iProcessedListNumberbyBlock;//ceilf (blocksByGrid.x/(iProcessedListNumberbyBlock) );
//    	exponent ++;
//    	exponent_b ++;
    	iSkipTopKBetweenMerges = iSkipTopKBetweenMerges << 1;//pow(iProcessedListNumberbyBlock, exponent);
    	iSkipTopKBetweenThreadBlocks = iSkipTopKBetweenThreadBlocks << 1;//pow(iProcessedListNumberbyBlock, exponent_b);
//    	checkCudaErrors(hipDeviceSynchronize());
    }

	#ifdef DEBUG
    	checkCudaErrors(hipDeviceSynchronize());
	#endif

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    checkCudaErrors(hipDeviceSynchronize());
	printf("Time spent executing by the GPU: %.4f ms\n", gpu_time);

    #ifdef DEBUG
		printf("Ok \n ");
		printf("Coping results from GPU to CPU... ");
	#endif

	nbytes = iTopk * sizeof(int);
	checkCudaErrors(hipMemcpy(h_iTopkDocList, d_iTopkDocList, nbytes, hipMemcpyDeviceToHost));

	nbytes = iTopk * sizeof(float);
    checkCudaErrors(hipMemcpy(h_dTopkScoreList, d_dTopkScoreList, nbytes, hipMemcpyDeviceToHost));
//    checkCudaErrors(hipDeviceSynchronize());

    #ifdef DEBUG
		printf("OK!\n");
	#endif

	for (int i = 0; i < iTopk; ++i) {
		printf("--- %d ",h_iTopkDocList[i]);
	}

	for (int i = 0; i < iTopk; ++i) {
		printf("--- %.2f ",h_dTopkScoreList[i]);
	}
	printf("\n");
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    printf("Result %d, %d, %d, %d, %d, %.4f \n",TOP_K,SHAREDTHESHOLD,DOC_QUANTITY_IN_MEMORY,threadsByBlock.x, iGlobalNumberRound, gpu_time);

}

__host__ void querySingleProcessingHost_Teste(int iTopk, int iBlockSize, int iBlockNumberRound,
											  int iGlobalNumberRound, int iMergeNumberByBlock,
											  int iQueryType, int iExperimentNumber){

	hipSetDevice(0);
	//	printf("TOPK: %d\n", TOP_K);
	//	printf("DOC_QUANTITY_IN_MEMORY: %d\n", DOC_QUANTITY_IN_MEMORY);

//const int* iDocIdList,
//const short int iTermNumber,
//const int* d_iDocNumberByTermList,
//const int* iInitialPositionPostingList,
//const int docIdNumberByBlock,
//int* extraPositions, int* docMaxList

		int iPostingTotalNumber = 0;
		int iMaxNumberInList = 0;
		int iTermNumber;
		int docTotalNumber = 0;

		//----------Host Variables-------------
		int** h_iDocIdList;
		int** h_iDocLenghtList;
		unsigned short int** h_iFreqList;

		float* h_dUBList;
		float* h_dIdfList;

		int* h_iDocNumberByTermList;
		int* h_iInitialPositionPostingList;

		int* h_iTopkDocList = (int*) malloc(iTopk * sizeof(int));
		float* h_dTopkScoreList = (float*) malloc(iTopk * sizeof(float));

		float dAverageDocumentLength;
		//-------------------------------------

		//----------Device Variables-------------
		float dInitialThreshold = 0;
		float* d_dUBList;
		float* d_dIdfList;

		int* d_iDocIdList;
		int* d_iDocLenghtList;
		unsigned short int* d_iFreqDocList;

		int* d_iDocNumberByTermList;

		int* d_iTopkDocList;
		float* d_dTopkScoreList;

		int* d_iInitialPositionPostingList;
		int* d_extraPositions;
		int* d_docMaxList;
		int docIdNumberByBlock;
		//-------------------------------------
		#ifdef DEBUG
			printf("Initializing variables in host memory...\n");
		#endif
		initializeSingleExperimentQuery(&iTermNumber, &dAverageDocumentLength, &iTopk, &h_iDocIdList, &h_iDocLenghtList,
								 &h_iFreqList, &h_dIdfList, &h_dUBList, &h_iDocNumberByTermList,
								 &iPostingTotalNumber, &iMaxNumberInList,iExperimentNumber);
		//-------------------------------------

		//------------Kernel Launch Configuration---------------------------------
		printf("Kernel Launch Configuration...\n");
		int iBlockNumber= (int) ceil((float)iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * iGlobalNumberRound));
	//	int iBlockNumber= (int) ceil((float)iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * iBlockNumberRound * iGlobalNumberRound));
		dim3 threadsByBlock = dim3(iBlockSize, 1);
		dim3 blocksByGrid   = dim3(iBlockNumber, 1);
		printf("#threads by block: %i, #blocks: %i ...\n",threadsByBlock.x, blocksByGrid.x) ;
		//-------------------------------------------------------------------------

		h_iInitialPositionPostingList = (int*) malloc(sizeof(int*) * iTermNumber);
		int positionInList = 0;
		for (int idTerm = 0; idTerm < iTermNumber; ++idTerm) {
			h_iInitialPositionPostingList[idTerm] = positionInList;
			positionInList += h_iDocNumberByTermList[idTerm];
		}
		int nb = sizeof(int) * iTermNumber;
		checkCudaErrors(hipMalloc((void**) &d_iInitialPositionPostingList, nb));
		checkCudaErrors(hipMemcpyAsync(d_iInitialPositionPostingList, h_iInitialPositionPostingList, nb, hipMemcpyHostToDevice, 0));

		nb = sizeof(int) * iBlockNumber * iTermNumber;
		checkCudaErrors(hipMalloc((void**) &d_extraPositions, nb));
		nb = sizeof(int) * iBlockNumber;
		checkCudaErrors(hipMalloc((void**) &d_docMaxList, nb));

		docIdNumberByBlock = DOC_QUANTITY_IN_MEMORY*iGlobalNumberRound;
		//------------------------------------

	//	float** h_dUBLocal;
	//	h_dUBLocal = (float**) malloc( sizeof(float*)*iTermNumber);
	//
	//	float score = 0.0;
	//	float maxScore = 0.0;
	//	float k_1 = 1.2;
	//	float b = 0.75;
	//	for (int term = 0; term < iTermNumber; ++term) {
	//		maxScore = 0.0;
	//		h_dUBLocal[term] = (float*) malloc(iBlockNumber * sizeof(float));
	//
	//		for (int partition = 0; partition < iBlockNumber; ++partition) {
	//
	//			int position= partition * DOC_QUANTITY_IN_MEMORY * iGlobalNumberRound;
	//			int count = 0;
	//			while((count < DOC_QUANTITY_IN_MEMORY * iGlobalNumberRound) && (position < h_iDocNumberByTermList[term])) {
	//				score = (k_1 * h_iFreqList[term][position]) / ( h_iFreqList[term][position] + (k_1 * ((1 - b) + (b * h_iDocLenghtList[term][position]) / dAverageDocumentLength)));
	//				if(score > maxScore){
	//					maxScore = score;
	//				}
	//				count++;
	//				position++;
	//			}
	//
	//			h_dUBLocal[term][partition] = 1.1 * maxScore * h_dIdfList[term];
	//		}
	//	}
	//
	//	float* d_dUBLocal;
	//	int nbytes = sizeof(float) * iTermNumber * iBlockNumber;
	//	checkCudaErrors(hipMalloc((void**) &(d_dUBLocal), nbytes));
	//
	//	int position = 0;
	//	for (int iTerm = 0; iTerm < iTermNumber; ++iTerm) {
	//		nbytes = iBlockNumber * sizeof(float);
	//		checkCudaErrors(hipMemcpyAsync(&d_dUBLocal[position], h_dUBLocal[iTerm], nbytes, hipMemcpyHostToDevice));
	//		position += iBlockNumber;
	//	}


		//-----------------------------------



		//-----------Initializing the initial value of Threshold if it is necessery ---------------------
		if(iQueryType){
			setThresholdForANDQuery(&dInitialThreshold, h_dUBList, iTermNumber);
		}
		//-----------------------------------------------------------------------------------------------

		for (int i = 0; i < iTermNumber; ++i) {
			docTotalNumber += h_iDocNumberByTermList[i];
		}

		//------------Allocating and Coping Pointer to Pointer in Device Memory--------------------------------------
		memoryAllocationOfSingleQueryInGPU(docTotalNumber, iTermNumber, iBlockNumber, iTopk,
										   &d_iDocIdList, &d_iDocLenghtList, &d_iFreqDocList,
										   &d_dUBList, &d_dIdfList, &d_iDocNumberByTermList, &d_iTopkDocList,
										   &d_dTopkScoreList);

		memoryCopyOfSingleQuery_To_Device(docTotalNumber, iTermNumber,h_iDocNumberByTermList,
			    						  d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
			    						  d_dUBList, d_dIdfList, d_iDocNumberByTermList, h_iDocIdList,
			    						  h_iDocLenghtList, h_iFreqList, h_dUBList, h_dIdfList);
		//----------------------------------------------------------------------------------------------------------

		//--------------------------Call to Kernels----------------------------
		callToKernels_Teste(blocksByGrid, threadsByBlock, iMergeNumberByBlock, iTopk,
					  dAverageDocumentLength, dInitialThreshold, iBlockNumberRound,
					  iGlobalNumberRound, iTermNumber,
					  d_dUBList, d_dIdfList,d_iDocIdList, d_iFreqDocList, d_iDocLenghtList,
					  d_iTopkDocList, d_dTopkScoreList, h_iTopkDocList, h_dTopkScoreList,d_iDocNumberByTermList,
					  d_iInitialPositionPostingList, d_extraPositions, d_docMaxList,
					  docIdNumberByBlock);

		//----------------------------------------------------------------------------------------------------------

		//-----------------------Free GPU Memory------------------------------

		freeAllocationOfSingleQueryInGPU(d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
										 d_dUBList, d_dIdfList, d_iDocNumberByTermList, d_iTopkDocList,
										 d_dTopkScoreList);
		//----------------------------------------------------------------------------------------------------------

		checkCudaErrors(hipFree(d_extraPositions));
		checkCudaErrors(hipFree(d_docMaxList));
		//------------------Release resources in CPU-------------------------------
		printf("Releasing resources in CPU... ");

		for (int i = 0; i < iTermNumber; ++i) {
			free(h_iDocIdList[i]);
			free(h_iDocLenghtList[i]);
			free(h_iFreqList[i]);
	//		free(h_dUBLocal[i]);
		}
		free(h_iInitialPositionPostingList);
		free(h_iDocIdList);
		free(h_iDocLenghtList);
		free(h_iFreqList);
		free(h_dIdfList);
		free(h_dUBList);
		free(h_iDocNumberByTermList);
		free(h_iTopkDocList);
		free(h_dTopkScoreList);
		printf("OK!\n");
		//-------------------------------------------------------------------------
		printf("Finish!\n");


}

__host__ void querySingleProcessingHost(int iTopk, int iBlockSize, int iBlockNumberRound,
										int iGlobalNumberRound, int iMergeNumberByBlock,
										int iQueryType, int iExperimentNumber){
//	printf("TOPK: %d\n", TOP_K);
//	printf("DOC_QUANTITY_IN_MEMORY: %d\n", DOC_QUANTITY_IN_MEMORY);

	hipSetDevice(0);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	printf("Device Name: %s\n",prop.name);

	int iPostingTotalNumber = 0;
	int iMaxNumberInList = 0;
	int iTermNumber;
	int docTotalNumber = 0;

	//----------Host Variables-------------
	int** h_iDocIdList;
	int** h_iDocLenghtList;
	unsigned short int** h_iFreqList;

	float* h_dUBList;
	float* h_dIdfList;

	int* h_iDocNumberByTermList;

	int* h_iTopkDocList = (int*) malloc(iTopk * sizeof(int));
	float* h_dTopkScoreList = (float*) malloc(iTopk * sizeof(float));

	float dAverageDocumentLength;
	//-------------------------------------

	//----------Device Variables-------------
	float dInitialThreshold = 0;
	float* d_dUBList;
	float* d_dIdfList;

	int* d_iDocIdList;
	int* d_iDocLenghtList;
	unsigned short int* d_iFreqDocList;

	int* d_iDocNumberByTermList;
	int* d_iTopkDocList;
	float* d_dTopkScoreList;
	//-------------------------------------
	#ifdef DEBUG
		printf("Initializing variables in host memory...\n");
	#endif
	initializeSingleExperimentQuery(&iTermNumber, &dAverageDocumentLength, &iTopk, &h_iDocIdList, &h_iDocLenghtList,
							 &h_iFreqList, &h_dIdfList, &h_dUBList, &h_iDocNumberByTermList,
							 &iPostingTotalNumber, &iMaxNumberInList,iExperimentNumber);
	//-------------------------------------

	//------------Kernel Launch Configuration---------------------------------
	printf("Kernel Launch Configuration...\n");
	int iBlockNumber= (int) ceil((float)iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * iGlobalNumberRound));
//	int iBlockNumber= (int) ceil((float)iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * iBlockNumberRound * iGlobalNumberRound));
	dim3 threadsByBlock = dim3(iBlockSize, 1);
	dim3 blocksByGrid   = dim3(iBlockNumber, 1);
	printf("#threads by block: %i, #blocks: %i ...\n",threadsByBlock.x, blocksByGrid.x) ;
	//-------------------------------------------------------------------------

	//------------------------------------

//	float** h_dUBLocal;
//	h_dUBLocal = (float**) malloc( sizeof(float*)*iTermNumber);
//
//	float score = 0.0;
//	float maxScore = 0.0;
//	float k_1 = 1.2;
//	float b = 0.75;
//	for (int term = 0; term < iTermNumber; ++term) {
//		maxScore = 0.0;
//		h_dUBLocal[term] = (float*) malloc(iBlockNumber * sizeof(float));
//
//		for (int partition = 0; partition < iBlockNumber; ++partition) {
//
//			int position= partition * DOC_QUANTITY_IN_MEMORY * iGlobalNumberRound;
//			int count = 0;
//			while((count < DOC_QUANTITY_IN_MEMORY * iGlobalNumberRound) && (position < h_iDocNumberByTermList[term])) {
//				score = (k_1 * h_iFreqList[term][position]) / ( h_iFreqList[term][position] + (k_1 * ((1 - b) + (b * h_iDocLenghtList[term][position]) / dAverageDocumentLength)));
//				if(score > maxScore){
//					maxScore = score;
//				}
//				count++;
//				position++;
//			}
//
//			h_dUBLocal[term][partition] = 1.1 * maxScore * h_dIdfList[term];
//		}
//	}
//
//	float* d_dUBLocal;
//	int nbytes = sizeof(float) * iTermNumber * iBlockNumber;
//	checkCudaErrors(hipMalloc((void**) &(d_dUBLocal), nbytes));
//
//	int position = 0;
//	for (int iTerm = 0; iTerm < iTermNumber; ++iTerm) {
//		nbytes = iBlockNumber * sizeof(float);
//		checkCudaErrors(hipMemcpyAsync(&d_dUBLocal[position], h_dUBLocal[iTerm], nbytes, hipMemcpyHostToDevice));
//		position += iBlockNumber;
//	}
//

	//-----------------------------------



	//-----------Initializing the initial value of Threshold if it is necessery ---------------------
	if(iQueryType){
		setThresholdForANDQuery(&dInitialThreshold, h_dUBList, iTermNumber);
	}
	//-----------------------------------------------------------------------------------------------

	for (int i = 0; i < iTermNumber; ++i) {
		docTotalNumber += h_iDocNumberByTermList[i];
	}

	//------------Allocating and Coping Pointer to Pointer in Device Memory--------------------------------------
	memoryAllocationOfSingleQueryInGPU(docTotalNumber, iTermNumber, iBlockNumber, iTopk,
									   &d_iDocIdList, &d_iDocLenghtList, &d_iFreqDocList,
									   &d_dUBList, &d_dIdfList, &d_iDocNumberByTermList, &d_iTopkDocList,
									   &d_dTopkScoreList);

	memoryCopyOfSingleQuery_To_Device(docTotalNumber, iTermNumber,h_iDocNumberByTermList,
		    						  d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
		    						  d_dUBList, d_dIdfList, d_iDocNumberByTermList, h_iDocIdList,
		    						  h_iDocLenghtList, h_iFreqList, h_dUBList, h_dIdfList);
	//----------------------------------------------------------------------------------------------------------

	//--------------------------Call to Kernels----------------------------
	callToKernels(blocksByGrid, threadsByBlock, iMergeNumberByBlock, iTopk,
				  dAverageDocumentLength, dInitialThreshold, iBlockNumberRound,
				  iGlobalNumberRound, iTermNumber,
				  d_dUBList, d_dIdfList,d_iDocIdList, d_iFreqDocList, d_iDocLenghtList,
				  d_iTopkDocList, d_dTopkScoreList, h_iTopkDocList, h_dTopkScoreList,d_iDocNumberByTermList);

	//----------------------------------------------------------------------------------------------------------

	//-----------------------Free GPU Memory------------------------------

	freeAllocationOfSingleQueryInGPU(d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
									 d_dUBList, d_dIdfList, d_iDocNumberByTermList, d_iTopkDocList,
									 d_dTopkScoreList);
	//----------------------------------------------------------------------------------------------------------

//	checkCudaErrors(hipFree(d_dUBLocal));
	//------------------Release resources in CPU-------------------------------
	printf("Releasing resources in CPU... ");

	for (int i = 0; i < iTermNumber; ++i) {
		free(h_iDocIdList[i]);
		free(h_iDocLenghtList[i]);
		free(h_iFreqList[i]);
//		free(h_dUBLocal[i]);
	}
//	free(h_dUBLocal);
	free(h_iDocIdList);
	free(h_iDocLenghtList);
	free(h_iFreqList);
	free(h_dIdfList);
	free(h_dUBList);
	free(h_iDocNumberByTermList);
	free(h_iTopkDocList);
	free(h_dTopkScoreList);
	printf("OK!\n");
	//-------------------------------------------------------------------------
	printf("Finish!\n");
}

__host__ void queryBatchProcessingHost_Mix(int iTopk, int iBlockSize, int iBlockNumberRound,
										int iGlobalNumberRound, int iMergeNumberByBlock,
									int iQueryType){



//	int iPostingTotalNumber = 0;
//	int iMaxNumberInList = 0;
	int iTermNumberInVocabulary;
	unsigned long long docTotalNumber = 0;

	//----------Host Variables-------------
	int **h_iDocIdList, **h_iDocLenghtList;
	unsigned short int** h_iFreqList;

	float *h_dUBList, *h_dIdfList;

	int* h_iDocNumberByTermList;

	int* h_iTopkDocList = (int*) malloc(iTopk * sizeof(int));
	float* h_dTopkScoreList = (float*) malloc(iTopk * sizeof(float));

	float dAverageDocumentLength;

	int *h_iTermNumberByQuery; //Número de termo por query do batch de query
	int **h_iQueryBatches; //Batch de Queries
	int iQueryTotalNumberByBatch = 500; //Número de query por batch
	//-------------------------------------

	//---------Mapped Variables in Host and Device------------------
	int *m_iTermNumberByQuery; //Número de termo por query do batch de query - Os indices representam as queries
	int *m_iQueryBatches; //Batch de Queries - Um conjunto de termos - Os termos pertence a um conjunto de queries
	int *m_ptrQueryPositions; //A posição das queries na lista de batch de queries; essa contagem é a realizada pela contagem dos termos

	long long* m_ptrInitPostingList; //Considera que há uma única lista no acelerador  - Posição inicial das listas de postings dos termos

	int** m_iTopkDocList; // Topk
	float** m_dTopkScoreList; // Topk
	//---------------------------------------------------------------

	//----------Device Variables------------------------------------


	float dInitialThreshold = 0, *d_dUBList, *d_dIdfList;

	int *d_iDocIdList, *d_iDocLenghtList;
	unsigned short int* d_iFreqDocList;

	int* d_iDocNumberByTermList;
	int* d_iTopkDocList;
	float* d_dTopkScoreList;

	int *d_iTermNumberByQuery, *d_iQueryBatches, *d_ptrQueryPositions;
	long long *d_ptrInitPostingList;

	//-------------------------------------

	initializeInvertedIndex(&iTermNumberInVocabulary, &dAverageDocumentLength, &h_iDocIdList,
							&h_iDocLenghtList, &h_iFreqList, &h_dIdfList, &h_dUBList,
							&h_iDocNumberByTermList, &docTotalNumber);

	//-------------------GPU Pre-Configuration---------------------
	hipSetDevice(0);
	hipSetDeviceFlags(hipDeviceMapHost);

	//-------------------------------------------------------------

	//------------Allocating and Coping Pointer to Pointer in Device Memory--------------------------------------
	memoryAllocationOfInvertedIndexInGPU(docTotalNumber, iTermNumberInVocabulary,&d_iDocIdList,
										 &d_iDocLenghtList, &d_iFreqDocList,&d_dUBList,
										 &d_dIdfList, &d_iDocNumberByTermList);

	memoryCopyOfSingleQuery_To_Device(docTotalNumber, iTermNumberInVocabulary,h_iDocNumberByTermList,
									  d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
									  d_dUBList, d_dIdfList, d_iDocNumberByTermList, h_iDocIdList,
									  h_iDocLenghtList, h_iFreqList, h_dUBList, h_dIdfList);
	//----------------------------------------------------------------------------------------------------------


	initializeQueryBatch(&h_iQueryBatches, &iQueryTotalNumberByBatch, &h_iTermNumberByQuery);
//	iQueryTotalNumberByBatch=500;
	//Obtém a soma do número de termos de todas as queries (Esse valor pode ser pre-computado)
	int iTermTotalNumberOfBatch = 0;
	for (int i = 0; i < iQueryTotalNumberByBatch; ++i) {
		iTermTotalNumberOfBatch += h_iTermNumberByQuery[i];
	}
//	checkCudaErrors(hipMalloc((void**)&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_ptrQueryPositions, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_ptrInitPostingList, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch));
//
	checkCudaErrors(hipHostAlloc(&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch, hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_ptrQueryPositions, sizeof(int)*iQueryTotalNumberByBatch, hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_ptrInitPostingList, sizeof(long long)*iTermNumberInVocabulary, hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_iQueryBatches, sizeof(int)*iTermTotalNumberOfBatch, hipHostMallocWriteCombined | hipHostMallocMapped));

	long long position = 0;
	for (int i = 0; i < iQueryTotalNumberByBatch; i++) {
		m_ptrQueryPositions[i] = position;
		m_iTermNumberByQuery[i] = h_iTermNumberByQuery[i];
		for (int term = 0; term < h_iTermNumberByQuery[i]; term++) {
			m_iQueryBatches[position] = h_iQueryBatches[i][term];//Alinha as queries para o mapeamento com a GPU
			position++;
		}
	}

	position = 0;
	for (int i = 0; i < iTermNumberInVocabulary; ++i) {//Obtém a posição inicial de todas as listas invertidas
		m_ptrInitPostingList[i] = position;
		position += h_iDocNumberByTermList[i];
	}

	checkCudaErrors(hipHostGetDevicePointer(&d_iTermNumberByQuery, m_iTermNumberByQuery, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_ptrQueryPositions, m_ptrQueryPositions, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_iQueryBatches,m_iQueryBatches, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_ptrInitPostingList, m_ptrInitPostingList, 0 ) );

	//-------------------------------------

	hipStream_t *streams = (hipStream_t*) malloc(sizeof(hipStream_t) * iQueryTotalNumberByBatch);

	int nbytes = sizeof(int**) * iQueryTotalNumberByBatch;
	checkCudaErrors(hipHostAlloc((void **)&(m_iTopkDocList), nbytes, hipHostMallocWriteCombined | hipHostMallocMapped));

	nbytes = sizeof(float**) * iQueryTotalNumberByBatch;
	checkCudaErrors(hipHostAlloc((void **)&(m_dTopkScoreList), nbytes, hipHostMallocWriteCombined | hipHostMallocMapped));

	int *queryListOrder = (int*) malloc(sizeof(int)*iQueryTotalNumberByBatch);
	int *sizePostingList = (int*) malloc(sizeof(int)*iQueryTotalNumberByBatch);

	int iMaxNumberInList;
	int* d_temp_iTopkDocList;
	float* d_temp_dTopkScoreList;
	for (int idQuery = 0; idQuery < iQueryTotalNumberByBatch; ++idQuery) {
		iMaxNumberInList = 0;
		for (int idTerm = 0; idTerm < h_iTermNumberByQuery[idQuery]; idTerm++) {
			if(iMaxNumberInList < h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm] ] )
				iMaxNumberInList = h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm] ];
		}

		sizePostingList[idQuery] = iMaxNumberInList;
		queryListOrder[idQuery] =  idQuery;

		int auxq=0, auxtamanho=0;

		int i = idQuery;
		while(i > 0){
			if(sizePostingList[i-1] < sizePostingList[i]){
				auxtamanho = sizePostingList[i-1];
				auxq = queryListOrder[i-1];

				sizePostingList[i-1] = sizePostingList[i];
				queryListOrder[i-1] = queryListOrder[i];

				sizePostingList[i] = auxtamanho;
				queryListOrder[i] = auxq;
			}
			else
				break;
			i--;
		}

		nbytes = iTopk * sizeof(int) * ((int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound)));
		checkCudaErrors(hipMalloc((void**)&d_temp_iTopkDocList, nbytes)  );
		checkCudaErrors(hipMemset(d_temp_iTopkDocList, -1, nbytes));

		nbytes = iTopk * sizeof(float)  * ((int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound)));
		checkCudaErrors(hipMalloc((void**)&d_temp_dTopkScoreList, nbytes)  );
		checkCudaErrors(hipMemset(d_temp_dTopkScoreList, 0.0, nbytes));

		m_iTopkDocList[idQuery]=d_temp_iTopkDocList;
		m_dTopkScoreList[idQuery]=d_temp_dTopkScoreList;

		d_temp_iTopkDocList = NULL;
		d_temp_dTopkScoreList = NULL;

		checkCudaErrors( hipStreamCreate(&streams[idQuery]) );
	}
//	free(sizePostingList);
	//Processamento das Queries em Paralelo:

	double start;
	double end;

	int idQuery;
//	int* d_iTopkDocList;
//	float* d_dTopkScoreList;

//	int iMaxNumberInList;
	int iBlockNumber;
	dim3 threadsByBlock;
	dim3 blocksByGrid;

	int iTotalElementos; //= blocksByGrid.x * iTopk;
	int iProcessedListNumberbyBlock;// = iMergeNumberByBlock + 1;
	int iSkipTopKBetweenThreadBlocks;// = iProcessedListNumberbyBlock;
	int iSkipTopKBetweenMerges;// = 1;
	int exponent;// = 0;
	int exponent_b;// = 1;

	int* d_OrderQueryList;
	int n = 32;
	nbytes = n * sizeof(int);
	checkCudaErrors(hipMalloc((void**)&d_OrderQueryList, nbytes)  );
	checkCudaErrors(hipMemcpy(d_OrderQueryList, queryListOrder, nbytes, hipMemcpyHostToDevice));

	start = omp_get_wtime();

	threadsByBlock = dim3(iBlockSize, 1);
	blocksByGrid   = dim3(n, 1);

	d_iTopkDocList = m_iTopkDocList[0];
	d_dTopkScoreList = m_dTopkScoreList[0];

	matchWandParallel_VARIABLE_Batch_Block_Test<<<blocksByGrid, threadsByBlock,0,
																streams[queryListOrder[0]]>>>(d_iDocIdList, d_iFreqDocList,
																			   d_dUBList, d_dIdfList, d_iDocLenghtList,
																			   (short int*)m_iTermNumberByQuery, d_iTopkDocList,
																			   d_dTopkScoreList,
																			   dAverageDocumentLength,
																			   iTopk,dInitialThreshold,
																			   d_iQueryBatches, d_ptrInitPostingList,
																			   d_ptrQueryPositions, d_iDocNumberByTermList,
																			   d_OrderQueryList);



	for(int i=n; i < 500;i++){
		idQuery = queryListOrder[i];

		iMaxNumberInList = sizePostingList[i];
		iBlockNumber= (int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound));
		threadsByBlock = dim3(iBlockSize, 1);
		blocksByGrid   = dim3(iBlockNumber, 1);

		sizePostingList[i] = iBlockNumber;

		d_iTopkDocList = m_iTopkDocList[idQuery];
		d_dTopkScoreList = m_dTopkScoreList[idQuery];

		matchWandParallel_BATCH_2<<<blocksByGrid,threadsByBlock, 0, streams[idQuery]>>>
												 (d_iDocIdList, d_iFreqDocList,
												 d_dUBList, d_dIdfList, d_iDocLenghtList,
												 m_iTermNumberByQuery[idQuery],
												 d_iTopkDocList, d_dTopkScoreList,
												 dAverageDocumentLength,iBlockNumberRound,
												 iGlobalNumberRound,iTopk,dInitialThreshold,
												 d_iQueryBatches, d_ptrInitPostingList,
												 d_ptrQueryPositions, idQuery, d_iDocNumberByTermList);

	}

	for(int i=n; i < 500;i++){
		idQuery = queryListOrder[i];
		d_iTopkDocList = m_iTopkDocList[idQuery];
		d_dTopkScoreList = m_dTopkScoreList[idQuery];

		iProcessedListNumberbyBlock = iMergeNumberByBlock + 1;
		iSkipTopKBetweenThreadBlocks = iProcessedListNumberbyBlock;
		iSkipTopKBetweenMerges = 1;
		exponent = 0;
		exponent_b = 1;

		iBlockNumber= ceilf( sizePostingList[i]/(iProcessedListNumberbyBlock) );
		threadsByBlock = dim3(iBlockSize, 1);
		blocksByGrid   = dim3(iBlockNumber, 1);
		iTotalElementos = blocksByGrid.x * iTopk;

		while(blocksByGrid.x >= 1){
			mergeTopkLists_v3<<<blocksByGrid, iTopk, 0,streams[idQuery]>>>(d_dTopkScoreList, d_iTopkDocList,
															 iTopk, iMergeNumberByBlock,
															 iSkipTopKBetweenMerges,
															 iSkipTopKBetweenThreadBlocks,iTotalElementos);

			blocksByGrid.x = ceilf (blocksByGrid.x/(iProcessedListNumberbyBlock) );
			exponent ++;
			exponent_b ++;
			iSkipTopKBetweenMerges = pow(iProcessedListNumberbyBlock, exponent);
			iSkipTopKBetweenThreadBlocks = pow(iProcessedListNumberbyBlock, exponent_b);
		}

//	    checkCudaErrors(hipDeviceSynchronize());
//			int nbytes = iTopk * sizeof(int);
//			int *h_iTopkDocList = (int*) malloc(sizeof(int) * iTopk);
//			checkCudaErrors(hipMemcpy(h_iTopkDocList, d_iTopkDocList, nbytes, hipMemcpyDeviceToHost));
//
//			float *h_dTopkScoreList = (float*) malloc(sizeof(float) * iTopk);
//			nbytes = iTopk * sizeof(float);
//		    checkCudaErrors(hipMemcpy(h_dTopkScoreList, d_dTopkScoreList, nbytes, hipMemcpyDeviceToHost));
//
//		    checkCudaErrors(hipDeviceSynchronize());
//
//		//    for (int j = 0; j < 500; ++j) {
//		    	printf("\n--- %d Query -----\n",idQuery);
//				for (int i = 0; i < iTopk; ++i) {
//					printf("--- %d %.2f----",h_iTopkDocList[i],h_dTopkScoreList[i]);
//				}
//		//	}
//
//		    free(h_iTopkDocList);
//		    free(h_dTopkScoreList);


	}
	checkCudaErrors(hipDeviceSynchronize());

//    for(int idQuery=0; idQuery < iQueryTotalNumberByBatch;idQuery++){
//		checkCudaErrors(hipStreamSynchronize(streams[idQuery]));
//	}

	end = omp_get_wtime();
	printf("Batch - Work took %f s\n", (end - start));
	printf("Batch %d, %d, %d, %d, %.4f \n",TOP_K,SHAREDTHESHOLD,DOC_QUANTITY_IN_MEMORY,iGlobalNumberRound, (end - start));

	free(queryListOrder);
	free(sizePostingList);
	for (int idQuery = 0; idQuery < iQueryTotalNumberByBatch; ++idQuery) {
		d_iTopkDocList = m_iTopkDocList[idQuery];
		d_dTopkScoreList = m_dTopkScoreList[idQuery];

		checkCudaErrors(hipFree(d_iTopkDocList));
		checkCudaErrors(hipFree(d_dTopkScoreList));
	}

	checkCudaErrors(hipFree(d_OrderQueryList));
	checkCudaErrors(hipHostFree(m_iTopkDocList));
	checkCudaErrors(hipHostFree(m_dTopkScoreList));



	//-----------------------Destroy Stream Objects------------------------------
	for (int i = 0; i < iQueryTotalNumberByBatch; ++i)
		hipStreamDestroy(streams[i]);
	//---------------------------------------------------------------------------

	//-----------------------Free GPU Memory------------------------------
	checkCudaErrors(hipHostFree(m_iTermNumberByQuery));
	checkCudaErrors(hipHostFree(m_iQueryBatches));
	checkCudaErrors(hipHostFree(m_ptrQueryPositions));
	checkCudaErrors(hipHostFree(m_ptrInitPostingList));

	freeAllocationOfInvertedListInGPU(d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
									  d_dUBList, d_dIdfList, d_iDocNumberByTermList);
	//----------------------------------------------------------------------------------------------------------

	//------------------Release resources in CPU-------------------------------
	#ifdef DEBUG
		printf("Releasing resources in CPU... ");
	#endif

	for (int i = 0; i < iTermNumberInVocabulary; ++i) {
		free(h_iDocIdList[i]);
		free(h_iDocLenghtList[i]);
		free(h_iFreqList[i]);
	}

	for (int i = 0; i < iQueryTotalNumberByBatch; ++i) {
		free(h_iQueryBatches[i]);
	}

//	free(streams);
	free(h_iDocIdList);
	free(h_iDocLenghtList);
	free(h_iFreqList);
	free(h_iQueryBatches);
	free(h_dIdfList);
	free(h_dUBList);
	free(h_iTermNumberByQuery);
	free(h_iDocNumberByTermList);
	free(h_iTopkDocList);
	free(h_dTopkScoreList);
	//-------------------------------------------------------------------------
	#ifdef DEBUG
		printf("OK!\n");
		printf("Finish!\n");
	#endif


}


__host__ void queryBatchProcessingHost(int iTopk, int iBlockSize, int iBlockNumberRound,
										int iGlobalNumberRound, int iMergeNumberByBlock,
										int iQueryType){

//	int iPostingTotalNumber = 0;
//	int iMaxNumberInList = 0;
	int iTermNumberInVocabulary;
	unsigned long long docTotalNumber = 0;

	//----------Host Variables-------------
	int **h_iDocIdList, **h_iDocLenghtList;
	unsigned short int** h_iFreqList;

	float *h_dUBList, *h_dIdfList;

	int* h_iDocNumberByTermList;

	int* h_iTopkDocList = (int*) malloc(iTopk * sizeof(int));
	float* h_dTopkScoreList = (float*) malloc(iTopk * sizeof(float));

	float dAverageDocumentLength;

	int *h_iTermNumberByQuery; //Número de termo por query do batch de query
	int **h_iQueryBatches; //Batch de Queries
	int iQueryTotalNumberByBatch = 500; //Número de query por batch
	//-------------------------------------

	//---------Mapped Variables in Host and Device------------------
	int *m_iTermNumberByQuery; //Número de termo por query do batch de query - Os indices representam as queries
	int *m_iQueryBatches; //Batch de Queries - Um conjunto de termos - Os termos pertence a um conjunto de queries
	int *m_ptrQueryPositions; //A posição das queries na lista de batch de queries; essa contagem é a realizada pela contagem dos termos

	long long* m_ptrInitPostingList; //Considera que há uma única lista no acelerador  - Posição inicial das listas de postings dos termos

	int** m_iTopkDocList; // Topk
	float** m_dTopkScoreList; // Topk
	//---------------------------------------------------------------

	//----------Device Variables------------------------------------


	float dInitialThreshold = 0, *d_dUBList, *d_dIdfList;

	int *d_iDocIdList, *d_iDocLenghtList;
	unsigned short int* d_iFreqDocList;

	int* d_iDocNumberByTermList;
	int* d_iTopkDocList;
	float* d_dTopkScoreList;

	int *d_iTermNumberByQuery, *d_iQueryBatches, *d_ptrQueryPositions;
	long long *d_ptrInitPostingList;

	//-------------------------------------

	initializeInvertedIndex(&iTermNumberInVocabulary, &dAverageDocumentLength, &h_iDocIdList,
							&h_iDocLenghtList, &h_iFreqList, &h_dIdfList, &h_dUBList,
							&h_iDocNumberByTermList, &docTotalNumber);

	//-------------------GPU Pre-Configuration---------------------
	hipSetDevice(0);
	hipSetDeviceFlags(hipDeviceMapHost);

	//-------------------------------------------------------------

	//------------Allocating and Coping Pointer to Pointer in Device Memory--------------------------------------
	memoryAllocationOfInvertedIndexInGPU(docTotalNumber, iTermNumberInVocabulary,&d_iDocIdList,
										 &d_iDocLenghtList, &d_iFreqDocList,&d_dUBList,
									     &d_dIdfList, &d_iDocNumberByTermList);

	memoryCopyOfSingleQuery_To_Device(docTotalNumber, iTermNumberInVocabulary,h_iDocNumberByTermList,
		    						  d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
		    						  d_dUBList, d_dIdfList, d_iDocNumberByTermList, h_iDocIdList,
		    						  h_iDocLenghtList, h_iFreqList, h_dUBList, h_dIdfList);
	//----------------------------------------------------------------------------------------------------------


	initializeQueryBatch(&h_iQueryBatches, &iQueryTotalNumberByBatch, &h_iTermNumberByQuery);
//	iQueryTotalNumberByBatch=500;
	//Obtém a soma do número de termos de todas as queries (Esse valor pode ser pre-computado)
	int iTermTotalNumberOfBatch = 0;
	for (int i = 0; i < iQueryTotalNumberByBatch; ++i) {
		iTermTotalNumberOfBatch += h_iTermNumberByQuery[i];
	}
//	checkCudaErrors(hipMalloc((void**)&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_ptrQueryPositions, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_ptrInitPostingList, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch));
//
	checkCudaErrors(hipHostAlloc(&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch, hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_ptrQueryPositions, sizeof(int)*iQueryTotalNumberByBatch, hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_ptrInitPostingList, sizeof(long long)*iTermNumberInVocabulary, hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_iQueryBatches, sizeof(int)*iTermTotalNumberOfBatch, hipHostMallocWriteCombined | hipHostMallocMapped));

	long long position = 0;
	for (int i = 0; i < iQueryTotalNumberByBatch; i++) {
		m_ptrQueryPositions[i] = position;
		m_iTermNumberByQuery[i] = h_iTermNumberByQuery[i];
		for (int term = 0; term < h_iTermNumberByQuery[i]; term++) {
			m_iQueryBatches[position] = h_iQueryBatches[i][term];//Alinha as queries para o mapeamento com a GPU
			position++;
		}
	}

	position = 0;
	for (int i = 0; i < iTermNumberInVocabulary; ++i) {//Obtém a posição inicial de todas as listas invertidas
		m_ptrInitPostingList[i] = position;
		position += h_iDocNumberByTermList[i];
	}

	checkCudaErrors(hipHostGetDevicePointer(&d_iTermNumberByQuery, m_iTermNumberByQuery, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_ptrQueryPositions, m_ptrQueryPositions, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_iQueryBatches,m_iQueryBatches, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_ptrInitPostingList, m_ptrInitPostingList, 0 ) );

	//-------------------------------------

	hipStream_t *streams = (hipStream_t*) malloc(sizeof(hipStream_t) * iQueryTotalNumberByBatch);

	int nbytes = sizeof(int**) * iQueryTotalNumberByBatch;
	checkCudaErrors(hipHostAlloc((void **)&(m_iTopkDocList), nbytes, hipHostMallocWriteCombined | hipHostMallocMapped));

	nbytes = sizeof(float**) * iQueryTotalNumberByBatch;
	checkCudaErrors(hipHostAlloc((void **)&(m_dTopkScoreList), nbytes, hipHostMallocWriteCombined | hipHostMallocMapped));

	int *queryListOrder = (int*) malloc(sizeof(int)*iQueryTotalNumberByBatch);
	int *sizePostingList = (int*) malloc(sizeof(int)*iQueryTotalNumberByBatch);

	int iMaxNumberInList;
	int* d_temp_iTopkDocList;
	float* d_temp_dTopkScoreList;
	for (int idQuery = 0; idQuery < iQueryTotalNumberByBatch; ++idQuery) {
		iMaxNumberInList = 0;
		for (int idTerm = 0; idTerm < h_iTermNumberByQuery[idQuery]; idTerm++) {
				if(iMaxNumberInList < h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm] ] )
					iMaxNumberInList = h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm] ];
		}

		sizePostingList[idQuery] = iMaxNumberInList;
		queryListOrder[idQuery] =  idQuery;

		int auxq=0, auxtamanho=0;

		int i = idQuery;
		while(i > 0){
			if(sizePostingList[i-1] < sizePostingList[i]){
				auxtamanho = sizePostingList[i-1];
				auxq = queryListOrder[i-1];

				sizePostingList[i-1] = sizePostingList[i];
				queryListOrder[i-1] = queryListOrder[i];

				sizePostingList[i] = auxtamanho;
				queryListOrder[i] = auxq;
			}
			else
				break;
			i--;
		}

		nbytes = iTopk * sizeof(int) * ((int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound)));
		checkCudaErrors(hipMalloc((void**)&d_temp_iTopkDocList, nbytes)  );
		checkCudaErrors(hipMemset(d_temp_iTopkDocList, -1, nbytes));

		nbytes = iTopk * sizeof(float)  * ((int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound)));
		checkCudaErrors(hipMalloc((void**)&d_temp_dTopkScoreList, nbytes)  );
		checkCudaErrors(hipMemset(d_temp_dTopkScoreList, 0.0, nbytes));

		m_iTopkDocList[idQuery]=d_temp_iTopkDocList;
		m_dTopkScoreList[idQuery]=d_temp_dTopkScoreList;

		d_temp_iTopkDocList = NULL;
		d_temp_dTopkScoreList = NULL;

		checkCudaErrors( hipStreamCreate(&streams[idQuery]) );
	}
//	free(sizePostingList);
	//Processamento das Queries em Paralelo:

	double start;
	double end;
//	int idQuery;

	start = omp_get_wtime();
//#pragma omp parallel num_threads(1)  //private(idQuery)
//{


//	#pragma omp single
//	{
//		int idThread = omp_get_thread_num();
//		int portion = iQueryTotalNumberByBatch/omp_get_num_threads();
//		int pointStart = portion * idThread;

//		#pragma omp single
//		{
//			start = omp_get_wtime();
//		}
//	   #pragma omp taskgroup
//		{
//			#pragma omp parallel for num_threads(8)
			for(int i=0; i < 500;i++){
//			for (int idQuery = pointStart; idQuery < pointStart+portion ; ++idQuery) {
//				#pragma omp task //firstprivate(idQuery)
//				{
					int idQuery = queryListOrder[i];
//					if(idQuery != 8)
//						continue;
					int* d_iTopkDocList;
					float* d_dTopkScoreList;
					#ifdef DEBUG
						printf("idThreadCPU: %d\n",omp_get_thread_num());
						printf("idQuery: %d\n",idQuery);
						printf("idTerm: ");

						for (int i = 0; i < h_iTermNumberByQuery[idQuery]; ++i) {
							printf("%d ",h_iQueryBatches[idQuery][i]);
						}

						printf("\n");
					#endif

					#ifdef DEBUG
						int totalDoc = 0;
					#endif

					int iMaxNumberInList= sizePostingList[i];
//					for (int idTerm = 0; idTerm < h_iTermNumberByQuery[idQuery]; idTerm++) {
//						if(iMaxNumberInList < h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm] ] )
//							iMaxNumberInList = h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm] ];
//
//						#ifdef DEBUG
//							totalDoc +=h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm]];
//						#endif
//					}
					#ifdef DEBUG
						printf("Total of Docs: %d - Max of Doc: %d \n",totalDoc, iMaxNumberInList);
					#endif
					//------------Kernel Launch Configuration---------------------------------
					#ifdef DEBUG
						printf("Kernel Launch Configuration... \n");
					#endif

					int iBlockNumber= (int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound));
					dim3 threadsByBlock = dim3(iBlockSize, 1);
					dim3 blocksByGrid   = dim3(iBlockNumber, 1);

					#ifdef DEBUG
						printf("#threads by block: %i, #blocks: %i ... OK\n",threadsByBlock.x, blocksByGrid.x) ;
					#endif
					//-------------------------------------------------------------------------

					//------------Allocation memory of Top-k Lists---------------------------------
					#ifdef DEBUG
						printf("Allocation memory of Top-k Lists... ");
					#endif

					d_iTopkDocList = m_iTopkDocList[idQuery];
					d_dTopkScoreList = m_dTopkScoreList[idQuery];

					#ifdef DEBUG
						printf("OK\n");
					#endif
					//-------------------------------------------------------------------------

					//-----------Initializing the initial value of Threshold if it is necessary ---------------------
			//		if(iQueryType){
			//			setThresholdForANDQueryOnInvertedList(&dInitialThreshold, h_dUBList,
			//												  h_iQueryBatches[idQuery], h_iTermNumberByQuery[idQuery]);
			//		}
					//-----------------------------------------------------------------------------------------------

					//--------------------------Call to Kernels----------------------------
					callToBatchKernels(blocksByGrid, threadsByBlock, iMergeNumberByBlock, iTopk,
								  dAverageDocumentLength, dInitialThreshold, iBlockNumberRound,
								  iGlobalNumberRound, m_iTermNumberByQuery[idQuery], d_dUBList, d_dIdfList,
								  d_iDocIdList, d_iFreqDocList, d_iDocLenghtList,d_iTopkDocList,
								  d_dTopkScoreList, idQuery,
								  streams[idQuery], d_iQueryBatches,d_ptrInitPostingList,d_ptrQueryPositions,
								  d_iDocNumberByTermList);

					//----------------------------------------------------------------------------------------------------------
				}
//			}//Task
//
//		}//TaskGroup-1

//		#pragma omp taskgroup
//		{
//			for (int idQuery = pointStart; idQuery < pointStart+portion ; ++idQuery) {
//			#pragma omp for nowait
			for(int idQuery=0; idQuery < iQueryTotalNumberByBatch;idQuery++){
//				#pragma omp task //firstprivate(idQuery)
//				{
					hipStreamSynchronize(streams[idQuery]);
//				}
			}
//		}
//	}//Single
//}

	end = omp_get_wtime();
	printf("Batch - Work took %f s\n", (end - start));
	printf("Batch %d, %d, %d, %d, %.4f \n",TOP_K,SHAREDTHESHOLD,DOC_QUANTITY_IN_MEMORY,iGlobalNumberRound, (end - start));

	free(queryListOrder);
	free(sizePostingList);
	for (int idQuery = 0; idQuery < iQueryTotalNumberByBatch; ++idQuery) {
		d_iTopkDocList = m_iTopkDocList[idQuery];
		d_dTopkScoreList = m_dTopkScoreList[idQuery];

		checkCudaErrors(hipFree(d_iTopkDocList));
		checkCudaErrors(hipFree(d_dTopkScoreList));
	}

	checkCudaErrors(hipHostFree(m_iTopkDocList));
	checkCudaErrors(hipHostFree(m_dTopkScoreList));



	//-----------------------Destroy Stream Objects------------------------------
	for (int i = 0; i < iQueryTotalNumberByBatch; ++i)
		hipStreamDestroy(streams[i]);
	//---------------------------------------------------------------------------

	//-----------------------Free GPU Memory------------------------------
	checkCudaErrors(hipHostFree(m_iTermNumberByQuery));
	checkCudaErrors(hipHostFree(m_iQueryBatches));
	checkCudaErrors(hipHostFree(m_ptrQueryPositions));
	checkCudaErrors(hipHostFree(m_ptrInitPostingList));

	freeAllocationOfInvertedListInGPU(d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
									  d_dUBList, d_dIdfList, d_iDocNumberByTermList);
	//----------------------------------------------------------------------------------------------------------

	//------------------Release resources in CPU-------------------------------
	#ifdef DEBUG
		printf("Releasing resources in CPU... ");
	#endif

	for (int i = 0; i < iTermNumberInVocabulary; ++i) {
		free(h_iDocIdList[i]);
		free(h_iDocLenghtList[i]);
		free(h_iFreqList[i]);
	}

	for (int i = 0; i < iQueryTotalNumberByBatch; ++i) {
		free(h_iQueryBatches[i]);
	}

//	free(streams);
	free(h_iDocIdList);
	free(h_iDocLenghtList);
	free(h_iFreqList);
	free(h_iQueryBatches);
	free(h_dIdfList);
	free(h_dUBList);
	free(h_iTermNumberByQuery);
	free(h_iDocNumberByTermList);
	free(h_iTopkDocList);
	free(h_dTopkScoreList);
	//-------------------------------------------------------------------------
	#ifdef DEBUG
		printf("OK!\n");
		printf("Finish!\n");
	#endif
}

__host__ void queryBatchProcessingHost_2(int iTopk, int iBlockSize, int iBlockNumberRound,
										int iGlobalNumberRound, int iMergeNumberByBlock,
										int iQueryType){

//	int iPostingTotalNumber = 0;
//	int iMaxNumberInList = 0;
	int iTermNumberInVocabulary;
	unsigned long long docTotalNumber = 0;

	//----------Host Variables-------------
	int **h_iDocIdList, **h_iDocLenghtList;
	unsigned short int** h_iFreqList;

	float *h_dUBList, *h_dIdfList;

	int* h_iDocNumberByTermList;

	int* h_iTopkDocList = (int*) malloc(iTopk * sizeof(int));
	float* h_dTopkScoreList = (float*) malloc(iTopk * sizeof(float));

	float dAverageDocumentLength;

	int *h_iTermNumberByQuery; //Número de termo por query do batch de query
	int **h_iQueryBatches; //Batch de Queries
	int iQueryTotalNumberByBatch = 500; //Número de query por batch
	//-------------------------------------

	//---------Mapped Variables in Host and Device------------------
	int *m_iTermNumberByQuery; //Número de termo por query do batch de query - Os indices representam as queries
	int *m_iQueryBatches; //Batch de Queries - Um conjunto de termos - Os termos pertence a um conjunto de queries
	int *m_ptrQueryPositions; //A posição das queries na lista de batch de queries; essa contagem é a realizada pela contagem dos termos

	long long* m_ptrInitPostingList; //Considera que há uma única lista no acelerador  - Posição inicial das listas de postings dos termos

	int** m_iTopkDocList; // Topk
	float** m_dTopkScoreList; // Topk
	//---------------------------------------------------------------

	//----------Device Variables------------------------------------


	float dInitialThreshold = 0, *d_dUBList, *d_dIdfList;

	int *d_iDocIdList, *d_iDocLenghtList;
	unsigned short int* d_iFreqDocList;

	int* d_iDocNumberByTermList;
	int* d_iTopkDocList;
	float* d_dTopkScoreList;

	int *d_iTermNumberByQuery, *d_iQueryBatches, *d_ptrQueryPositions;
	long long *d_ptrInitPostingList;

	//-------------------------------------

	initializeInvertedIndex(&iTermNumberInVocabulary, &dAverageDocumentLength, &h_iDocIdList,
							&h_iDocLenghtList, &h_iFreqList, &h_dIdfList, &h_dUBList,
							&h_iDocNumberByTermList, &docTotalNumber);

	//-------------------GPU Pre-Configuration---------------------
	hipSetDevice(0);
	hipSetDeviceFlags(hipDeviceMapHost);

	//-------------------------------------------------------------

	//------------Allocating and Coping Pointer to Pointer in Device Memory--------------------------------------
	memoryAllocationOfInvertedIndexInGPU(docTotalNumber, iTermNumberInVocabulary,&d_iDocIdList,
										 &d_iDocLenghtList, &d_iFreqDocList,&d_dUBList,
									     &d_dIdfList, &d_iDocNumberByTermList);

	memoryCopyOfSingleQuery_To_Device(docTotalNumber, iTermNumberInVocabulary,h_iDocNumberByTermList,
		    						  d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
		    						  d_dUBList, d_dIdfList, d_iDocNumberByTermList, h_iDocIdList,
		    						  h_iDocLenghtList, h_iFreqList, h_dUBList, h_dIdfList);
	//----------------------------------------------------------------------------------------------------------


	initializeQueryBatch(&h_iQueryBatches, &iQueryTotalNumberByBatch, &h_iTermNumberByQuery);
//	iQueryTotalNumberByBatch=500;
	//Obtém a soma do número de termos de todas as queries (Esse valor pode ser pre-computado)
	int iTermTotalNumberOfBatch = 0;
	for (int i = 0; i < iQueryTotalNumberByBatch; ++i) {
		iTermTotalNumberOfBatch += h_iTermNumberByQuery[i];
	}
//	checkCudaErrors(hipMalloc((void**)&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_ptrQueryPositions, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_ptrInitPostingList, sizeof(int)*iQueryTotalNumberByBatch));
//	checkCudaErrors(hipMalloc((void**)&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch));
//
	checkCudaErrors(hipHostAlloc(&m_iTermNumberByQuery, sizeof(int)*iQueryTotalNumberByBatch, hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_ptrQueryPositions, sizeof(int)*iQueryTotalNumberByBatch, hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_ptrInitPostingList, sizeof(long long)*iTermNumberInVocabulary, hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_iQueryBatches, sizeof(int)*iTermTotalNumberOfBatch, hipHostMallocWriteCombined | hipHostMallocMapped));

	long long position = 0;
	for (int i = 0; i < iQueryTotalNumberByBatch; i++) {
		m_ptrQueryPositions[i] = position;
		m_iTermNumberByQuery[i] = h_iTermNumberByQuery[i];
		for (int term = 0; term < h_iTermNumberByQuery[i]; term++) {
			m_iQueryBatches[position] = h_iQueryBatches[i][term];//Alinha as queries para o mapeamento com a GPU
			position++;
		}
	}

	position = 0;
	for (int i = 0; i < iTermNumberInVocabulary; ++i) {//Obtém a posição inicial de todas as listas invertidas
		m_ptrInitPostingList[i] = position;
		position += h_iDocNumberByTermList[i];
	}

	checkCudaErrors(hipHostGetDevicePointer(&d_iTermNumberByQuery, m_iTermNumberByQuery, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_ptrQueryPositions, m_ptrQueryPositions, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_iQueryBatches,m_iQueryBatches, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_ptrInitPostingList, m_ptrInitPostingList, 0 ) );

	//-------------------------------------

	hipStream_t *streams = (hipStream_t*) malloc(sizeof(hipStream_t) * iQueryTotalNumberByBatch);

	int nbytes = sizeof(int**) * iQueryTotalNumberByBatch;
	checkCudaErrors(hipHostAlloc((void **)&(m_iTopkDocList), nbytes, hipHostMallocWriteCombined | hipHostMallocMapped));

	nbytes = sizeof(float**) * iQueryTotalNumberByBatch;
	checkCudaErrors(hipHostAlloc((void **)&(m_dTopkScoreList), nbytes, hipHostMallocWriteCombined | hipHostMallocMapped));

	int *queryListOrder = (int*) malloc(sizeof(int)*iQueryTotalNumberByBatch);
	int *sizePostingList = (int*) malloc(sizeof(int)*iQueryTotalNumberByBatch);

	int iMaxNumberInList;
	int* d_temp_iTopkDocList;
	float* d_temp_dTopkScoreList;
	for (int idQuery = 0; idQuery < iQueryTotalNumberByBatch; ++idQuery) {
		iMaxNumberInList = 0;
		for (int idTerm = 0; idTerm < h_iTermNumberByQuery[idQuery]; idTerm++) {
			if(iMaxNumberInList < h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm] ] )
				iMaxNumberInList = h_iDocNumberByTermList[ h_iQueryBatches[idQuery][idTerm] ];
		}

		sizePostingList[idQuery] = iMaxNumberInList;
		queryListOrder[idQuery] =  idQuery;

		int auxq=0, auxtamanho=0;

		int i = idQuery;
		while(i > 0){
			if(sizePostingList[i-1] < sizePostingList[i]){
				auxtamanho = sizePostingList[i-1];
				auxq = queryListOrder[i-1];

				sizePostingList[i-1] = sizePostingList[i];
				queryListOrder[i-1] = queryListOrder[i];

				sizePostingList[i] = auxtamanho;
				queryListOrder[i] = auxq;
			}
			else
				break;
			i--;
		}

		nbytes = iTopk * sizeof(int) * ((int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound)));
		checkCudaErrors(hipMalloc((void**)&d_temp_iTopkDocList, nbytes)  );
		checkCudaErrors(hipMemset(d_temp_iTopkDocList, -1, nbytes));

		nbytes = iTopk * sizeof(float)  * ((int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound)));
		checkCudaErrors(hipMalloc((void**)&d_temp_dTopkScoreList, nbytes)  );
		checkCudaErrors(hipMemset(d_temp_dTopkScoreList, 0.0, nbytes));

		m_iTopkDocList[idQuery]=d_temp_iTopkDocList;
		m_dTopkScoreList[idQuery]=d_temp_dTopkScoreList;

		d_temp_iTopkDocList = NULL;
		d_temp_dTopkScoreList = NULL;

		checkCudaErrors( hipStreamCreate(&streams[idQuery]) );
	}
//	free(sizePostingList);
	//Processamento das Queries em Paralelo:

	double start;
	double end;

	int idQuery;
//	int* d_iTopkDocList;
//	float* d_dTopkScoreList;

//	int iMaxNumberInList;
	int iBlockNumber;
	dim3 threadsByBlock;
	dim3 blocksByGrid;

	int iTotalElementos; //= blocksByGrid.x * iTopk;
	int iProcessedListNumberbyBlock;// = iMergeNumberByBlock + 1;
	int iSkipTopKBetweenThreadBlocks;// = iProcessedListNumberbyBlock;
	int iSkipTopKBetweenMerges;// = 1;
	int exponent;// = 0;
	int exponent_b;// = 1;

	start = omp_get_wtime();

	for(int i=0; i < iQueryTotalNumberByBatch;i++){
		idQuery = queryListOrder[i];

		iMaxNumberInList = sizePostingList[i];
		iBlockNumber= (int) ceil((float) iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * 1 * iGlobalNumberRound));
		threadsByBlock = dim3(iBlockSize, 1);
		blocksByGrid   = dim3(iBlockNumber, 1);

		sizePostingList[i] = iBlockNumber;

		d_iTopkDocList = m_iTopkDocList[idQuery];
		d_dTopkScoreList = m_dTopkScoreList[idQuery];

		matchWandParallel_BATCH_2<<<blocksByGrid,threadsByBlock, 0, streams[idQuery]>>>
												 (d_iDocIdList, d_iFreqDocList,
												 d_dUBList, d_dIdfList, d_iDocLenghtList,
												 m_iTermNumberByQuery[idQuery],
												 d_iTopkDocList, d_dTopkScoreList,
												 dAverageDocumentLength,iBlockNumberRound,
												 iGlobalNumberRound,iTopk,dInitialThreshold,
												 d_iQueryBatches, d_ptrInitPostingList,
												 d_ptrQueryPositions, idQuery, d_iDocNumberByTermList);

	}

	for(int i=0; i < iQueryTotalNumberByBatch;i++){
		idQuery = queryListOrder[i];
		d_iTopkDocList = m_iTopkDocList[idQuery];
		d_dTopkScoreList = m_dTopkScoreList[idQuery];

		iProcessedListNumberbyBlock = iMergeNumberByBlock + 1;
		iSkipTopKBetweenThreadBlocks = iProcessedListNumberbyBlock;
		iSkipTopKBetweenMerges = 1;
		exponent = 0;
		exponent_b = 1;

		iBlockNumber= ceilf( sizePostingList[i]/(iProcessedListNumberbyBlock) );
		threadsByBlock = dim3(iBlockSize, 1);
		blocksByGrid   = dim3(iBlockNumber, 1);
		iTotalElementos = blocksByGrid.x * iTopk;

		while(blocksByGrid.x >= 1){
			mergeTopkLists_v3<<<blocksByGrid, iTopk, 0,streams[idQuery]>>>(d_dTopkScoreList, d_iTopkDocList,
															 iTopk, iMergeNumberByBlock,
															 iSkipTopKBetweenMerges,
															 iSkipTopKBetweenThreadBlocks,iTotalElementos);

			blocksByGrid.x = ceilf (blocksByGrid.x/(iProcessedListNumberbyBlock) );
			exponent ++;
			exponent_b ++;
			iSkipTopKBetweenMerges = pow(iProcessedListNumberbyBlock, exponent);
			iSkipTopKBetweenThreadBlocks = pow(iProcessedListNumberbyBlock, exponent_b);
		}

//	    checkCudaErrors(hipDeviceSynchronize());
//			int nbytes = iTopk * sizeof(int);
//			int *h_iTopkDocList = (int*) malloc(sizeof(int) * iTopk);
//			checkCudaErrors(hipMemcpy(h_iTopkDocList, d_iTopkDocList, nbytes, hipMemcpyDeviceToHost));
//
//			float *h_dTopkScoreList = (float*) malloc(sizeof(float) * iTopk);
//			nbytes = iTopk * sizeof(float);
//		    checkCudaErrors(hipMemcpy(h_dTopkScoreList, d_dTopkScoreList, nbytes, hipMemcpyDeviceToHost));
//
//		    checkCudaErrors(hipDeviceSynchronize());
//
//		//    for (int j = 0; j < 500; ++j) {
//		    	printf("\n--- %d Query -----\n",idQuery);
//				for (int i = 0; i < iTopk; ++i) {
//					printf("--- %d %.2f----",h_iTopkDocList[i],h_dTopkScoreList[i]);
//				}
//		//	}
//
//		    free(h_iTopkDocList);
//		    free(h_dTopkScoreList);


	}
    checkCudaErrors(hipDeviceSynchronize());

//    for(int idQuery=0; idQuery < iQueryTotalNumberByBatch;idQuery++){
//		checkCudaErrors(hipStreamSynchronize(streams[idQuery]));
//	}

	end = omp_get_wtime();
	printf("Batch - Work took %f s\n", (end - start));
	printf("Batch %d, %d, %d, %d, %.4f \n",TOP_K,SHAREDTHESHOLD,DOC_QUANTITY_IN_MEMORY,iGlobalNumberRound, (end - start));

	free(queryListOrder);
	free(sizePostingList);
	for (int idQuery = 0; idQuery < iQueryTotalNumberByBatch; ++idQuery) {
		d_iTopkDocList = m_iTopkDocList[idQuery];
		d_dTopkScoreList = m_dTopkScoreList[idQuery];

		checkCudaErrors(hipFree(d_iTopkDocList));
		checkCudaErrors(hipFree(d_dTopkScoreList));
	}

	checkCudaErrors(hipHostFree(m_iTopkDocList));
	checkCudaErrors(hipHostFree(m_dTopkScoreList));



	//-----------------------Destroy Stream Objects------------------------------
	for (int i = 0; i < iQueryTotalNumberByBatch; ++i)
		hipStreamDestroy(streams[i]);
	//---------------------------------------------------------------------------

	//-----------------------Free GPU Memory------------------------------
	checkCudaErrors(hipHostFree(m_iTermNumberByQuery));
	checkCudaErrors(hipHostFree(m_iQueryBatches));
	checkCudaErrors(hipHostFree(m_ptrQueryPositions));
	checkCudaErrors(hipHostFree(m_ptrInitPostingList));

	freeAllocationOfInvertedListInGPU(d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
									  d_dUBList, d_dIdfList, d_iDocNumberByTermList);
	//----------------------------------------------------------------------------------------------------------

	//------------------Release resources in CPU-------------------------------
	#ifdef DEBUG
		printf("Releasing resources in CPU... ");
	#endif

	for (int i = 0; i < iTermNumberInVocabulary; ++i) {
		free(h_iDocIdList[i]);
		free(h_iDocLenghtList[i]);
		free(h_iFreqList[i]);
	}

	for (int i = 0; i < iQueryTotalNumberByBatch; ++i) {
		free(h_iQueryBatches[i]);
	}

//	free(streams);
	free(h_iDocIdList);
	free(h_iDocLenghtList);
	free(h_iFreqList);
	free(h_iQueryBatches);
	free(h_dIdfList);
	free(h_dUBList);
	free(h_iTermNumberByQuery);
	free(h_iDocNumberByTermList);
	free(h_iTopkDocList);
	free(h_dTopkScoreList);
	//-------------------------------------------------------------------------
	#ifdef DEBUG
		printf("OK!\n");
		printf("Finish!\n");
	#endif
}



__host__ void queryBatchProcessingHost_ByBlock(int iTopk, int iBlockSize, int iBlockNumberRound,
											   int iGlobalNumberRound, int iMergeNumberByBlock,
											   int iQueryType, int iBatchSize){

//	int iPostingTotalNumber = 0;
//	int iMaxNumberInList = 0;
	int iTermNumberInVocabulary;
	unsigned long long docTotalNumber = 0;

	//----------Host Variables-------------
	int **h_iDocIdList, **h_iDocLenghtList;
	unsigned short int** h_iFreqList;


	float *h_dUBList, *h_dIdfList;

	int* h_iDocNumberByTermList;

//	int* h_iTopkDocList = (int*) malloc(iTopk * sizeof(int) * iBatchSize);
//	float* h_dTopkScoreList = (float*) malloc(iTopk * sizeof(float) * iBatchSize);

	float dAverageDocumentLength;

	int *h_iTermNumberByQuery; //Número de termo por query do batch de query
	int **h_iQueryBatches; //Batch de Queries contém as queries
	int iQueryTotalNumberByBatch=500; //Número total de query
	//-------------------------------------

	//---------Mapped Variables in Host and Device------------------
	short int *m_iTermNumberByQuery; //Número de termo por query do batch de query - Os indices representam as queries
	int *m_iQueryBatches; //Batch de Queries - Um conjunto de termos - Os termos pertence a um conjunto de queries
	int *m_ptrQueryPositions; //A posição das queries na lista de batch de queries; essa contagem é a realizada pela contagem dos termos

	long long* m_ptrInitPostingList; //Contém as listas de postings de todos os termos do vocabulário - Considera que há uma única lista no acelerador

//	int* m_iTopkDocList;
//	float* m_dTopkScoreList;
	//---------------------------------------------------------------

	//----------Device Variables------------------------------------


	float dInitialThreshold = 0, *d_dUBList, *d_dIdfList;

	int *d_iDocIdList, *d_iDocLenghtList;
	unsigned short int* d_iFreqDocList;

	int* d_iDocNumberByTermList;
	int* d_iTopkDocList;
	float* d_dTopkScoreList;

	int *d_iTermNumberByQuery, *d_iQueryBatches, *d_ptrQueryPositions;
	long long *d_ptrInitPostingList;

	//-------------------------------------

	initializeInvertedIndex(&iTermNumberInVocabulary, &dAverageDocumentLength, &h_iDocIdList,
							&h_iDocLenghtList, &h_iFreqList, &h_dIdfList, &h_dUBList,
							&h_iDocNumberByTermList, &docTotalNumber);

	//-------------------GPU Pre-Configuration---------------------
	hipSetDevice(0);
	hipSetDeviceFlags(hipDeviceMapHost);
	//-------------------------------------------------------------

	//------------Allocating and Coping Pointer to Pointer in Device Memory--------------------------------------
	memoryAllocationOfInvertedIndexInGPU(docTotalNumber, iTermNumberInVocabulary,&d_iDocIdList,
										 &d_iDocLenghtList, &d_iFreqDocList,&d_dUBList,
									     &d_dIdfList, &d_iDocNumberByTermList);

	memoryCopyOfSingleQuery_To_Device(docTotalNumber, iTermNumberInVocabulary,h_iDocNumberByTermList,
		    						  d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
		    						  d_dUBList, d_dIdfList, d_iDocNumberByTermList, h_iDocIdList,
		    						  h_iDocLenghtList, h_iFreqList, h_dUBList, h_dIdfList);
	//----------------------------------------------------------------------------------------------------------


	initializeQueryBatch(&h_iQueryBatches, &iQueryTotalNumberByBatch, &h_iTermNumberByQuery);
	//Obtém o número total de termos nas queries (Esse valor pode ser pre-computado)
	int iTermTotalNumberOfBatch = 0;
	for (int i = 0; i < iQueryTotalNumberByBatch; ++i) {
		iTermTotalNumberOfBatch += h_iTermNumberByQuery[i];
	}

	checkCudaErrors(hipHostAlloc(&m_iTermNumberByQuery, sizeof(short int)*iQueryTotalNumberByBatch, hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_ptrQueryPositions, sizeof(int)*iQueryTotalNumberByBatch, hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_ptrInitPostingList, sizeof(long long)*iTermNumberInVocabulary, hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc(&m_iQueryBatches, sizeof(int)*iTermTotalNumberOfBatch, hipHostMallocWriteCombined | hipHostMallocMapped));

	long long position = 0;
	for (int i = 0; i < iQueryTotalNumberByBatch; i++) {
			printf("\n--- %d Query ",i);
			m_ptrQueryPositions[i] = position;
			m_iTermNumberByQuery[i] = h_iTermNumberByQuery[i];
		for (int term = 0; term < h_iTermNumberByQuery[i]; term++) {
			printf("%d ",h_iQueryBatches[i][term]);
			m_iQueryBatches[position] = h_iQueryBatches[i][term];

			position++;
		}
	}

	position = 0;
	for (int i = 0; i < iTermNumberInVocabulary; ++i) {

//		#ifdef DEBUG
//			if (i == 4558 || i == 2515)
//				printf("STOP");
//		#endif
		m_ptrInitPostingList[i] = position;
		position += h_iDocNumberByTermList[i];
	}

	checkCudaErrors(hipHostGetDevicePointer(&d_iTermNumberByQuery, m_iTermNumberByQuery, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_ptrQueryPositions, m_ptrQueryPositions, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_iQueryBatches,m_iQueryBatches, 0 ) );
	checkCudaErrors(hipHostGetDevicePointer(&d_ptrInitPostingList, m_ptrInitPostingList, 0 ) );

	//------------Allocation memory of Top-k Lists---------------------------------
	#ifdef DEBUG
		printf("Allocation memory of Top-k Lists... ");
	#endif


	int nbytes = iTopk * sizeof(int) * iBatchSize;
	checkCudaErrors(hipMalloc((void**)&d_iTopkDocList, nbytes)  );
	checkCudaErrors(hipMemset(d_iTopkDocList, -1, nbytes));

	nbytes = iTopk * sizeof(float) * iBatchSize;
	checkCudaErrors(hipMalloc((void**)&d_dTopkScoreList, nbytes)  );
	checkCudaErrors(hipMemset(d_dTopkScoreList, 0.0, nbytes));

//	int nbytes = iTopk * sizeof(int) * iBatchSize;
//	checkCudaErrors(hipHostAlloc((void **) &m_iTopkDocList, nbytes,hipHostMallocWriteCombined | hipHostMallocMapped));
//	memset(m_iTopkDocList, -1, nbytes);
//
//	nbytes = iTopk * sizeof(float) * iBatchSize;
//	checkCudaErrors(hipHostAlloc((void **) &m_dTopkScoreList, nbytes, hipHostMallocWriteCombined | hipHostMallocMapped));
//	memset(m_dTopkScoreList, 0.0, nbytes);
//
//	checkCudaErrors(hipHostGetDevicePointer(&d_iTopkDocList, m_iTopkDocList, 0 ) );
//	checkCudaErrors(hipHostGetDevicePointer(&d_dTopkScoreList,m_dTopkScoreList, 0 ) );

	#ifdef DEBUG
		printf("OK\n");
	#endif

	//-------------------------------------------------------------------------

	//-------------------------------------

//	hipStream_t *streams = (hipStream_t*) malloc(sizeof(hipStream_t) * iQueryTotalNumberByBatch);


	//Processamento das Queries em Paralelo:
	//(1) Processar as 1000 queries em um único round (todas estarão no Buffer do acelerador)
	//1 2 3 4 5 6 7 8 9 10 11 12 13 14 15
//	for (int idQuery = 0; idQuery < h_iQueryNumber; ++idQuery) {


//		checkCudaErrors( hipStreamCreate(&streams[idQuery]) );

//		iMaxNumberInList = 0;
//		for (int idTerm = 0; idTerm < h_iTermNumberByQuery[idQuery]; idTerm++) {
//			if(iMaxNumberInList < h_iDocNumberByTermList[ m_iQueryBatches[ m_ptrQueryPositions[idQuery] +idTerm] ] )
//				iMaxNumberInList = h_iDocNumberByTermList[ m_iQueryBatches[ m_ptrQueryPositions[idQuery] +idTerm] ];
//		}

		//------------Kernel Launch Configuration---------------------------------
		#ifdef DEBUG
			printf("Kernel Launch Configuration... \n");
		#endif

		int iBlockNumber = iBatchSize;//(int) ceil((float)iMaxNumberInList/(DOC_QUANTITY_IN_MEMORY * iBlockNumberRound * iGlobalNumberRound));
		dim3 threadsByBlock = dim3(iBlockSize, 1);
		dim3 blocksByGrid   = dim3(iBlockNumber, 1);

		#ifdef DEBUG
			printf("#threads by block: %i, #blocks: %i ... OK\n",threadsByBlock.x, blocksByGrid.x) ;
		#endif
		//-------------------------------------------------------------------------


		//-----------Initializing the initial value of Threshold if it is necessary ---------------------
//		if(iQueryType){
//			setThresholdForANDQueryOnInvertedList(&dInitialThreshold, h_dUBList,
//												  h_iQueryBatches[idQuery], h_iTermNumberByQuery[idQuery]);
//		}
		//-----------------------------------------------------------------------------------------------

		//--------------------------Call to Kernels----------------------------
//		callToBatchKernels_byBlock(blocksByGrid, threadsByBlock, iMergeNumberByBlock, iTopk,
//								   dAverageDocumentLength, dInitialThreshold, iBlockNumberRound,
//								   iGlobalNumberRound, m_iTermNumberByQuery, d_dUBList, d_dIdfList,
//								   d_iDocIdList, d_iFreqDocList, d_iDocLenghtList,d_iTopkDocList,
//								   d_dTopkScoreList, d_iQueryBatches,d_ptrPostingPositions,
//								   d_ptrQueryPositions, d_iDocNumberByTermList);

		callToBatchKernels_byBlock(blocksByGrid, threadsByBlock, iMergeNumberByBlock, iTopk,
							  dAverageDocumentLength, dInitialThreshold,
							  m_iTermNumberByQuery, d_dUBList, d_dIdfList,
							  d_iDocIdList, d_iFreqDocList, d_iDocLenghtList,d_iTopkDocList,
							  d_dTopkScoreList, d_iQueryBatches,d_ptrInitPostingList,d_ptrQueryPositions,
							  d_iDocNumberByTermList);

//		dim3 blocksByGrid, dim3 threadsByBlock, int iMergeNumberByBlock, int iTopk,
//										float dAverageDocumentLength, float dInitialThreshold,
//										short int* iTermNumberInQuery,
//										float *d_dUBList, float *d_dIdfList, int *d_iDocIdList,
//										unsigned short int *d_iFreqDocList, int *d_iDocLenghtList,
//										int *d_iTopkDocList, float *d_dTopkScoreList,
//										int *iQueryTerms,long long* d_ptrPostingPositions,
//										int* d_ptrQueryPositions, int *d_iDocNumberByTermList)

		//----------------------------------------------------------------------------------------------------------
//	}


//   checkCudaErrors(hipHostFree(m_iTopkDocList));
//   checkCudaErrors(hipHostFree(m_dTopkScoreList));

//	//-----------------------Destroy Stream Objects------------------------------
//	for (int i = 0; i < h_iQueryNumber; ++i)
//		hipStreamDestroy(streams[i]);
//	//---------------------------------------------------------------------------

	//-----------------------Free GPU Memory------------------------------
	checkCudaErrors(hipHostFree(m_iTermNumberByQuery));
	checkCudaErrors(hipHostFree(m_iQueryBatches));
	checkCudaErrors(hipHostFree(m_ptrQueryPositions));
	checkCudaErrors(hipHostFree(m_ptrInitPostingList));

	checkCudaErrors(hipFree(d_iTopkDocList));
	checkCudaErrors(hipFree(d_dTopkScoreList));

	freeAllocationOfInvertedListInGPU(d_iDocIdList, d_iDocLenghtList, d_iFreqDocList,
									  d_dUBList, d_dIdfList, d_iDocNumberByTermList);
	//----------------------------------------------------------------------------------------------------------

	//------------------Release resources in CPU-------------------------------
	#ifdef DEBUG
		printf("Releasing resources in CPU... ");
	#endif
	for (int i = 0; i < iTermNumberInVocabulary; ++i) {
		free(h_iDocIdList[i]);
		free(h_iDocLenghtList[i]);
		free(h_iFreqList[i]);
	}

	for (int i = 0; i < iQueryTotalNumberByBatch; ++i) {
		free(h_iQueryBatches[i]);
	}

//	free(streams);
	free(h_iQueryBatches);
	free(h_iDocIdList);
	free(h_iDocLenghtList);
	free(h_iFreqList);
	free(h_dIdfList);
	free(h_dUBList);
	free(h_iTermNumberByQuery);
	free(h_iDocNumberByTermList);

//	free(h_iTopkDocList);
//	free(h_dTopkScoreList);

	//-------------------------------------------------------------------------
	#ifdef DEBUG
		printf("OK!\n");
	#endif

	printf("Finish!\n");
}




